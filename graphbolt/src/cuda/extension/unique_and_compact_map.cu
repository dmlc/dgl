#include "hip/hip_runtime.h"
/**
 *   Copyright (c) 2023, GT-TDAlab (Muhammed Fatih Balin & Umit V. Catalyurek)
 *   All rights reserved.
 *
 *   Licensed under the Apache License, Version 2.0 (the "License");
 *   you may not use this file except in compliance with the License.
 *   You may obtain a copy of the License at
 *
 *       http://www.apache.org/licenses/LICENSE-2.0
 *
 *   Unless required by applicable law or agreed to in writing, software
 *   distributed under the License is distributed on an "AS IS" BASIS,
 *   WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *   See the License for the specific language governing permissions and
 *   limitations under the License.
 *
 * @file cuda/unique_and_compact_map.cu
 * @brief Unique and compact operator implementation on CUDA using hash table.
 */
#include <graphbolt/cuda_ops.h>
#include <thrust/gather.h>

#include <cuco/static_map.cuh>
#include <cuda/std/atomic>
#include <numeric>

#include "../common.h"
#include "../utils.h"
#include "./unique_and_compact.h"

namespace graphbolt {
namespace ops {

// Support graphs with up to 2^kNodeIdBits nodes.
constexpr int kNodeIdBits = 40;

template <typename index_t, typename map_t>
__global__ void _InsertAndSetMinBatched(
    const int64_t num_edges, const int32_t* const indexes, index_t** pointers,
    const int64_t* const offsets, map_t map) {
  int64_t i = blockIdx.x * blockDim.x + threadIdx.x;
  const int stride = gridDim.x * blockDim.x;

  while (i < num_edges) {
    const int64_t tensor_index = indexes[i];
    const auto tensor_offset = i - offsets[tensor_index];
    const int64_t node_id = pointers[tensor_index][tensor_offset];
    const auto batch_index = tensor_index / 2;
    const int64_t key = node_id | (batch_index << kNodeIdBits);

    auto [slot, is_new_key] = map.insert_and_find(cuco::pair{key, i});

    if (!is_new_key) {
      auto ref = ::cuda::atomic_ref<int64_t, ::cuda::thread_scope_device>{
          slot->second};
      ref.fetch_min(i, ::cuda::memory_order_relaxed);
    }

    i += stride;
  }
}

template <typename index_t, typename map_t>
__global__ void _IsInsertedBatched(
    const int64_t num_edges, const int32_t* const indexes, index_t** pointers,
    const int64_t* const offsets, map_t map, int64_t* valid) {
  int64_t i = blockIdx.x * blockDim.x + threadIdx.x;
  const int stride = gridDim.x * blockDim.x;

  while (i < num_edges) {
    const int64_t tensor_index = indexes[i];
    const auto tensor_offset = i - offsets[tensor_index];
    const int64_t node_id = pointers[tensor_index][tensor_offset];
    const auto batch_index = tensor_index / 2;
    const int64_t key = node_id | (batch_index << kNodeIdBits);

    auto slot = map.find(key);
    valid[i] = slot->second == i;

    i += stride;
  }
}

template <typename index_t, typename map_t>
__global__ void _GetInsertedBatched(
    const int64_t num_edges, const int32_t* const indexes, index_t** pointers,
    const int64_t* const offsets, map_t map, const int64_t* const valid,
    index_t* unique_ids) {
  int64_t i = blockIdx.x * blockDim.x + threadIdx.x;
  const int stride = gridDim.x * blockDim.x;

  while (i < num_edges) {
    const auto valid_i = valid[i];

    if (valid_i + 1 == valid[i + 1]) {
      const int64_t tensor_index = indexes[i];
      const auto tensor_offset = i - offsets[tensor_index];
      const int64_t node_id = pointers[tensor_index][tensor_offset];
      const auto batch_index = tensor_index / 2;
      const int64_t key = node_id | (batch_index << kNodeIdBits);

      auto slot = map.find(key);
      const auto batch_offset = offsets[batch_index * 2];
      const auto new_id = valid_i - valid[batch_offset];
      unique_ids[valid_i] = node_id;
      slot->second = new_id;
    }

    i += stride;
  }
}

template <typename index_t, typename map_t>
__global__ void _MapIdsBatched(
    const int num_batches, const int64_t num_edges,
    const int32_t* const indexes, index_t** pointers,
    const int64_t* const offsets, map_t map, index_t* mapped_ids) {
  int64_t i = blockIdx.x * blockDim.x + threadIdx.x;
  const int stride = gridDim.x * blockDim.x;

  while (i < num_edges) {
    const int64_t tensor_index = indexes[i];
    int64_t batch_index;

    if (tensor_index >= 2 * num_batches) {
      batch_index = tensor_index - 2 * num_batches;
    } else if (tensor_index & 1) {
      batch_index = tensor_index / 2;
    } else {
      batch_index = -1;
    }

    // Only map src or dst ids.
    if (batch_index >= 0) {
      const auto tensor_offset = i - offsets[tensor_index];
      const int64_t node_id = pointers[tensor_index][tensor_offset];
      const int64_t key = node_id | (batch_index << kNodeIdBits);

      auto slot = map.find(key);
      mapped_ids[i] = slot->second;
    }

    i += stride;
  }
}

std::vector<std::tuple<torch::Tensor, torch::Tensor, torch::Tensor> >
UniqueAndCompactBatchedHashMapBased(
    const std::vector<torch::Tensor>& src_ids,
    const std::vector<torch::Tensor>& dst_ids,
    const std::vector<torch::Tensor>& unique_dst_ids) {
  auto allocator = cuda::GetAllocator();
  auto stream = cuda::GetCurrentStream();
  auto scalar_type = src_ids.at(0).scalar_type();
  constexpr int BLOCK_SIZE = 512;
  const auto num_batches = src_ids.size();
  static_assert(
      sizeof(std::ptrdiff_t) == sizeof(int64_t),
      "Need to be compiled on a 64-bit system.");
  constexpr int batch_id_bits = sizeof(int64_t) * 8 - 1 - kNodeIdBits;
  TORCH_CHECK(
      num_batches <= (1 << batch_id_bits),
      "UniqueAndCompactBatched supports a batch size of up to ",
      1 << batch_id_bits);
  return AT_DISPATCH_INDEX_TYPES(
      scalar_type, "unique_and_compact", ([&] {
        // For 2 batches of inputs, stores the input tensor pointers in the
        // unique_dst, src, unique_dst, src, dst, dst order. Since there are
        // 3 * num_batches input tensors, we need the first 3 * num_batches to
        // store the input tensor pointers. Then, we store offsets in the rest
        // of the 3 * num_batches + 1 space as if they were stored contiguously.
        auto pointers_and_offsets = torch::empty(
            6 * num_batches + 1,
            c10::TensorOptions().dtype(torch::kInt64).pinned_memory(true));
        // Points to the input tensor pointers.
        auto pointers_ptr =
            reinterpret_cast<index_t**>(pointers_and_offsets.data_ptr());
        // Points to the input tensor storage logical offsets.
        auto offsets_ptr =
            pointers_and_offsets.data_ptr<int64_t>() + 3 * num_batches;
        for (std::size_t i = 0; i < num_batches; i++) {
          pointers_ptr[2 * i] = unique_dst_ids.at(i).data_ptr<index_t>();
          offsets_ptr[2 * i] = unique_dst_ids[i].size(0);
          pointers_ptr[2 * i + 1] = src_ids.at(i).data_ptr<index_t>();
          offsets_ptr[2 * i + 1] = src_ids[i].size(0);
          pointers_ptr[2 * num_batches + i] = dst_ids.at(i).data_ptr<index_t>();
          offsets_ptr[2 * num_batches + i] = dst_ids[i].size(0);
        }
        // Finish computing the offsets by taking a cumulative sum.
        std::exclusive_scan(
            offsets_ptr, offsets_ptr + 3 * num_batches + 1, offsets_ptr, 0ll);
        // Device version of the tensors defined above. We store the information
        // initially on the CPU, which are later copied to the device.
        auto pointers_and_offsets_dev = torch::empty(
            pointers_and_offsets.size(0),
            src_ids[0].options().dtype(pointers_and_offsets.scalar_type()));
        auto offsets_dev = pointers_and_offsets_dev.slice(0, 3 * num_batches);
        auto pointers_dev_ptr =
            reinterpret_cast<index_t**>(pointers_and_offsets_dev.data_ptr());
        auto offsets_dev_ptr = offsets_dev.data_ptr<int64_t>();
        CUDA_CALL(hipMemcpyAsync(
            pointers_dev_ptr, pointers_ptr,
            sizeof(int64_t) * pointers_and_offsets.size(0),
            hipMemcpyHostToDevice, stream));
        auto indexes = ExpandIndptrImpl(
            offsets_dev, torch::kInt32, torch::nullopt,
            offsets_ptr[3 * num_batches]);
        cuco::static_map map{
            offsets_ptr[2 * num_batches],
            0.5,  // load_factor
            cuco::empty_key{static_cast<int64_t>(-1)},
            cuco::empty_value{static_cast<int64_t>(-1)},
            {},
            cuco::linear_probing<1, cuco::default_hash_function<int64_t> >{},
            {},
            {},
            cuda::CUDAWorkspaceAllocator<cuco::pair<int64_t, int64_t> >{},
            cuco::cuda_stream_ref{stream},
        };
        C10_CUDA_KERNEL_LAUNCH_CHECK();  // Check the map constructor's success.
        const dim3 block(BLOCK_SIZE);
        const dim3 grid(
            (offsets_ptr[2 * num_batches] + BLOCK_SIZE - 1) / BLOCK_SIZE);
        CUDA_KERNEL_CALL(
            _InsertAndSetMinBatched, grid, block, 0,
            offsets_ptr[2 * num_batches], indexes.data_ptr<int32_t>(),
            pointers_dev_ptr, offsets_dev_ptr, map.ref(cuco::insert_and_find));
        auto valid = torch::empty(
            offsets_ptr[2 * num_batches] + 1,
            src_ids[0].options().dtype(torch::kInt64));
        CUDA_KERNEL_CALL(
            _IsInsertedBatched, grid, block, 0, offsets_ptr[2 * num_batches],
            indexes.data_ptr<int32_t>(), pointers_dev_ptr, offsets_dev_ptr,
            map.ref(cuco::find), valid.data_ptr<int64_t>());
        valid = ExclusiveCumSum(valid);
        auto unique_ids_offsets = torch::empty(
            num_batches + 1,
            c10::TensorOptions().dtype(torch::kInt64).pinned_memory(true));
        auto unique_ids_offsets_ptr = unique_ids_offsets.data_ptr<int64_t>();
        for (int64_t i = 0; i <= num_batches; i++) {
          unique_ids_offsets_ptr[i] = offsets_ptr[2 * i];
        }
        THRUST_CALL(
            gather, unique_ids_offsets_ptr,
            unique_ids_offsets_ptr + unique_ids_offsets.size(0),
            valid.data_ptr<int64_t>(), unique_ids_offsets_ptr);
        at::cuda::CUDAEvent unique_ids_offsets_event;
        unique_ids_offsets_event.record();
        auto unique_ids =
            torch::empty(offsets_ptr[2 * num_batches], src_ids[0].options());
        CUDA_KERNEL_CALL(
            _GetInsertedBatched, grid, block, 0, offsets_ptr[2 * num_batches],
            indexes.data_ptr<int32_t>(), pointers_dev_ptr, offsets_dev_ptr,
            map.ref(cuco::find), valid.data_ptr<int64_t>(),
            unique_ids.data_ptr<index_t>());
        auto mapped_ids =
            torch::empty(offsets_ptr[3 * num_batches], unique_ids.options());
        CUDA_KERNEL_CALL(
            _MapIdsBatched, grid, block, 0, num_batches,
            offsets_ptr[3 * num_batches], indexes.data_ptr<int32_t>(),
            pointers_dev_ptr, offsets_dev_ptr, map.ref(cuco::find),
            mapped_ids.data_ptr<index_t>());
        std::vector<std::tuple<torch::Tensor, torch::Tensor, torch::Tensor> >
            results;
        unique_ids_offsets_event.synchronize();
        for (int64_t i = 0; i < num_batches; i++) {
          results.emplace_back(
              unique_ids.slice(
                  0, unique_ids_offsets_ptr[i], unique_ids_offsets_ptr[i + 1]),
              mapped_ids.slice(
                  0, offsets_ptr[2 * i + 1], offsets_ptr[2 * i + 2]),
              mapped_ids.slice(
                  0, offsets_ptr[2 * num_batches + i],
                  offsets_ptr[2 * num_batches + i + 1]));
        }
        return results;
      }));
}

}  // namespace ops
}  // namespace graphbolt
