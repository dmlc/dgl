#include "hip/hip_runtime.h"
/**
 *  Copyright (c) 2021 by Contributors
 * @file graph/sampling/frequency_hashmap.cu
 * @brief frequency hashmap - used to select top-k frequency edges of each node
 */

#include <algorithm>
#include <hipcub/hipcub.hpp>  // NOLINT
#include <tuple>
#include <utility>

#include "../../../array/cuda/atomic.cuh"
#include "../../../runtime/cuda/cuda_common.h"
#include "frequency_hashmap.cuh"

namespace dgl {

namespace sampling {

namespace impl {

namespace {

int64_t _table_size(const int64_t num, const int64_t scale) {
  /**
   * Calculate the number of buckets in the hashtable. To guarantee we can
   * fill the hashtable in the worst case, we must use a number of buckets which
   * is a power of two.
   * https://en.wikipedia.org/wiki/Quadratic_probing#Limitations
   */
  const int64_t next_pow2 = 1 << static_cast<int64_t>(1 + std::log2(num >> 1));
  return next_pow2 << scale;
}

template <typename IdxType, int BLOCK_SIZE, int TILE_SIZE>
__global__ void _init_edge_table(void *edge_hashmap, int64_t edges_len) {
  using EdgeItem = typename DeviceEdgeHashmap<IdxType>::EdgeItem;
  auto edge_hashmap_t = static_cast<EdgeItem *>(edge_hashmap);
  int64_t start_idx = (blockIdx.x * TILE_SIZE) + threadIdx.x;
  int64_t last_idx = start_idx + TILE_SIZE;
#pragma unroll(4)
  for (int64_t idx = start_idx; idx < last_idx; idx += BLOCK_SIZE) {
    if (idx < edges_len) {
      EdgeItem *edge = (edge_hashmap_t + idx);
      edge->src = static_cast<IdxType>(-1);
      edge->cnt = static_cast<IdxType>(0);
    }
  }
}

template <typename IdxType, int BLOCK_SIZE, int TILE_SIZE>
__global__ void _count_frequency(
    const IdxType *src_data, const int64_t num_edges,
    const int64_t num_edges_per_node, IdxType *edge_blocks_prefix,
    bool *is_first_position, DeviceEdgeHashmap<IdxType> device_edge_hashmap) {
  int64_t start_idx = (blockIdx.x * TILE_SIZE) + threadIdx.x;
  int64_t last_idx = start_idx + TILE_SIZE;

  IdxType count = 0;
  for (int64_t idx = start_idx; idx < last_idx; idx += BLOCK_SIZE) {
    if (idx < num_edges) {
      IdxType src = src_data[idx];
      if (src == static_cast<IdxType>(-1)) {
        continue;
      }
      IdxType dst_idx = (idx / num_edges_per_node);
      if (device_edge_hashmap.InsertEdge(src, dst_idx) == 0) {
        is_first_position[idx] = true;
        ++count;
      }
    }
  }

  using BlockReduce = typename hipcub::BlockReduce<IdxType, BLOCK_SIZE>;
  __shared__ typename BlockReduce::TempStorage temp_space;

  count = BlockReduce(temp_space).Sum(count);
  if (threadIdx.x == 0) {
    edge_blocks_prefix[blockIdx.x] = count;
    if (blockIdx.x == 0) {
      edge_blocks_prefix[gridDim.x] = 0;
    }
  }
}

/**
 * This structure is used with cub's block-level prefixscan in order to
 * keep a running sum as items are iteratively processed.
 */
template <typename T>
struct BlockPrefixCallbackOp {
  T _running_total;

  __device__ BlockPrefixCallbackOp(const T running_total)
      : _running_total(running_total) {}

  __device__ T operator()(const T block_aggregate) {
    const T old_prefix = _running_total;
    _running_total += block_aggregate;
    return old_prefix;
  }
};

template <typename IdxType, typename Idx64Type, int BLOCK_SIZE, int TILE_SIZE>
__global__ void _compact_frequency(
    const IdxType *src_data, const IdxType *dst_data, const int64_t num_edges,
    const int64_t num_edges_per_node, const IdxType *edge_blocks_prefix,
    const bool *is_first_position, IdxType *num_unique_each_node,
    IdxType *unique_src_edges, Idx64Type *unique_frequency,
    DeviceEdgeHashmap<IdxType> device_edge_hashmap) {
  int64_t start_idx = (blockIdx.x * TILE_SIZE) + threadIdx.x;
  int64_t last_idx = start_idx + TILE_SIZE;
  const IdxType block_offset = edge_blocks_prefix[blockIdx.x];

  using BlockScan = typename hipcub::BlockScan<IdxType, BLOCK_SIZE>;
  __shared__ typename BlockScan::TempStorage temp_space;
  BlockPrefixCallbackOp<IdxType> prefix_op(0);

  for (int64_t idx = start_idx; idx < last_idx; idx += BLOCK_SIZE) {
    IdxType flag = 0;
    if (idx < num_edges) {
      IdxType src = src_data[idx];
      IdxType dst_idx = (idx / num_edges_per_node);
      if (idx % num_edges_per_node == 0) {
        num_unique_each_node[dst_idx] =
            device_edge_hashmap.GetDstCount(dst_idx);
      }
      if (is_first_position[idx] == true) {
        flag = 1;
      }
      BlockScan(temp_space).ExclusiveSum(flag, flag, prefix_op);
      __syncthreads();
      if (is_first_position[idx] == true) {
        const IdxType pos = (block_offset + flag);
        unique_src_edges[pos] = src;
        if (sizeof(IdxType) != sizeof(Idx64Type) &&
            sizeof(IdxType) == 4) {  // if IdxType is a 32-bit data
          unique_frequency[pos] =
              ((static_cast<Idx64Type>(num_edges / num_edges_per_node - dst_idx)
                << 32) |
               device_edge_hashmap.GetEdgeCount(src, dst_idx));
        } else {
          unique_frequency[pos] =
              device_edge_hashmap.GetEdgeCount(src, dst_idx);
        }
      }
    }
  }
}

template <typename IdxType, int BLOCK_SIZE, int TILE_SIZE>
__global__ void _get_pick_num(
    IdxType *num_unique_each_node, const int64_t num_pick,
    const int64_t num_dst_nodes) {
  int64_t start_idx = (blockIdx.x * TILE_SIZE) + threadIdx.x;
  int64_t last_idx = start_idx + TILE_SIZE;
#pragma unroll(4)
  for (int64_t idx = start_idx; idx < last_idx; idx += BLOCK_SIZE) {
    if (idx < num_dst_nodes) {
      IdxType &num_unique = num_unique_each_node[idx];
      num_unique = min(num_unique, static_cast<IdxType>(num_pick));
    }
  }
}

template <typename IdxType, typename Idx64Type, int BLOCK_SIZE, int TILE_SIZE>
__global__ void _pick_data(
    const Idx64Type *unique_frequency, const IdxType *unique_src_edges,
    const IdxType *unique_input_offsets, const IdxType *dst_data,
    const int64_t num_edges_per_node, const int64_t num_dst_nodes,
    const int64_t num_edges, const IdxType *unique_output_offsets,
    IdxType *output_src, IdxType *output_dst, IdxType *output_frequency) {
  int64_t start_idx = (blockIdx.x * TILE_SIZE) + threadIdx.x;
  int64_t last_idx = start_idx + TILE_SIZE;

  for (int64_t idx = start_idx; idx < last_idx; idx += BLOCK_SIZE) {
    if (idx < num_dst_nodes) {
      const int64_t dst_pos = (idx * num_edges_per_node);
      assert(dst_pos < num_edges);
      const IdxType dst = dst_data[dst_pos];
      const IdxType last_output_offset = unique_output_offsets[idx + 1];
      assert(
          (last_output_offset - unique_output_offsets[idx]) <=
          (unique_input_offsets[idx + 1] - unique_input_offsets[idx]));
      for (IdxType output_idx = unique_output_offsets[idx],
                   input_idx = unique_input_offsets[idx];
           output_idx < last_output_offset; ++output_idx, ++input_idx) {
        output_src[output_idx] = unique_src_edges[input_idx];
        output_dst[output_idx] = dst;
        output_frequency[output_idx] =
            static_cast<IdxType>(unique_frequency[input_idx]);
      }
    }
  }
}

}  // namespace

// return the old cnt of this edge
template <typename IdxType>
inline __device__ IdxType DeviceEdgeHashmap<IdxType>::InsertEdge(
    const IdxType &src, const IdxType &dst_idx) {
  IdxType start_off = dst_idx * _num_items_each_dst;
  IdxType pos = EdgeHash(src);
  IdxType delta = 1;
  IdxType old_cnt = static_cast<IdxType>(-1);
  while (true) {
    IdxType old_src = dgl::aten::cuda::AtomicCAS(
        &_edge_hashmap[start_off + pos].src, static_cast<IdxType>(-1), src);
    if (old_src == static_cast<IdxType>(-1) || old_src == src) {
      // first insert
      old_cnt = dgl::aten::cuda::AtomicAdd(
          &_edge_hashmap[start_off + pos].cnt, static_cast<IdxType>(1));
      if (old_src == static_cast<IdxType>(-1)) {
        assert(dst_idx < _num_dst);
        dgl::aten::cuda::AtomicAdd(
            &_dst_unique_edges[dst_idx], static_cast<IdxType>(1));
      }
      break;
    }
    pos = EdgeHash(pos + delta);
    delta += 1;
  }
  return old_cnt;
}

template <typename IdxType>
inline __device__ IdxType
DeviceEdgeHashmap<IdxType>::GetDstCount(const IdxType &dst_idx) {
  return _dst_unique_edges[dst_idx];
}

template <typename IdxType>
inline __device__ IdxType DeviceEdgeHashmap<IdxType>::GetEdgeCount(
    const IdxType &src, const IdxType &dst_idx) {
  IdxType start_off = dst_idx * _num_items_each_dst;
  IdxType pos = EdgeHash(src);
  IdxType delta = 1;
  while (_edge_hashmap[start_off + pos].src != src) {
    pos = EdgeHash(pos + delta);
    delta += 1;
  }
  return _edge_hashmap[start_off + pos].cnt;
}

template <typename IdxType>
FrequencyHashmap<IdxType>::FrequencyHashmap(
    int64_t num_dst, int64_t num_items_each_dst, DGLContext ctx,
    hipStream_t stream, int64_t edge_table_scale) {
  _ctx = ctx;
  _stream = stream;
  num_items_each_dst = _table_size(num_items_each_dst, edge_table_scale);
  auto device = dgl::runtime::DeviceAPI::Get(_ctx);
  auto dst_unique_edges = static_cast<IdxType *>(
      device->AllocWorkspace(_ctx, (num_dst) * sizeof(IdxType)));
  auto edge_hashmap = static_cast<EdgeItem *>(device->AllocWorkspace(
      _ctx, (num_dst * num_items_each_dst) * sizeof(EdgeItem)));
  constexpr int BLOCK_SIZE = 256;
  constexpr int TILE_SIZE = BLOCK_SIZE * 8;
  dim3 block(BLOCK_SIZE);
  dim3 grid((num_dst * num_items_each_dst + TILE_SIZE - 1) / TILE_SIZE);
  CUDA_CALL(hipMemset(dst_unique_edges, 0, (num_dst) * sizeof(IdxType)));
  CUDA_KERNEL_CALL(
      (_init_edge_table<IdxType, BLOCK_SIZE, TILE_SIZE>), grid, block, 0,
      _stream, edge_hashmap, (num_dst * num_items_each_dst));
  _device_edge_hashmap = new DeviceEdgeHashmap<IdxType>(
      num_dst, num_items_each_dst, dst_unique_edges, edge_hashmap);
  _dst_unique_edges = dst_unique_edges;
  _edge_hashmap = edge_hashmap;
}

template <typename IdxType>
FrequencyHashmap<IdxType>::~FrequencyHashmap() {
  auto device = dgl::runtime::DeviceAPI::Get(_ctx);
  delete _device_edge_hashmap;
  _device_edge_hashmap = nullptr;
  device->FreeWorkspace(_ctx, _dst_unique_edges);
  _dst_unique_edges = nullptr;
  device->FreeWorkspace(_ctx, _edge_hashmap);
  _edge_hashmap = nullptr;
}

template <typename IdxType>
std::tuple<IdArray, IdArray, IdArray> FrequencyHashmap<IdxType>::Topk(
    const IdxType *src_data, const IdxType *dst_data, DGLDataType dtype,
    const int64_t num_edges, const int64_t num_edges_per_node,
    const int64_t num_pick) {
  using Idx64Type = int64_t;
  const int64_t num_dst_nodes = (num_edges / num_edges_per_node);
  constexpr int BLOCK_SIZE = 256;
  // XXX: a experienced value, best performance in GV100
  constexpr int TILE_SIZE = BLOCK_SIZE * 32;
  const dim3 block(BLOCK_SIZE);
  const dim3 edges_grid((num_edges + TILE_SIZE - 1) / TILE_SIZE);
  auto device = dgl::runtime::DeviceAPI::Get(_ctx);
  const IdxType num_edge_blocks = static_cast<IdxType>(edges_grid.x);
  IdxType num_unique_edges = 0;

  // to mark if this position of edges is the first inserting position for
  // _edge_hashmap
  bool *is_first_position = static_cast<bool *>(
      device->AllocWorkspace(_ctx, sizeof(bool) * (num_edges)));
  CUDA_CALL(hipMemset(is_first_position, 0, sizeof(bool) * (num_edges)));
  // double space to use ExclusiveSum
  auto edge_blocks_prefix_data = static_cast<IdxType *>(device->AllocWorkspace(
      _ctx, 2 * sizeof(IdxType) * (num_edge_blocks + 1)));
  IdxType *edge_blocks_prefix = edge_blocks_prefix_data;
  IdxType *edge_blocks_prefix_alternate =
      (edge_blocks_prefix_data + (num_edge_blocks + 1));
  // triple space to use ExclusiveSum and unique_output_offsets
  auto num_unique_each_node_data = static_cast<IdxType *>(
      device->AllocWorkspace(_ctx, 3 * sizeof(IdxType) * (num_dst_nodes + 1)));
  IdxType *num_unique_each_node = num_unique_each_node_data;
  IdxType *num_unique_each_node_alternate =
      (num_unique_each_node_data + (num_dst_nodes + 1));
  IdxType *unique_output_offsets =
      (num_unique_each_node_data + 2 * (num_dst_nodes + 1));

  // 1. Scan the all edges and count the unique edges and unique edges for each
  // dst node
  CUDA_KERNEL_CALL(
      (_count_frequency<IdxType, BLOCK_SIZE, TILE_SIZE>), edges_grid, block, 0,
      _stream, src_data, num_edges, num_edges_per_node, edge_blocks_prefix,
      is_first_position, *_device_edge_hashmap);

  // 2. Compact the unique edges frequency
  // 2.1 ExclusiveSum the edge_blocks_prefix
  void *d_temp_storage = nullptr;
  size_t temp_storage_bytes = 0;
  CUDA_CALL(hipcub::DeviceScan::ExclusiveSum(
      d_temp_storage, temp_storage_bytes, edge_blocks_prefix,
      edge_blocks_prefix_alternate, num_edge_blocks + 1, _stream));
  d_temp_storage = device->AllocWorkspace(_ctx, temp_storage_bytes);
  CUDA_CALL(hipcub::DeviceScan::ExclusiveSum(
      d_temp_storage, temp_storage_bytes, edge_blocks_prefix,
      edge_blocks_prefix_alternate, num_edge_blocks + 1, _stream));
  device->FreeWorkspace(_ctx, d_temp_storage);
  std::swap(edge_blocks_prefix, edge_blocks_prefix_alternate);
  device->CopyDataFromTo(
      &edge_blocks_prefix[num_edge_blocks], 0, &num_unique_edges, 0,
      sizeof(num_unique_edges), _ctx, DGLContext{kDGLCPU, 0}, dtype);
  device->StreamSync(_ctx, _stream);
  // 2.2 Allocate the data of unique edges and frequency
  // double space to use SegmentedRadixSort
  auto unique_src_edges_data = static_cast<IdxType *>(
      device->AllocWorkspace(_ctx, 2 * sizeof(IdxType) * (num_unique_edges)));
  IdxType *unique_src_edges = unique_src_edges_data;
  IdxType *unique_src_edges_alternate =
      unique_src_edges_data + num_unique_edges;
  // double space to use SegmentedRadixSort
  auto unique_frequency_data = static_cast<Idx64Type *>(
      device->AllocWorkspace(_ctx, 2 * sizeof(Idx64Type) * (num_unique_edges)));
  Idx64Type *unique_frequency = unique_frequency_data;
  Idx64Type *unique_frequency_alternate =
      unique_frequency_data + num_unique_edges;
  // 2.3 Compact the unique edges and their frequency
  CUDA_KERNEL_CALL(
      (_compact_frequency<IdxType, Idx64Type, BLOCK_SIZE, TILE_SIZE>),
      edges_grid, block, 0, _stream, src_data, dst_data, num_edges,
      num_edges_per_node, edge_blocks_prefix, is_first_position,
      num_unique_each_node, unique_src_edges, unique_frequency,
      *_device_edge_hashmap);

  // 3. SegmentedRadixSort the unique edges and unique_frequency
  // 3.1 ExclusiveSum the num_unique_each_node
  d_temp_storage = nullptr;
  temp_storage_bytes = 0;
  CUDA_CALL(hipcub::DeviceScan::ExclusiveSum(
      d_temp_storage, temp_storage_bytes, num_unique_each_node,
      num_unique_each_node_alternate, num_dst_nodes + 1, _stream));
  d_temp_storage = device->AllocWorkspace(_ctx, temp_storage_bytes);
  CUDA_CALL(hipcub::DeviceScan::ExclusiveSum(
      d_temp_storage, temp_storage_bytes, num_unique_each_node,
      num_unique_each_node_alternate, num_dst_nodes + 1, _stream));
  device->FreeWorkspace(_ctx, d_temp_storage);
  // 3.2 SegmentedRadixSort the unique_src_edges and unique_frequency
  // Create a set of DoubleBuffers to wrap pairs of device pointers
  hipcub::DoubleBuffer<Idx64Type> d_unique_frequency(
      unique_frequency, unique_frequency_alternate);
  hipcub::DoubleBuffer<IdxType> d_unique_src_edges(
      unique_src_edges, unique_src_edges_alternate);
  // Determine temporary device storage requirements
  d_temp_storage = nullptr;
  temp_storage_bytes = 0;
  // the DeviceRadixSort is faster than DeviceSegmentedRadixSort,
  // especially when num_dst_nodes is large (about ~10000)
  if (dtype.bits == 32) {
    CUDA_CALL(hipcub::DeviceRadixSort::SortPairsDescending(
        d_temp_storage, temp_storage_bytes, d_unique_frequency,
        d_unique_src_edges, num_unique_edges, 0, sizeof(Idx64Type) * 8,
        _stream));
  } else {
    CUDA_CALL(hipcub::DeviceSegmentedRadixSort::SortPairsDescending(
        d_temp_storage, temp_storage_bytes, d_unique_frequency,
        d_unique_src_edges, num_unique_edges, num_dst_nodes,
        num_unique_each_node_alternate, num_unique_each_node_alternate + 1, 0,
        sizeof(Idx64Type) * 8, _stream));
  }
  d_temp_storage = device->AllocWorkspace(_ctx, temp_storage_bytes);
  if (dtype.bits == 32) {
    CUDA_CALL(hipcub::DeviceRadixSort::SortPairsDescending(
        d_temp_storage, temp_storage_bytes, d_unique_frequency,
        d_unique_src_edges, num_unique_edges, 0, sizeof(Idx64Type) * 8,
        _stream));
  } else {
    CUDA_CALL(hipcub::DeviceSegmentedRadixSort::SortPairsDescending(
        d_temp_storage, temp_storage_bytes, d_unique_frequency,
        d_unique_src_edges, num_unique_edges, num_dst_nodes,
        num_unique_each_node_alternate, num_unique_each_node_alternate + 1, 0,
        sizeof(Idx64Type) * 8, _stream));
  }
  device->FreeWorkspace(_ctx, d_temp_storage);

  // 4. Get the final pick number for each dst node
  // 4.1 Reset the min(num_pick, num_unique_each_node) to num_unique_each_node
  constexpr int NODE_TILE_SIZE = BLOCK_SIZE * 2;
  const dim3 nodes_grid((num_dst_nodes + NODE_TILE_SIZE - 1) / NODE_TILE_SIZE);
  CUDA_KERNEL_CALL(
      (_get_pick_num<IdxType, BLOCK_SIZE, NODE_TILE_SIZE>), nodes_grid, block,
      0, _stream, num_unique_each_node, num_pick, num_dst_nodes);
  // 4.2 ExclusiveSum the new num_unique_each_node as unique_output_offsets
  // use unique_output_offsets;
  d_temp_storage = nullptr;
  temp_storage_bytes = 0;
  CUDA_CALL(hipcub::DeviceScan::ExclusiveSum(
      d_temp_storage, temp_storage_bytes, num_unique_each_node,
      unique_output_offsets, num_dst_nodes + 1, _stream));
  d_temp_storage = device->AllocWorkspace(_ctx, temp_storage_bytes);
  CUDA_CALL(hipcub::DeviceScan::ExclusiveSum(
      d_temp_storage, temp_storage_bytes, num_unique_each_node,
      unique_output_offsets, num_dst_nodes + 1, _stream));
  device->FreeWorkspace(_ctx, d_temp_storage);

  // 5. Pick the data to result
  IdxType num_output = 0;
  device->CopyDataFromTo(
      &unique_output_offsets[num_dst_nodes], 0, &num_output, 0,
      sizeof(num_output), _ctx, DGLContext{kDGLCPU, 0}, dtype);
  device->StreamSync(_ctx, _stream);

  IdArray res_src =
      IdArray::Empty({static_cast<int64_t>(num_output)}, dtype, _ctx);
  IdArray res_dst =
      IdArray::Empty({static_cast<int64_t>(num_output)}, dtype, _ctx);
  IdArray res_cnt =
      IdArray::Empty({static_cast<int64_t>(num_output)}, dtype, _ctx);
  CUDA_KERNEL_CALL(
      (_pick_data<IdxType, Idx64Type, BLOCK_SIZE, NODE_TILE_SIZE>), nodes_grid,
      block, 0, _stream, d_unique_frequency.Current(),
      d_unique_src_edges.Current(), num_unique_each_node_alternate, dst_data,
      num_edges_per_node, num_dst_nodes, num_edges, unique_output_offsets,
      res_src.Ptr<IdxType>(), res_dst.Ptr<IdxType>(), res_cnt.Ptr<IdxType>());

  device->FreeWorkspace(_ctx, is_first_position);
  device->FreeWorkspace(_ctx, edge_blocks_prefix_data);
  device->FreeWorkspace(_ctx, num_unique_each_node_data);
  device->FreeWorkspace(_ctx, unique_src_edges_data);
  device->FreeWorkspace(_ctx, unique_frequency_data);

  return std::make_tuple(res_src, res_dst, res_cnt);
}

template class FrequencyHashmap<int64_t>;

template class FrequencyHashmap<int32_t>;

};  // namespace impl

};  // namespace sampling

};  // namespace dgl
