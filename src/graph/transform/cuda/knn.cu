#include "hip/hip_runtime.h"
/**
 *  Copyright (c) 2020 by Contributors
 * @file graph/transform/cuda/knn.cu
 * @brief k-nearest-neighbor (KNN) implementation (cuda)
 */

#include <hiprand/hiprand_kernel.h>
#include <dgl/array.h>
#include <dgl/random.h>
#include <dgl/runtime/device_api.h>

#include <algorithm>
#include <hipcub/hipcub.hpp>  // NOLINT
#include <limits>
#include <string>
#include <type_traits>
#include <vector>

#include "../../../array/cuda/utils.h"
#include "../../../runtime/cuda/cuda_common.h"
#include "../knn.h"

namespace dgl {
namespace transform {
namespace impl {

/**
 * @brief Given input `size`, find the smallest value
 * greater or equal to `size` that is a multiple of `align`.
 *
 * e.g. Pow2Align(17, 4) = 20, Pow2Align(17, 8) = 24
 */
template <typename Type>
static __host__ __device__ std::enable_if_t<std::is_unsigned<Type>::value, Type>
Pow2Align(Type size, Type align) {
  if (align <= 1 || size <= 0) return size;
  return ((size - 1) | (align - 1)) + 1;
}

/**
 * @brief Utility class used to avoid linker errors with extern
 *  unsized shared memory arrays with templated type
 */
template <typename Type>
struct SharedMemory {
  __device__ inline operator Type*() {
    extern __shared__ int __smem[];
    return reinterpret_cast<Type*>(__smem);
  }

  __device__ inline operator const Type*() const {
    extern __shared__ int __smem[];
    return reinterpret_cast<Type*>(__smem);
  }
};

// specialize for double to avoid unaligned memory
// access compile errors
template <>
struct SharedMemory<double> {
  __device__ inline operator double*() {
    extern __shared__ double __smem_d[];
    return reinterpret_cast<double*>(__smem_d);
  }

  __device__ inline operator const double*() const {
    extern __shared__ double __smem_d[];
    return reinterpret_cast<double*>(__smem_d);
  }
};

/** @brief Compute Euclidean distance between two vectors in a cuda kernel */
template <typename FloatType, typename IdType>
__device__ FloatType
EuclideanDist(const FloatType* vec1, const FloatType* vec2, const int64_t dim) {
  FloatType dist = 0;
  IdType idx = 0;
  for (; idx < dim - 3; idx += 4) {
    FloatType diff0 = vec1[idx] - vec2[idx];
    FloatType diff1 = vec1[idx + 1] - vec2[idx + 1];
    FloatType diff2 = vec1[idx + 2] - vec2[idx + 2];
    FloatType diff3 = vec1[idx + 3] - vec2[idx + 3];

    dist += diff0 * diff0 + diff1 * diff1 + diff2 * diff2 + diff3 * diff3;
  }

  for (; idx < dim; ++idx) {
    FloatType diff = vec1[idx] - vec2[idx];
    dist += diff * diff;
  }

  return dist;
}

/**
 * @brief Compute Euclidean distance between two vectors in a cuda kernel,
 *  return positive infinite value if the intermediate distance is greater
 *  than the worst distance.
 */
template <typename FloatType, typename IdType>
__device__ FloatType EuclideanDistWithCheck(
    const FloatType* vec1, const FloatType* vec2, const int64_t dim,
    const FloatType worst_dist) {
  FloatType dist = 0;
  IdType idx = 0;
  bool early_stop = false;

  for (; idx < dim - 3; idx += 4) {
    FloatType diff0 = vec1[idx] - vec2[idx];
    FloatType diff1 = vec1[idx + 1] - vec2[idx + 1];
    FloatType diff2 = vec1[idx + 2] - vec2[idx + 2];
    FloatType diff3 = vec1[idx + 3] - vec2[idx + 3];

    dist += diff0 * diff0 + diff1 * diff1 + diff2 * diff2 + diff3 * diff3;
    if (dist > worst_dist) {
      early_stop = true;
      idx = dim;
      break;
    }
  }

  for (; idx < dim; ++idx) {
    FloatType diff = vec1[idx] - vec2[idx];
    dist += diff * diff;
    if (dist > worst_dist) {
      early_stop = true;
      break;
    }
  }

  if (early_stop) {
    return std::numeric_limits<FloatType>::max();
  } else {
    return dist;
  }
}

template <typename FloatType, typename IdType>
__device__ void BuildHeap(IdType* indices, FloatType* dists, int size) {
  for (int i = size / 2 - 1; i >= 0; --i) {
    IdType idx = i;
    while (true) {
      IdType largest = idx;
      IdType left = idx * 2 + 1;
      IdType right = left + 1;
      if (left < size && dists[left] > dists[largest]) {
        largest = left;
      }
      if (right < size && dists[right] > dists[largest]) {
        largest = right;
      }
      if (largest != idx) {
        IdType tmp_idx = indices[largest];
        indices[largest] = indices[idx];
        indices[idx] = tmp_idx;

        FloatType tmp_dist = dists[largest];
        dists[largest] = dists[idx];
        dists[idx] = tmp_dist;
        idx = largest;
      } else {
        break;
      }
    }
  }
}

template <typename FloatType, typename IdType>
__device__ void HeapInsert(
    IdType* indices, FloatType* dist, IdType new_idx, FloatType new_dist,
    int size, bool check_repeat = false) {
  if (new_dist > dist[0]) return;

  // check if we have it
  if (check_repeat) {
    for (IdType i = 0; i < size; ++i) {
      if (indices[i] == new_idx) return;
    }
  }

  IdType left = 0, right = 0, idx = 0, largest = 0;
  dist[0] = new_dist;
  indices[0] = new_idx;
  while (true) {
    left = idx * 2 + 1;
    right = left + 1;
    if (left < size && dist[left] > dist[largest]) {
      largest = left;
    }
    if (right < size && dist[right] > dist[largest]) {
      largest = right;
    }
    if (largest != idx) {
      IdType tmp_idx = indices[idx];
      indices[idx] = indices[largest];
      indices[largest] = tmp_idx;

      FloatType tmp_dist = dist[idx];
      dist[idx] = dist[largest];
      dist[largest] = tmp_dist;

      idx = largest;
    } else {
      break;
    }
  }
}

template <typename FloatType, typename IdType>
__device__ bool FlaggedHeapInsert(
    IdType* indices, FloatType* dist, bool* flags, IdType new_idx,
    FloatType new_dist, bool new_flag, int size, bool check_repeat = false) {
  if (new_dist > dist[0]) return false;

  // check if we have it
  if (check_repeat) {
    for (IdType i = 0; i < size; ++i) {
      if (indices[i] == new_idx) return false;
    }
  }

  IdType left = 0, right = 0, idx = 0, largest = 0;
  dist[0] = new_dist;
  indices[0] = new_idx;
  flags[0] = new_flag;
  while (true) {
    left = idx * 2 + 1;
    right = left + 1;
    if (left < size && dist[left] > dist[largest]) {
      largest = left;
    }
    if (right < size && dist[right] > dist[largest]) {
      largest = right;
    }
    if (largest != idx) {
      IdType tmp_idx = indices[idx];
      indices[idx] = indices[largest];
      indices[largest] = tmp_idx;

      FloatType tmp_dist = dist[idx];
      dist[idx] = dist[largest];
      dist[largest] = tmp_dist;

      bool tmp_flag = flags[idx];
      flags[idx] = flags[largest];
      flags[largest] = tmp_flag;

      idx = largest;
    } else {
      break;
    }
  }
  return true;
}

/**
 * @brief Brute force kNN kernel. Compute distance for each pair of input points
 * and get the result directly (without a distance matrix).
 */
template <typename FloatType, typename IdType>
__global__ void BruteforceKnnKernel(
    const FloatType* data_points, const IdType* data_offsets,
    const FloatType* query_points, const IdType* query_offsets, const int k,
    FloatType* dists, IdType* query_out, IdType* data_out,
    const int64_t num_batches, const int64_t feature_size) {
  const IdType q_idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (q_idx >= query_offsets[num_batches]) return;
  IdType batch_idx = 0;
  for (IdType b = 0; b < num_batches + 1; ++b) {
    if (query_offsets[b] > q_idx) {
      batch_idx = b - 1;
      break;
    }
  }
  const IdType data_start = data_offsets[batch_idx],
               data_end = data_offsets[batch_idx + 1];

  for (IdType k_idx = 0; k_idx < k; ++k_idx) {
    query_out[q_idx * k + k_idx] = q_idx;
    dists[q_idx * k + k_idx] = std::numeric_limits<FloatType>::max();
  }
  FloatType worst_dist = std::numeric_limits<FloatType>::max();

  for (IdType d_idx = data_start; d_idx < data_end; ++d_idx) {
    FloatType tmp_dist = EuclideanDistWithCheck<FloatType, IdType>(
        query_points + q_idx * feature_size, data_points + d_idx * feature_size,
        feature_size, worst_dist);

    IdType out_offset = q_idx * k;
    HeapInsert<FloatType, IdType>(
        data_out + out_offset, dists + out_offset, d_idx, tmp_dist, k);
    worst_dist = dists[q_idx * k];
  }
}

/**
 * @brief Same as BruteforceKnnKernel, but use shared memory as buffer.
 *  This kernel divides query points and data points into blocks. For each
 *  query block, it will make a loop over all data blocks and compute distances.
 *  This kernel is faster when the dimension of input points is not large.
 */
template <typename FloatType, typename IdType>
__global__ void BruteforceKnnShareKernel(
    const FloatType* data_points, const IdType* data_offsets,
    const FloatType* query_points, const IdType* query_offsets,
    const IdType* block_batch_id, const IdType* local_block_id, const int k,
    FloatType* dists, IdType* query_out, IdType* data_out,
    const int64_t num_batches, const int64_t feature_size) {
  const IdType block_idx = static_cast<IdType>(blockIdx.x);
  const IdType block_size = static_cast<IdType>(blockDim.x);
  const IdType batch_idx = block_batch_id[block_idx];
  const IdType local_bid = local_block_id[block_idx];
  const IdType query_start = query_offsets[batch_idx] + block_size * local_bid;
  const IdType query_end =
      min(query_start + block_size, query_offsets[batch_idx + 1]);
  if (query_start >= query_end) return;
  const IdType query_idx = query_start + threadIdx.x;
  const IdType data_start = data_offsets[batch_idx];
  const IdType data_end = data_offsets[batch_idx + 1];

  // shared memory: points in block + distance buffer + result buffer
  FloatType* data_buff = SharedMemory<FloatType>();
  FloatType* query_buff = data_buff + block_size * feature_size;
  FloatType* dist_buff = query_buff + block_size * feature_size;
  IdType* res_buff = reinterpret_cast<IdType*>(Pow2Align<uint64_t>(
      reinterpret_cast<uint64_t>(dist_buff + block_size * k), sizeof(IdType)));
  FloatType worst_dist = std::numeric_limits<FloatType>::max();

  // initialize dist buff with inf value
  for (auto i = 0; i < k; ++i) {
    dist_buff[threadIdx.x + i * block_size] =
        std::numeric_limits<FloatType>::max();
  }

  // load query data to shared memory
  // TODO(tianqi): could be better here to exploit coalesce global memory
  // access.
  if (query_idx < query_end) {
    for (auto i = 0; i < feature_size; ++i) {
      // to avoid bank conflict, we use transpose here
      query_buff[threadIdx.x + i * block_size] =
          query_points[query_idx * feature_size + i];
    }
  }

  // perform computation on each tile
  for (auto tile_start = data_start; tile_start < data_end;
       tile_start += block_size) {
    // each thread load one data point into the shared memory
    IdType load_idx = tile_start + threadIdx.x;
    if (load_idx < data_end) {
      for (auto i = 0; i < feature_size; ++i) {
        data_buff[threadIdx.x * feature_size + i] =
            data_points[load_idx * feature_size + i];
      }
    }
    __syncthreads();

    // compute distance for one tile
    IdType true_block_size = min(data_end - tile_start, block_size);
    if (query_idx < query_end) {
      for (IdType d_idx = 0; d_idx < true_block_size; ++d_idx) {
        FloatType tmp_dist = 0;
        bool early_stop = false;
        IdType dim_idx = 0;

        for (; dim_idx < feature_size - 3; dim_idx += 4) {
          FloatType diff0 = query_buff[threadIdx.x + block_size * (dim_idx)] -
                            data_buff[d_idx * feature_size + dim_idx];
          FloatType diff1 =
              query_buff[threadIdx.x + block_size * (dim_idx + 1)] -
              data_buff[d_idx * feature_size + dim_idx + 1];
          FloatType diff2 =
              query_buff[threadIdx.x + block_size * (dim_idx + 2)] -
              data_buff[d_idx * feature_size + dim_idx + 2];
          FloatType diff3 =
              query_buff[threadIdx.x + block_size * (dim_idx + 3)] -
              data_buff[d_idx * feature_size + dim_idx + 3];

          tmp_dist +=
              diff0 * diff0 + diff1 * diff1 + diff2 * diff2 + diff3 * diff3;

          if (tmp_dist > worst_dist) {
            early_stop = true;
            dim_idx = feature_size;
            break;
          }
        }

        for (; dim_idx < feature_size; ++dim_idx) {
          const FloatType diff =
              query_buff[threadIdx.x + dim_idx * block_size] -
              data_buff[d_idx * feature_size + dim_idx];
          tmp_dist += diff * diff;

          if (tmp_dist > worst_dist) {
            early_stop = true;
            break;
          }
        }

        if (early_stop) continue;

        HeapInsert<FloatType, IdType>(
            res_buff + threadIdx.x * k, dist_buff + threadIdx.x * k,
            d_idx + tile_start, tmp_dist, k);
        worst_dist = dist_buff[threadIdx.x * k];
      }
    }
    __syncthreads();
  }

  // copy result to global memory
  if (query_idx < query_end) {
    for (auto i = 0; i < k; ++i) {
      dists[query_idx * k + i] = dist_buff[threadIdx.x * k + i];
      data_out[query_idx * k + i] = res_buff[threadIdx.x * k + i];
      query_out[query_idx * k + i] = query_idx;
    }
  }
}

/** @brief determine the number of blocks for each segment */
template <typename IdType>
__global__ void GetNumBlockPerSegment(
    const IdType* offsets, IdType* out, const int64_t batch_size,
    const int64_t block_size) {
  const IdType idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < batch_size) {
    out[idx] = (offsets[idx + 1] - offsets[idx] - 1) / block_size + 1;
  }
}

/** @brief Get the batch index and local index in segment for each block */
template <typename IdType>
__global__ void GetBlockInfo(
    const IdType* num_block_prefixsum, IdType* block_batch_id,
    IdType* local_block_id, size_t batch_size, size_t num_blocks) {
  const IdType idx = blockIdx.x * blockDim.x + threadIdx.x;
  IdType i = 0;

  if (idx < num_blocks) {
    for (; i < batch_size; ++i) {
      if (num_block_prefixsum[i] > idx) break;
    }
    i--;
    block_batch_id[idx] = i;
    local_block_id[idx] = idx - num_block_prefixsum[i];
  }
}

/**
 * @brief Brute force kNN. Compute distance for each pair of input points and
 * get the result directly (without a distance matrix).
 *
 * @tparam FloatType The type of input points.
 * @tparam IdType The type of id.
 * @param data_points NDArray of dataset points.
 * @param data_offsets offsets of point index in data points.
 * @param query_points NDArray of query points
 * @param query_offsets offsets of point index in query points.
 * @param k the number of nearest points
 * @param result output array
 */
template <typename FloatType, typename IdType>
void BruteForceKNNCuda(
    const NDArray& data_points, const IdArray& data_offsets,
    const NDArray& query_points, const IdArray& query_offsets, const int k,
    IdArray result) {
  hipStream_t stream = runtime::getCurrentCUDAStream();
  const auto& ctx = data_points->ctx;
  auto device = runtime::DeviceAPI::Get(ctx);
  const int64_t batch_size = data_offsets->shape[0] - 1;
  const int64_t feature_size = data_points->shape[1];
  const IdType* data_offsets_data = data_offsets.Ptr<IdType>();
  const IdType* query_offsets_data = query_offsets.Ptr<IdType>();
  const FloatType* data_points_data = data_points.Ptr<FloatType>();
  const FloatType* query_points_data = query_points.Ptr<FloatType>();
  IdType* query_out = result.Ptr<IdType>();
  IdType* data_out = query_out + k * query_points->shape[0];

  FloatType* dists = static_cast<FloatType*>(device->AllocWorkspace(
      ctx, k * query_points->shape[0] * sizeof(FloatType)));

  const int64_t block_size = cuda::FindNumThreads(query_points->shape[0]);
  const int64_t num_blocks = (query_points->shape[0] - 1) / block_size + 1;
  CUDA_KERNEL_CALL(
      BruteforceKnnKernel, num_blocks, block_size, 0, stream, data_points_data,
      data_offsets_data, query_points_data, query_offsets_data, k, dists,
      query_out, data_out, batch_size, feature_size);

  device->FreeWorkspace(ctx, dists);
}

/**
 * @brief Brute force kNN with shared memory.
 *  This function divides query points and data points into blocks. For each
 *  query block, it will make a loop over all data blocks and compute distances.
 *  It will be faster when the dimension of input points is not large.
 *
 * @tparam FloatType The type of input points.
 * @tparam IdType The type of id.
 * @param data_points NDArray of dataset points.
 * @param data_offsets offsets of point index in data points.
 * @param query_points NDArray of query points
 * @param query_offsets offsets of point index in query points.
 * @param k the number of nearest points
 * @param result output array
 */
template <typename FloatType, typename IdType>
void BruteForceKNNSharedCuda(
    const NDArray& data_points, const IdArray& data_offsets,
    const NDArray& query_points, const IdArray& query_offsets, const int k,
    IdArray result) {
  hipStream_t stream = runtime::getCurrentCUDAStream();
  const auto& ctx = data_points->ctx;
  auto device = runtime::DeviceAPI::Get(ctx);
  const int64_t batch_size = data_offsets->shape[0] - 1;
  const int64_t feature_size = data_points->shape[1];
  const IdType* data_offsets_data = data_offsets.Ptr<IdType>();
  const IdType* query_offsets_data = query_offsets.Ptr<IdType>();
  const FloatType* data_points_data = data_points.Ptr<FloatType>();
  const FloatType* query_points_data = query_points.Ptr<FloatType>();
  IdType* query_out = result.Ptr<IdType>();
  IdType* data_out = query_out + k * query_points->shape[0];
  constexpr size_t smem_align = std::max(sizeof(IdType), sizeof(FloatType));

  // get max shared memory per block in bytes
  // determine block size according to this value
  int max_sharedmem_per_block = 0;
  CUDA_CALL(hipDeviceGetAttribute(
      &max_sharedmem_per_block, hipDeviceAttributeMaxSharedMemoryPerBlock,
      ctx.device_id));
  const int64_t single_shared_mem = static_cast<int64_t>(Pow2Align<size_t>(
      (k + 2 * feature_size) * sizeof(FloatType) + k * sizeof(IdType),
      smem_align));

  const int64_t block_size =
      cuda::FindNumThreads(max_sharedmem_per_block / single_shared_mem);

  // Determine the number of blocks. We first get the number of blocks for each
  // segment. Then we get the block id offset via prefix sum.
  IdType* num_block_per_segment = static_cast<IdType*>(
      device->AllocWorkspace(ctx, batch_size * sizeof(IdType)));
  IdType* num_block_prefixsum = static_cast<IdType*>(
      device->AllocWorkspace(ctx, batch_size * sizeof(IdType)));

  // block size for GetNumBlockPerSegment computation
  int64_t temp_block_size = cuda::FindNumThreads(batch_size);
  int64_t temp_num_blocks = (batch_size - 1) / temp_block_size + 1;
  CUDA_KERNEL_CALL(
      GetNumBlockPerSegment, temp_num_blocks, temp_block_size, 0, stream,
      query_offsets_data, num_block_per_segment, batch_size, block_size);
  size_t prefix_temp_size = 0;
  CUDA_CALL(hipcub::DeviceScan::ExclusiveSum(
      nullptr, prefix_temp_size, num_block_per_segment, num_block_prefixsum,
      batch_size, stream));
  void* prefix_temp = device->AllocWorkspace(ctx, prefix_temp_size);
  CUDA_CALL(hipcub::DeviceScan::ExclusiveSum(
      prefix_temp, prefix_temp_size, num_block_per_segment, num_block_prefixsum,
      batch_size, stream));
  device->FreeWorkspace(ctx, prefix_temp);

  // wait for results
  CUDA_CALL(hipStreamSynchronize(stream));

  int64_t num_blocks = 0, final_elem = 0,
          copyoffset = (batch_size - 1) * sizeof(IdType);
  device->CopyDataFromTo(
      num_block_prefixsum, copyoffset, &num_blocks, 0, sizeof(IdType), ctx,
      DGLContext{kDGLCPU, 0}, query_offsets->dtype);
  device->CopyDataFromTo(
      num_block_per_segment, copyoffset, &final_elem, 0, sizeof(IdType), ctx,
      DGLContext{kDGLCPU, 0}, query_offsets->dtype);
  num_blocks += final_elem;
  device->FreeWorkspace(ctx, num_block_per_segment);

  // get batch id and local id in segment
  temp_block_size = cuda::FindNumThreads(num_blocks);
  temp_num_blocks = (num_blocks - 1) / temp_block_size + 1;
  IdType* block_batch_id = static_cast<IdType*>(
      device->AllocWorkspace(ctx, num_blocks * sizeof(IdType)));
  IdType* local_block_id = static_cast<IdType*>(
      device->AllocWorkspace(ctx, num_blocks * sizeof(IdType)));
  CUDA_KERNEL_CALL(
      GetBlockInfo, temp_num_blocks, temp_block_size, 0, stream,
      num_block_prefixsum, block_batch_id, local_block_id, batch_size,
      num_blocks);

  FloatType* dists = static_cast<FloatType*>(device->AllocWorkspace(
      ctx, k * query_points->shape[0] * sizeof(FloatType)));
  CUDA_KERNEL_CALL(
      BruteforceKnnShareKernel, num_blocks, block_size,
      single_shared_mem * block_size, stream, data_points_data,
      data_offsets_data, query_points_data, query_offsets_data, block_batch_id,
      local_block_id, k, dists, query_out, data_out, batch_size, feature_size);

  device->FreeWorkspace(ctx, num_block_prefixsum);
  device->FreeWorkspace(ctx, dists);
  device->FreeWorkspace(ctx, local_block_id);
  device->FreeWorkspace(ctx, block_batch_id);
}

/** @brief Setup rng state for nn-descent */
__global__ void SetupRngKernel(
    hiprandState* states, const uint64_t seed, const size_t n) {
  size_t id = blockIdx.x * blockDim.x + threadIdx.x;
  if (id < n) {
    hiprand_init(seed, id, 0, states + id);
  }
}

/**
 * @brief Randomly initialize neighbors (sampling without replacement)
 * for each nodes
 */
template <typename FloatType, typename IdType>
__global__ void RandomInitNeighborsKernel(
    const FloatType* points, const IdType* offsets, IdType* central_nodes,
    IdType* neighbors, FloatType* dists, bool* flags, const int k,
    const int64_t feature_size, const int64_t batch_size, const uint64_t seed) {
  const IdType point_idx = blockIdx.x * blockDim.x + threadIdx.x;
  IdType batch_idx = 0;
  if (point_idx >= offsets[batch_size]) return;
  hiprandState state;
  hiprand_init(seed, point_idx, 0, &state);

  // find the segment location in the input batch
  for (IdType b = 0; b < batch_size + 1; ++b) {
    if (offsets[b] > point_idx) {
      batch_idx = b - 1;
      break;
    }
  }

  const IdType segment_size = offsets[batch_idx + 1] - offsets[batch_idx];
  IdType* current_neighbors = neighbors + point_idx * k;
  IdType* current_central_nodes = central_nodes + point_idx * k;
  bool* current_flags = flags + point_idx * k;
  FloatType* current_dists = dists + point_idx * k;
  IdType segment_start = offsets[batch_idx];

  // reservoir sampling
  for (IdType i = 0; i < k; ++i) {
    current_neighbors[i] = i + segment_start;
    current_central_nodes[i] = point_idx;
  }
  for (IdType i = k; i < segment_size; ++i) {
    const IdType j = static_cast<IdType>(hiprand(&state) % (i + 1));
    if (j < k) current_neighbors[j] = i + segment_start;
  }

  // compute distances and set flags
  for (IdType i = 0; i < k; ++i) {
    current_flags[i] = true;
    current_dists[i] = EuclideanDist<FloatType, IdType>(
        points + point_idx * feature_size,
        points + current_neighbors[i] * feature_size, feature_size);
  }

  // build heap
  BuildHeap<FloatType, IdType>(neighbors + point_idx * k, current_dists, k);
}

/**
 * @brief Randomly select candidates from current knn and reverse-knn graph for
 *        nn-descent.
 */
template <typename IdType>
__global__ void FindCandidatesKernel(
    const IdType* offsets, IdType* new_candidates, IdType* old_candidates,
    IdType* neighbors, bool* flags, const uint64_t seed,
    const int64_t batch_size, const int num_candidates, const int k) {
  const IdType point_idx = blockIdx.x * blockDim.x + threadIdx.x;
  IdType batch_idx = 0;
  if (point_idx >= offsets[batch_size]) return;
  hiprandState state;
  hiprand_init(seed, point_idx, 0, &state);

  // find the segment location in the input batch
  for (IdType b = 0; b < batch_size + 1; ++b) {
    if (offsets[b] > point_idx) {
      batch_idx = b - 1;
      break;
    }
  }

  IdType segment_start = offsets[batch_idx],
         segment_end = offsets[batch_idx + 1];
  IdType* current_neighbors = neighbors + point_idx * k;
  bool* current_flags = flags + point_idx * k;

  // reset candidates
  IdType* new_candidates_ptr =
      new_candidates + point_idx * (num_candidates + 1);
  IdType* old_candidates_ptr =
      old_candidates + point_idx * (num_candidates + 1);
  new_candidates_ptr[0] = 0;
  old_candidates_ptr[0] = 0;

  // select candidates from current knn graph
  // here we use candidate[0] for reservoir sampling temporarily
  for (IdType i = 0; i < k; ++i) {
    IdType candidate = current_neighbors[i];
    IdType* candidate_array =
        current_flags[i] ? new_candidates_ptr : old_candidates_ptr;
    IdType curr_num = candidate_array[0];
    IdType* candidate_data = candidate_array + 1;

    // reservoir sampling
    if (curr_num < num_candidates) {
      candidate_data[curr_num] = candidate;
    } else {
      IdType pos = static_cast<IdType>(hiprand(&state) % (curr_num + 1));
      if (pos < num_candidates) candidate_data[pos] = candidate;
    }
    ++candidate_array[0];
  }

  // select candidates from current reverse knn graph
  // here we use candidate[0] for reservoir sampling temporarily
  IdType index_start = segment_start * k, index_end = segment_end * k;
  for (IdType i = index_start; i < index_end; ++i) {
    if (neighbors[i] == point_idx) {
      IdType reverse_candidate = (i - index_start) / k + segment_start;
      IdType* candidate_array =
          flags[i] ? new_candidates_ptr : old_candidates_ptr;
      IdType curr_num = candidate_array[0];
      IdType* candidate_data = candidate_array + 1;

      // reservoir sampling
      if (curr_num < num_candidates) {
        candidate_data[curr_num] = reverse_candidate;
      } else {
        IdType pos = static_cast<IdType>(hiprand(&state) % (curr_num + 1));
        if (pos < num_candidates) candidate_data[pos] = reverse_candidate;
      }
      ++candidate_array[0];
    }
  }

  // set candidate[0] back to length
  if (new_candidates_ptr[0] > num_candidates)
    new_candidates_ptr[0] = num_candidates;
  if (old_candidates_ptr[0] > num_candidates)
    old_candidates_ptr[0] = num_candidates;

  // mark new_candidates as old
  IdType num_new_candidates = new_candidates_ptr[0];
  for (IdType i = 0; i < k; ++i) {
    IdType neighbor_idx = current_neighbors[i];

    if (current_flags[i]) {
      for (IdType j = 1; j < num_new_candidates + 1; ++j) {
        if (new_candidates_ptr[j] == neighbor_idx) {
          current_flags[i] = false;
          break;
        }
      }
    }
  }
}

/** @brief Update knn graph according to selected candidates for nn-descent */
template <typename FloatType, typename IdType>
__global__ void UpdateNeighborsKernel(
    const FloatType* points, const IdType* offsets, IdType* neighbors,
    IdType* new_candidates, IdType* old_candidates, FloatType* distances,
    bool* flags, IdType* num_updates, const int64_t batch_size,
    const int num_candidates, const int k, const int64_t feature_size) {
  const IdType point_idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (point_idx >= offsets[batch_size]) return;
  IdType* current_neighbors = neighbors + point_idx * k;
  bool* current_flags = flags + point_idx * k;
  FloatType* current_dists = distances + point_idx * k;
  IdType* new_candidates_ptr =
      new_candidates + point_idx * (num_candidates + 1);
  IdType* old_candidates_ptr =
      old_candidates + point_idx * (num_candidates + 1);
  IdType num_new_candidates = new_candidates_ptr[0];
  IdType num_old_candidates = old_candidates_ptr[0];
  IdType current_num_updates = 0;

  // process new candidates
  for (IdType i = 1; i <= num_new_candidates; ++i) {
    IdType new_c = new_candidates_ptr[i];

    // new/old candidates of the current new candidate
    IdType* twohop_new_ptr = new_candidates + new_c * (num_candidates + 1);
    IdType* twohop_old_ptr = old_candidates + new_c * (num_candidates + 1);
    IdType num_twohop_new = twohop_new_ptr[0];
    IdType num_twohop_old = twohop_old_ptr[0];
    FloatType worst_dist = current_dists[0];

    // new - new
    for (IdType j = 1; j <= num_twohop_new; ++j) {
      IdType twohop_new_c = twohop_new_ptr[j];
      FloatType new_dist = EuclideanDistWithCheck<FloatType, IdType>(
          points + point_idx * feature_size,
          points + twohop_new_c * feature_size, feature_size, worst_dist);

      if (FlaggedHeapInsert<FloatType, IdType>(
              current_neighbors, current_dists, current_flags, twohop_new_c,
              new_dist, true, k, true)) {
        ++current_num_updates;
        worst_dist = current_dists[0];
      }
    }

    // new - old
    for (IdType j = 1; j <= num_twohop_old; ++j) {
      IdType twohop_old_c = twohop_old_ptr[j];
      FloatType new_dist = EuclideanDistWithCheck<FloatType, IdType>(
          points + point_idx * feature_size,
          points + twohop_old_c * feature_size, feature_size, worst_dist);

      if (FlaggedHeapInsert<FloatType, IdType>(
              current_neighbors, current_dists, current_flags, twohop_old_c,
              new_dist, true, k, true)) {
        ++current_num_updates;
        worst_dist = current_dists[0];
      }
    }
  }

  // process old candidates
  for (IdType i = 1; i <= num_old_candidates; ++i) {
    IdType old_c = old_candidates_ptr[i];

    // new candidates of the current old candidate
    IdType* twohop_new_ptr = new_candidates + old_c * (num_candidates + 1);
    IdType num_twohop_new = twohop_new_ptr[0];
    FloatType worst_dist = current_dists[0];

    // old - new
    for (IdType j = 1; j <= num_twohop_new; ++j) {
      IdType twohop_new_c = twohop_new_ptr[j];
      FloatType new_dist = EuclideanDistWithCheck<FloatType, IdType>(
          points + point_idx * feature_size,
          points + twohop_new_c * feature_size, feature_size, worst_dist);

      if (FlaggedHeapInsert<FloatType, IdType>(
              current_neighbors, current_dists, current_flags, twohop_new_c,
              new_dist, true, k, true)) {
        ++current_num_updates;
        worst_dist = current_dists[0];
      }
    }
  }

  num_updates[point_idx] = current_num_updates;
}

}  // namespace impl

template <DGLDeviceType XPU, typename FloatType, typename IdType>
void KNN(
    const NDArray& data_points, const IdArray& data_offsets,
    const NDArray& query_points, const IdArray& query_offsets, const int k,
    IdArray result, const std::string& algorithm) {
  if (algorithm == std::string("bruteforce")) {
    impl::BruteForceKNNCuda<FloatType, IdType>(
        data_points, data_offsets, query_points, query_offsets, k, result);
  } else if (algorithm == std::string("bruteforce-sharemem")) {
    impl::BruteForceKNNSharedCuda<FloatType, IdType>(
        data_points, data_offsets, query_points, query_offsets, k, result);
  } else {
    LOG(FATAL) << "Algorithm " << algorithm << " is not supported on CUDA.";
  }
}

template <DGLDeviceType XPU, typename FloatType, typename IdType>
void NNDescent(
    const NDArray& points, const IdArray& offsets, IdArray result, const int k,
    const int num_iters, const int num_candidates, const double delta) {
  hipStream_t stream = runtime::getCurrentCUDAStream();
  const auto& ctx = points->ctx;
  auto device = runtime::DeviceAPI::Get(ctx);
  const int64_t num_nodes = points->shape[0];
  const int64_t feature_size = points->shape[1];
  const int64_t batch_size = offsets->shape[0] - 1;
  const IdType* offsets_data = offsets.Ptr<IdType>();
  const FloatType* points_data = points.Ptr<FloatType>();

  IdType* central_nodes = result.Ptr<IdType>();
  IdType* neighbors = central_nodes + k * num_nodes;
  uint64_t seed;
  int warp_size = 0;
  CUDA_CALL(
      hipDeviceGetAttribute(&warp_size, hipDeviceAttributeWarpSize, ctx.device_id));
  // We don't need large block sizes, since there's not much inter-thread
  // communication
  int64_t block_size = warp_size;
  int64_t num_blocks = (num_nodes - 1) / block_size + 1;

  // allocate space for candidates, distances and flags
  // we use the first element in candidate array to represent length
  IdType* new_candidates = static_cast<IdType*>(device->AllocWorkspace(
      ctx, num_nodes * (num_candidates + 1) * sizeof(IdType)));
  IdType* old_candidates = static_cast<IdType*>(device->AllocWorkspace(
      ctx, num_nodes * (num_candidates + 1) * sizeof(IdType)));
  IdType* num_updates = static_cast<IdType*>(
      device->AllocWorkspace(ctx, num_nodes * sizeof(IdType)));
  FloatType* distances = static_cast<FloatType*>(
      device->AllocWorkspace(ctx, num_nodes * k * sizeof(IdType)));
  bool* flags = static_cast<bool*>(
      device->AllocWorkspace(ctx, num_nodes * k * sizeof(IdType)));

  size_t sum_temp_size = 0;
  IdType total_num_updates = 0;
  IdType* total_num_updates_d =
      static_cast<IdType*>(device->AllocWorkspace(ctx, sizeof(IdType)));

  CUDA_CALL(hipcub::DeviceReduce::Sum(
      nullptr, sum_temp_size, num_updates, total_num_updates_d, num_nodes,
      stream));
  IdType* sum_temp_storage =
      static_cast<IdType*>(device->AllocWorkspace(ctx, sum_temp_size));

  // random initialize neighbors
  seed = RandomEngine::ThreadLocal()->RandInt<uint64_t>(
      std::numeric_limits<uint64_t>::max());
  CUDA_KERNEL_CALL(
      impl::RandomInitNeighborsKernel, num_blocks, block_size, 0, stream,
      points_data, offsets_data, central_nodes, neighbors, distances, flags, k,
      feature_size, batch_size, seed);

  for (int i = 0; i < num_iters; ++i) {
    // select candidates
    seed = RandomEngine::ThreadLocal()->RandInt<uint64_t>(
        std::numeric_limits<uint64_t>::max());
    CUDA_KERNEL_CALL(
        impl::FindCandidatesKernel, num_blocks, block_size, 0, stream,
        offsets_data, new_candidates, old_candidates, neighbors, flags, seed,
        batch_size, num_candidates, k);

    // update
    CUDA_KERNEL_CALL(
        impl::UpdateNeighborsKernel, num_blocks, block_size, 0, stream,
        points_data, offsets_data, neighbors, new_candidates, old_candidates,
        distances, flags, num_updates, batch_size, num_candidates, k,
        feature_size);

    total_num_updates = 0;
    CUDA_CALL(hipcub::DeviceReduce::Sum(
        sum_temp_storage, sum_temp_size, num_updates, total_num_updates_d,
        num_nodes, stream));
    device->CopyDataFromTo(
        total_num_updates_d, 0, &total_num_updates, 0, sizeof(IdType), ctx,
        DGLContext{kDGLCPU, 0}, offsets->dtype);

    if (total_num_updates <= static_cast<IdType>(delta * k * num_nodes)) {
      break;
    }
  }

  device->FreeWorkspace(ctx, new_candidates);
  device->FreeWorkspace(ctx, old_candidates);
  device->FreeWorkspace(ctx, num_updates);
  device->FreeWorkspace(ctx, distances);
  device->FreeWorkspace(ctx, flags);
  device->FreeWorkspace(ctx, total_num_updates_d);
  device->FreeWorkspace(ctx, sum_temp_storage);
}

template void KNN<kDGLCUDA, float, int32_t>(
    const NDArray& data_points, const IdArray& data_offsets,
    const NDArray& query_points, const IdArray& query_offsets, const int k,
    IdArray result, const std::string& algorithm);
template void KNN<kDGLCUDA, float, int64_t>(
    const NDArray& data_points, const IdArray& data_offsets,
    const NDArray& query_points, const IdArray& query_offsets, const int k,
    IdArray result, const std::string& algorithm);
template void KNN<kDGLCUDA, double, int32_t>(
    const NDArray& data_points, const IdArray& data_offsets,
    const NDArray& query_points, const IdArray& query_offsets, const int k,
    IdArray result, const std::string& algorithm);
template void KNN<kDGLCUDA, double, int64_t>(
    const NDArray& data_points, const IdArray& data_offsets,
    const NDArray& query_points, const IdArray& query_offsets, const int k,
    IdArray result, const std::string& algorithm);

template void NNDescent<kDGLCUDA, float, int32_t>(
    const NDArray& points, const IdArray& offsets, IdArray result, const int k,
    const int num_iters, const int num_candidates, const double delta);
template void NNDescent<kDGLCUDA, float, int64_t>(
    const NDArray& points, const IdArray& offsets, IdArray result, const int k,
    const int num_iters, const int num_candidates, const double delta);
template void NNDescent<kDGLCUDA, double, int32_t>(
    const NDArray& points, const IdArray& offsets, IdArray result, const int k,
    const int num_iters, const int num_candidates, const double delta);
template void NNDescent<kDGLCUDA, double, int64_t>(
    const NDArray& points, const IdArray& offsets, IdArray result, const int k,
    const int num_iters, const int num_candidates, const double delta);

}  // namespace transform
}  // namespace dgl
