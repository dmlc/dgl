#include "hip/hip_runtime.h"
/**
 *  Copyright (c) 2019 by Contributors
 * @file geometry/cuda/geometry_op_impl.cc
 * @brief Geometry operator CUDA implementation
 */
#include <dgl/array.h>

#include "../../c_api_common.h"
#include "../../runtime/cuda/cuda_common.h"
#include "../geometry_op.h"

#define THREADS 1024

namespace dgl {
namespace geometry {
namespace impl {

/**
 * @brief Farthest Point Sampler without the need to compute all pairs of
 * distance.
 *
 * The input array has shape (N, d), where N is the number of points, and d is
 * the dimension. It consists of a (flatten) batch of point clouds.
 *
 * In each batch, the algorithm starts with the sample index specified by
 * ``start_idx``. Then for each point, we maintain the minimum to-sample
 * distance. Finally, we pick the point with the maximum such distance. This
 * process will be repeated for ``sample_points`` - 1 times.
 */
template <typename FloatType, typename IdType>
__global__ void fps_kernel(
    const FloatType* array_data, const int64_t batch_size,
    const int64_t sample_points, const int64_t point_in_batch,
    const int64_t dim, const IdType* start_idx, FloatType* dist_data,
    IdType* ret_data) {
  const int64_t thread_idx = threadIdx.x;
  const int64_t batch_idx = blockIdx.x;

  const int64_t array_start = point_in_batch * batch_idx;
  const int64_t ret_start = sample_points * batch_idx;

  __shared__ FloatType dist_max_ht[THREADS];
  __shared__ int64_t dist_argmax_ht[THREADS];

  // start with random initialization
  if (thread_idx == 0) {
    ret_data[ret_start] = (IdType)(start_idx[batch_idx]);
  }

  // sample the rest `sample_points - 1` points
  for (auto i = 0; i < sample_points - 1; i++) {
    __syncthreads();

    // the last sampled point
    int64_t sample_idx = (int64_t)(ret_data[ret_start + i]);
    dist_argmax_ht[thread_idx] = 0;
    dist_max_ht[thread_idx] = (FloatType)(-1.);

    // multi-thread distance calculation
    for (auto j = thread_idx; j < point_in_batch; j += THREADS) {
      FloatType one_dist = (FloatType)(0.);
      for (auto d = 0; d < dim; d++) {
        FloatType tmp = array_data[(array_start + j) * dim + d] -
                        array_data[(array_start + sample_idx) * dim + d];
        one_dist += tmp * tmp;
      }

      if (i == 0 || dist_data[array_start + j] > one_dist) {
        dist_data[array_start + j] = one_dist;
      }

      if (dist_data[array_start + j] > dist_max_ht[thread_idx]) {
        dist_argmax_ht[thread_idx] = j;
        dist_max_ht[thread_idx] = dist_data[array_start + j];
      }
    }

    __syncthreads();

    if (thread_idx == 0) {
      FloatType best = dist_max_ht[0];
      int64_t best_idx = dist_argmax_ht[0];
      for (auto j = 1; j < THREADS; j++) {
        if (dist_max_ht[j] > best) {
          best = dist_max_ht[j];
          best_idx = dist_argmax_ht[j];
        }
      }
      ret_data[ret_start + i + 1] = (IdType)(best_idx);
    }
  }
}

template <DGLDeviceType XPU, typename FloatType, typename IdType>
void FarthestPointSampler(
    NDArray array, int64_t batch_size, int64_t sample_points, NDArray dist,
    IdArray start_idx, IdArray result) {
  hipStream_t stream = runtime::getCurrentCUDAStream();

  const FloatType* array_data = static_cast<FloatType*>(array->data);

  const int64_t point_in_batch = array->shape[0] / batch_size;
  const int64_t dim = array->shape[1];

  // return value
  IdType* ret_data = static_cast<IdType*>(result->data);

  // distance
  FloatType* dist_data = static_cast<FloatType*>(dist->data);

  // sample for each cloud in the batch
  IdType* start_idx_data = static_cast<IdType*>(start_idx->data);
  CUDA_CALL(hipSetDevice(array->ctx.device_id));

  CUDA_KERNEL_CALL(
      fps_kernel, batch_size, THREADS, 0, stream, array_data, batch_size,
      sample_points, point_in_batch, dim, start_idx_data, dist_data, ret_data);
}

template void FarthestPointSampler<kDGLCUDA, float, int32_t>(
    NDArray array, int64_t batch_size, int64_t sample_points, NDArray dist,
    IdArray start_idx, IdArray result);
template void FarthestPointSampler<kDGLCUDA, float, int64_t>(
    NDArray array, int64_t batch_size, int64_t sample_points, NDArray dist,
    IdArray start_idx, IdArray result);
template void FarthestPointSampler<kDGLCUDA, double, int32_t>(
    NDArray array, int64_t batch_size, int64_t sample_points, NDArray dist,
    IdArray start_idx, IdArray result);
template void FarthestPointSampler<kDGLCUDA, double, int64_t>(
    NDArray array, int64_t batch_size, int64_t sample_points, NDArray dist,
    IdArray start_idx, IdArray result);
template void FarthestPointSampler<kDGLCUDA, __half, int32_t>(
    NDArray array, int64_t batch_size, int64_t sample_points, NDArray dist,
    IdArray start_idx, IdArray result);
template void FarthestPointSampler<kDGLCUDA, __half, int64_t>(
    NDArray array, int64_t batch_size, int64_t sample_points, NDArray dist,
    IdArray start_idx, IdArray result);
#if BF16_ENABLED
template void FarthestPointSampler<kDGLCUDA, __hip_bfloat16, int32_t>(
    NDArray array, int64_t batch_size, int64_t sample_points, NDArray dist,
    IdArray start_idx, IdArray result);
template void FarthestPointSampler<kDGLCUDA, __hip_bfloat16, int64_t>(
    NDArray array, int64_t batch_size, int64_t sample_points, NDArray dist,
    IdArray start_idx, IdArray result);
#endif  // BF16_ENABLED
}  // namespace impl
}  // namespace geometry
}  // namespace dgl
