#include "hip/hip_runtime.h"
/*!
 *   Copyright (c) 2022, NVIDIA Corporation
 *   Copyright (c) 2022, GT-TDAlab (Muhammed Fatih Balin & Umit V. Catalyurek)
 *   All rights reserved.
 *
 *   Licensed under the Apache License, Version 2.0 (the "License");
 *   you may not use this file except in compliance with the License.
 *   You may obtain a copy of the License at
 *
 *       http://www.apache.org/licenses/LICENSE-2.0
 *
 *   Unless required by applicable law or agreed to in writing, software
 *   distributed under the License is distributed on an "AS IS" BASIS,
 *   WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *   See the License for the specific language governing permissions and
 *   limitations under the License.
 *
 * @file array/cuda/labor_sampling.cu
 * @brief labor sampling
 */

#include <dgl/aten/coo.h>
#include <dgl/random.h>
#include <dgl/runtime/device_api.h>
#include <thrust/binary_search.h>
#include <thrust/copy.h>
#include <thrust/execution_policy.h>
#include <thrust/gather.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/reduce.h>
#include <thrust/shuffle.h>
#include <thrust/transform.h>
#include <thrust/zip_function.h>

#include <algorithm>
#include <hipcub/hipcub.hpp>  // NOLINT
#include <limits>
#include <numeric>
#include <type_traits>
#include <utility>

#include "../../array/cuda/utils.h"
#include "../../random/continuous_seed.h"
#include "../../runtime/cuda/cuda_common.h"
#include "./functor.cuh"
#include "./spmm.cuh"

namespace dgl {
namespace aten {
namespace impl {

using dgl::random::continuous_seed;

constexpr int BLOCK_SIZE = 128;
constexpr int CTA_SIZE = 128;
constexpr double eps = 0.0001;

namespace {

template <typename IdType>
struct TransformOp {
  const IdType* idx_coo;
  const IdType* rows;
  const IdType* indptr;
  const IdType* subindptr;
  const IdType* indices;
  const IdType* data_arr;
  bool is_pinned;
  __host__ __device__ auto operator()(IdType idx) {
    const auto in_row = idx_coo[idx];
    const auto row = rows[in_row];
    const auto in_idx = indptr[in_row] + idx - subindptr[in_row];
    const auto u = indices[is_pinned ? idx : in_idx];
    const auto data = data_arr ? data_arr[in_idx] : in_idx;
    return thrust::make_tuple(row, u, data);
  }
};

template <
    typename IdType, typename FloatType, typename probs_t, typename A_t,
    typename B_t>
struct TransformOpImp {
  probs_t probs;
  A_t A;
  B_t B;
  const IdType* idx_coo;
  const IdType* rows;
  const FloatType* cs;
  const IdType* indptr;
  const IdType* subindptr;
  const IdType* indices;
  const IdType* data_arr;
  bool is_pinned;
  __host__ __device__ auto operator()(IdType idx) {
    const auto ps = probs[idx];
    const auto in_row = idx_coo[idx];
    const auto c = cs[in_row];
    const auto row = rows[in_row];
    const auto in_idx = indptr[in_row] + idx - subindptr[in_row];
    const auto u = indices[is_pinned ? idx : in_idx];
    const auto w = A[in_idx];
    const auto w2 = B[in_idx];
    const auto data = data_arr ? data_arr[in_idx] : in_idx;
    return thrust::make_tuple(
        in_row, row, u, data, w / min((FloatType)1, c * w2 * ps));
  }
};

template <typename FloatType>
struct StencilOp {
  const FloatType* cs;
  template <typename IdType>
  __host__ __device__ auto operator()(
      IdType in_row, FloatType ps, FloatType rnd) {
    return rnd <= cs[in_row] * ps;
  }
};

template <typename IdType, typename FloatType, typename ps_t, typename A_t>
struct StencilOpFused {
  const continuous_seed seed;
  const IdType* idx_coo;
  const FloatType* cs;
  const ps_t probs;
  const A_t A;
  const IdType* subindptr;
  const IdType* indptr;
  const IdType* indices;
  const IdType* nids;
  bool is_pinned;
  __device__ auto operator()(IdType idx) {
    const auto in_row = idx_coo[idx];
    const auto ps = probs[idx];
    IdType rofs = idx - subindptr[in_row];
    const auto in_idx = indptr[in_row] + rofs;
    const auto u = indices[is_pinned ? idx : in_idx];
    const auto t = nids ? nids[u] : u;  // t in the paper
    // rolled random number r_t is a function of the random_seed and t
    const float rnd = seed.uniform(t);
    return rnd <= cs[in_row] * A[in_idx] * ps;
  }
};

template <typename IdType, typename FloatType>
struct TransformOpMean {
  const IdType* ds;
  const FloatType* ws;
  __host__ __device__ auto operator()(IdType idx, FloatType ps) {
    return ps * ds[idx] / ws[idx];
  }
};

struct TransformOpMinWith1 {
  template <typename FloatType>
  __host__ __device__ auto operator()(FloatType x) {
    return min((FloatType)1, x);
  }
};

template <typename IdType>
struct IndptrFunc {
  const IdType* indptr;
  const IdType* in_deg;
  __host__ __device__ auto operator()(IdType row) {
    return indptr[row] + (in_deg ? in_deg[row] : 0);
  }
};

template <typename FloatType>
struct SquareFunc {
  __host__ __device__ auto operator()(FloatType x) {
    return thrust::make_tuple(x, x * x);
  }
};

struct TupleSum {
  template <typename T>
  __host__ __device__ T operator()(const T& a, const T& b) const {
    return thrust::make_tuple(
        thrust::get<0>(a) + thrust::get<0>(b),
        thrust::get<1>(a) + thrust::get<1>(b));
  }
};

template <typename IdType, typename FloatType>
struct DegreeFunc {
  const IdType num_picks;
  const IdType* rows;
  const IdType* indptr;
  IdType* in_deg;
  IdType* inrow_indptr;
  FloatType* cs;
  __host__ __device__ auto operator()(IdType tIdx) {
    const auto out_row = rows[tIdx];
    const auto indptr_val = indptr[out_row];
    const auto d = indptr[out_row + 1] - indptr_val;
    in_deg[tIdx] = d;
    inrow_indptr[tIdx] = indptr_val;
    cs[tIdx] = num_picks / (FloatType)d;
  }
};

template <typename IdType, typename FloatType>
__global__ void _CSRRowWiseOneHopExtractorKernel(
    const continuous_seed seed, const IdType hop_size,
    const IdType* const indptr, const IdType* const subindptr,
    const IdType* const indices, const IdType* const idx_coo,
    const IdType* const nids, const FloatType* const A, FloatType* const rands,
    IdType* const hop, FloatType* const A_l) {
  IdType tx = static_cast<IdType>(blockIdx.x) * blockDim.x + threadIdx.x;
  const int stride_x = gridDim.x * blockDim.x;

  while (tx < hop_size) {
    IdType rpos = idx_coo[tx];
    IdType rofs = tx - subindptr[rpos];
    const auto in_idx = indptr[rpos] + rofs;
    const auto not_pinned = indices != hop;
    const auto u = indices[not_pinned ? in_idx : tx];
    if (not_pinned) hop[tx] = u;
    const auto t = nids ? nids[u] : u;
    if (A) A_l[tx] = A[in_idx];
    // rolled random number r_t is a function of the random_seed and t
    rands[tx] = (FloatType)seed.uniform(t);
    tx += stride_x;
  }
}

constexpr int CACHE_LINE_SIZE = 128;

template <typename IdType>
struct AlignmentFunc {
  static_assert(CACHE_LINE_SIZE % sizeof(IdType) == 0);
  const IdType* in_deg;
  const int64_t* perm;
  IdType num_rows;
  __host__ __device__ auto operator()(IdType row) {
    constexpr int num_elements = CACHE_LINE_SIZE / sizeof(IdType);
    return in_deg[perm ? perm[row % num_rows] : row] + num_elements - 1;
  }
};

template <typename IdType>
__global__ void _CSRRowWiseOneHopExtractorAlignedKernel(
    const IdType hop_size, const IdType num_rows, const IdType* const indptr,
    const IdType* const subindptr, const IdType* const subindptr_aligned,
    const IdType* const indices, IdType* const hop, const int64_t* const perm) {
  IdType tx = static_cast<IdType>(blockIdx.x) * blockDim.x + threadIdx.x;
  const int stride_x = gridDim.x * blockDim.x;

  while (tx < hop_size) {
    const IdType rpos_ =
        dgl::cuda::_UpperBound(subindptr_aligned, num_rows, tx) - 1;
    const IdType rpos = perm ? perm[rpos_] : rpos_;
    const auto out_row = subindptr[rpos];
    const auto d = subindptr[rpos + 1] - out_row;
    const int offset =
        ((uint64_t)(indices + indptr[rpos] - subindptr_aligned[rpos_]) %
         CACHE_LINE_SIZE) /
        sizeof(IdType);
    const IdType rofs = tx - subindptr_aligned[rpos_] - offset;
    if (rofs >= 0 && rofs < d) {
      const auto in_idx = indptr[rpos] + rofs;
      assert((uint64_t)(indices + in_idx - tx) % CACHE_LINE_SIZE == 0);
      const auto u = indices[in_idx];
      hop[out_row + rofs] = u;
    }
    tx += stride_x;
  }
}

template <typename IdType, typename FloatType, int BLOCK_CTAS, int TILE_SIZE>
__global__ void _CSRRowWiseLayerSampleDegreeKernel(
    const IdType num_picks, const IdType num_rows, FloatType* const cs,
    const FloatType* const ds, const FloatType* const d2s,
    const IdType* const indptr, const FloatType* const probs,
    const FloatType* const A, const IdType* const subindptr) {
  typedef hipcub::BlockReduce<FloatType, BLOCK_SIZE> BlockReduce;
  __shared__ typename BlockReduce::TempStorage temp_storage;
  __shared__ FloatType var_1_bcast[BLOCK_CTAS];

  // we assign one warp per row
  assert(blockDim.x == CTA_SIZE);
  assert(blockDim.y == BLOCK_CTAS);

  IdType out_row = blockIdx.x * TILE_SIZE + threadIdx.y;
  const auto last_row =
      min(static_cast<IdType>(blockIdx.x + 1) * TILE_SIZE, num_rows);

  constexpr FloatType ONE = 1;

  while (out_row < last_row) {
    const auto in_row_start = indptr[out_row];
    const auto out_row_start = subindptr[out_row];

    const IdType degree = subindptr[out_row + 1] - out_row_start;

    if (degree > 0) {
      // stands for k in in arXiv:2210.13339, i.e. fanout
      const auto k = min(num_picks, degree);
      // slightly better than NS
      const FloatType d_ = ds ? ds[out_row] : degree;
      // stands for right handside of Equation (22) in arXiv:2210.13339
      FloatType var_target =
          d_ * d_ / k + (ds ? d2s[out_row] - d_ * d_ / degree : 0);

      auto c = cs[out_row];
      const int num_valid = min(degree, (IdType)CTA_SIZE);
      // stands for left handside of Equation (22) in arXiv:2210.13339
      FloatType var_1;
      do {
        var_1 = 0;
        if (A) {
          for (int idx = threadIdx.x; idx < degree; idx += CTA_SIZE) {
            const auto w = A[in_row_start + idx];
            const auto ps = probs ? probs[out_row_start + idx] : w;
            var_1 += w > 0 ? w * w / min(ONE, c * ps) : 0;
          }
        } else {
          for (int idx = threadIdx.x; idx < degree; idx += CTA_SIZE) {
            const auto ps = probs[out_row_start + idx];
            var_1 += 1 / min(ONE, c * ps);
          }
        }
        var_1 = BlockReduce(temp_storage).Sum(var_1, num_valid);
        if (threadIdx.x == 0) var_1_bcast[threadIdx.y] = var_1;
        __syncthreads();
        var_1 = var_1_bcast[threadIdx.y];

        c *= var_1 / var_target;
      } while (min(var_1, var_target) / max(var_1, var_target) < 1 - eps);

      if (threadIdx.x == 0) cs[out_row] = c;
    }

    out_row += BLOCK_CTAS;
  }
}

}  // namespace

template <typename IdType>
int log_size(const IdType size) {
  if (size <= 0) return 0;
  for (int i = 0; i < static_cast<int>(sizeof(IdType)) * 8; i++)
    if (((size - 1) >> i) == 0) return i;
  return sizeof(IdType) * 8;
}

template <typename IdType, typename FloatType, typename exec_policy_t>
void compute_importance_sampling_probabilities(
    CSRMatrix mat, const IdType hop_size, hipStream_t stream,
    const continuous_seed seed, const IdType num_rows, const IdType* indptr,
    const IdType* subindptr, const IdType* indices, IdArray idx_coo_arr,
    const IdType* nids,
    FloatArray cs_arr,  // holds the computed cs values, has size num_rows
    const bool weighted, const FloatType* A, const FloatType* ds,
    const FloatType* d2s, const IdType num_picks, DGLContext ctx,
    const runtime::CUDAWorkspaceAllocator& allocator,
    const exec_policy_t& exec_policy, const int importance_sampling,
    IdType* hop_1,  // holds the contiguous one-hop neighborhood, has size |E|
    FloatType* rands,  // holds the rolled random numbers r_t for each edge, has
                       // size |E|
    FloatType* probs_found) {  // holds the computed pi_t values for each edge,
                               // has size |E|
  auto device = runtime::DeviceAPI::Get(ctx);
  auto idx_coo = idx_coo_arr.Ptr<IdType>();
  auto cs = cs_arr.Ptr<FloatType>();
  FloatArray A_l_arr = weighted
                           ? NewFloatArray(hop_size, ctx, sizeof(FloatType) * 8)
                           : NullArray();
  auto A_l = A_l_arr.Ptr<FloatType>();

  const int max_log_num_vertices = log_size(mat.num_cols);

  {  // extracts the onehop neighborhood cols to a contiguous range into hop_1
    const dim3 block(BLOCK_SIZE);
    const dim3 grid((hop_size + BLOCK_SIZE - 1) / BLOCK_SIZE);
    CUDA_KERNEL_CALL(
        (_CSRRowWiseOneHopExtractorKernel<IdType, FloatType>), grid, block, 0,
        stream, seed, hop_size, indptr, subindptr, indices, idx_coo, nids,
        weighted ? A : nullptr, rands, hop_1, A_l);
  }
  int64_t hop_uniq_size = 0;
  IdArray hop_new_arr = NewIdArray(hop_size, ctx, sizeof(IdType) * 8);
  auto hop_new = hop_new_arr.Ptr<IdType>();
  auto hop_unique = allocator.alloc_unique<IdType>(hop_size);
  // After this block, hop_unique holds the unique set of one-hop neighborhood
  // and hop_new holds the relabeled hop_1, idx_coo already holds relabeled
  // destination. hop_unique[hop_new] == hop_1 holds
  {
    auto hop_2 = allocator.alloc_unique<IdType>(hop_size);
    auto hop_3 = allocator.alloc_unique<IdType>(hop_size);

    device->CopyDataFromTo(
        hop_1, 0, hop_2.get(), 0, sizeof(IdType) * hop_size, ctx, ctx,
        mat.indptr->dtype);

    hipcub::DoubleBuffer<IdType> hop_b(hop_2.get(), hop_3.get());

    {
      std::size_t temp_storage_bytes = 0;
      CUDA_CALL(hipcub::DeviceRadixSort::SortKeys(
          nullptr, temp_storage_bytes, hop_b, hop_size, 0, max_log_num_vertices,
          stream));

      auto temp = allocator.alloc_unique<char>(temp_storage_bytes);

      CUDA_CALL(hipcub::DeviceRadixSort::SortKeys(
          temp.get(), temp_storage_bytes, hop_b, hop_size, 0,
          max_log_num_vertices, stream));
    }

    auto hop_counts = allocator.alloc_unique<IdType>(hop_size + 1);
    auto hop_unique_size = allocator.alloc_unique<int64_t>(1);

    {
      std::size_t temp_storage_bytes = 0;
      CUDA_CALL(hipcub::DeviceRunLengthEncode::Encode(
          nullptr, temp_storage_bytes, hop_b.Current(), hop_unique.get(),
          hop_counts.get(), hop_unique_size.get(), hop_size, stream));

      auto temp = allocator.alloc_unique<char>(temp_storage_bytes);

      CUDA_CALL(hipcub::DeviceRunLengthEncode::Encode(
          temp.get(), temp_storage_bytes, hop_b.Current(), hop_unique.get(),
          hop_counts.get(), hop_unique_size.get(), hop_size, stream));

      device->CopyDataFromTo(
          hop_unique_size.get(), 0, &hop_uniq_size, 0, sizeof(hop_uniq_size),
          ctx, DGLContext{kDGLCPU, 0}, mat.indptr->dtype);
    }

    thrust::lower_bound(
        exec_policy, hop_unique.get(), hop_unique.get() + hop_uniq_size, hop_1,
        hop_1 + hop_size, hop_new);
  }

  // @todo Consider creating a CSC because the SpMV will be done multiple times.
  COOMatrix rmat(
      num_rows, hop_uniq_size, idx_coo_arr, hop_new_arr, NullArray(), true,
      mat.sorted);

  BcastOff bcast_off;
  bcast_off.use_bcast = false;
  bcast_off.out_len = 1;
  bcast_off.lhs_len = 1;
  bcast_off.rhs_len = 1;

  FloatArray probs_arr =
      NewFloatArray(hop_uniq_size, ctx, sizeof(FloatType) * 8);
  auto probs_1 = probs_arr.Ptr<FloatType>();
  FloatArray probs_arr_2 =
      NewFloatArray(hop_uniq_size, ctx, sizeof(FloatType) * 8);
  auto probs = probs_arr_2.Ptr<FloatType>();
  auto arg_u = NewIdArray(hop_uniq_size, ctx, sizeof(IdType) * 8);
  auto arg_e = NewIdArray(hop_size, ctx, sizeof(IdType) * 8);

  double prev_ex_nodes = hop_uniq_size;

  for (int iters = 0; iters < importance_sampling || importance_sampling < 0;
       iters++) {
    if (weighted && iters == 0) {
      cuda::SpMMCoo<
          IdType, FloatType, cuda::binary::Mul<FloatType>,
          cuda::reduce::Max<IdType, FloatType, true>>(
          bcast_off, rmat, cs_arr, A_l_arr, probs_arr_2, arg_u, arg_e);
    } else {
      cuda::SpMMCoo<
          IdType, FloatType, cuda::binary::CopyLhs<FloatType>,
          cuda::reduce::Max<IdType, FloatType, true>>(
          bcast_off, rmat, cs_arr, NullArray(), iters ? probs_arr : probs_arr_2,
          arg_u, arg_e);
    }

    if (iters)
      thrust::transform(
          exec_policy, probs_1, probs_1 + hop_uniq_size, probs, probs,
          thrust::multiplies<FloatType>{});

    thrust::gather(
        exec_policy, hop_new, hop_new + hop_size, probs, probs_found);

    {
      constexpr int BLOCK_CTAS = BLOCK_SIZE / CTA_SIZE;
      // the number of rows each thread block will cover
      constexpr int TILE_SIZE = BLOCK_CTAS;
      const dim3 block(CTA_SIZE, BLOCK_CTAS);
      const dim3 grid((num_rows + TILE_SIZE - 1) / TILE_SIZE);
      CUDA_KERNEL_CALL(
          (_CSRRowWiseLayerSampleDegreeKernel<
              IdType, FloatType, BLOCK_CTAS, TILE_SIZE>),
          grid, block, 0, stream, (IdType)num_picks, num_rows, cs,
          weighted ? ds : nullptr, weighted ? d2s : nullptr, indptr,
          probs_found, A, subindptr);
    }

    {
      auto probs_min_1 =
          thrust::make_transform_iterator(probs, TransformOpMinWith1{});
      const double cur_ex_nodes = thrust::reduce(
          exec_policy, probs_min_1, probs_min_1 + hop_uniq_size, 0.0);
      if (cur_ex_nodes / prev_ex_nodes >= 1 - eps) break;
      prev_ex_nodes = cur_ex_nodes;
    }
  }
}

/////////////////////////////// CSR ///////////////////////////////

template <DGLDeviceType XPU, typename IdType, typename FloatType>
std::pair<COOMatrix, FloatArray> CSRLaborSampling(
    CSRMatrix mat, IdArray rows_arr, const int64_t num_picks,
    FloatArray prob_arr, const int importance_sampling, IdArray random_seed_arr,
    float seed2_contribution, IdArray NIDs) {
  const bool weighted = !IsNullArray(prob_arr);

  const auto& ctx = rows_arr->ctx;

  runtime::CUDAWorkspaceAllocator allocator(ctx);

  const auto stream = runtime::getCurrentCUDAStream();
  const auto exec_policy = thrust::cuda::par_nosync(allocator).on(stream);

  auto device = runtime::DeviceAPI::Get(ctx);

  const IdType num_rows = rows_arr->shape[0];
  IdType* const rows = rows_arr.Ptr<IdType>();
  IdType* const nids = IsNullArray(NIDs) ? nullptr : NIDs.Ptr<IdType>();
  FloatType* const A = prob_arr.Ptr<FloatType>();

  IdType* const indptr_ = mat.indptr.Ptr<IdType>();
  IdType* const indices_ = mat.indices.Ptr<IdType>();
  IdType* const data = CSRHasData(mat) ? mat.data.Ptr<IdType>() : nullptr;

  // Read indptr only once in case it is pinned and access is slow.
  auto indptr = allocator.alloc_unique<IdType>(num_rows);
  // compute in-degrees
  auto in_deg = allocator.alloc_unique<IdType>(num_rows + 1);
  // cs stands for c_s in arXiv:2210.13339
  FloatArray cs_arr = NewFloatArray(num_rows, ctx, sizeof(FloatType) * 8);
  auto cs = cs_arr.Ptr<FloatType>();
  // ds stands for A_{*s} in arXiv:2210.13339
  FloatArray ds_arr = weighted
                          ? NewFloatArray(num_rows, ctx, sizeof(FloatType) * 8)
                          : NullArray();
  auto ds = ds_arr.Ptr<FloatType>();
  // d2s stands for (A^2)_{*s} in arXiv:2210.13339, ^2 is elementwise.
  FloatArray d2s_arr = weighted
                           ? NewFloatArray(num_rows, ctx, sizeof(FloatType) * 8)
                           : NullArray();
  auto d2s = d2s_arr.Ptr<FloatType>();

  thrust::counting_iterator<IdType> iota(0);
  thrust::for_each(
      exec_policy, iota, iota + num_rows,
      DegreeFunc<IdType, FloatType>{
          (IdType)num_picks, rows, indptr_, in_deg.get(), indptr.get(), cs});

  if (weighted) {
    auto b_offsets = thrust::make_transform_iterator(
        iota, IndptrFunc<IdType>{indptr.get(), nullptr});
    auto e_offsets = thrust::make_transform_iterator(
        iota, IndptrFunc<IdType>{indptr.get(), in_deg.get()});

    auto A_A2 = thrust::make_transform_iterator(A, SquareFunc<FloatType>{});
    auto ds_d2s = thrust::make_zip_iterator(ds, d2s);

    size_t prefix_temp_size = 0;
    CUDA_CALL(hipcub::DeviceSegmentedReduce::Reduce(
        nullptr, prefix_temp_size, A_A2, ds_d2s, num_rows, b_offsets, e_offsets,
        TupleSum{}, thrust::make_tuple((FloatType)0, (FloatType)0), stream));
    auto temp = allocator.alloc_unique<char>(prefix_temp_size);
    CUDA_CALL(hipcub::DeviceSegmentedReduce::Reduce(
        temp.get(), prefix_temp_size, A_A2, ds_d2s, num_rows, b_offsets,
        e_offsets, TupleSum{}, thrust::make_tuple((FloatType)0, (FloatType)0),
        stream));
  }

  // fill subindptr
  IdArray subindptr_arr = NewIdArray(num_rows + 1, ctx, sizeof(IdType) * 8);
  auto subindptr = subindptr_arr.Ptr<IdType>();

  IdType hop_size;
  {
    size_t prefix_temp_size = 0;
    CUDA_CALL(hipcub::DeviceScan::ExclusiveSum(
        nullptr, prefix_temp_size, in_deg.get(), subindptr, num_rows + 1,
        stream));
    auto temp = allocator.alloc_unique<char>(prefix_temp_size);
    CUDA_CALL(hipcub::DeviceScan::ExclusiveSum(
        temp.get(), prefix_temp_size, in_deg.get(), subindptr, num_rows + 1,
        stream));

    device->CopyDataFromTo(
        subindptr, num_rows * sizeof(hop_size), &hop_size, 0, sizeof(hop_size),
        ctx, DGLContext{kDGLCPU, 0}, mat.indptr->dtype);
  }
  IdArray hop_arr = NewIdArray(hop_size, ctx, sizeof(IdType) * 8);
  CSRMatrix smat(
      num_rows, mat.num_cols, subindptr_arr, hop_arr, NullArray(), mat.sorted);
  // @todo Consider fusing CSRToCOO into StencilOpFused kernel
  auto smatcoo = CSRToCOO(smat, false);

  auto idx_coo_arr = smatcoo.row;
  auto idx_coo = idx_coo_arr.Ptr<IdType>();

  auto hop_1 = hop_arr.Ptr<IdType>();
  const bool is_pinned = mat.indices.IsPinned();
  if (is_pinned) {
    const auto res = Sort(rows_arr, log_size(mat.num_rows));
    const int64_t* perm = static_cast<int64_t*>(res.second->data);

    IdType hop_size;  // Shadows the original one as this is temporary
    auto subindptr_aligned = allocator.alloc_unique<IdType>(num_rows + 1);
    {
      auto modified_in_deg = thrust::make_transform_iterator(
          iota, AlignmentFunc<IdType>{in_deg.get(), perm, num_rows});
      size_t prefix_temp_size = 0;
      CUDA_CALL(hipcub::DeviceScan::ExclusiveSum(
          nullptr, prefix_temp_size, modified_in_deg, subindptr_aligned.get(),
          num_rows + 1, stream));
      auto temp = allocator.alloc_unique<char>(prefix_temp_size);
      CUDA_CALL(hipcub::DeviceScan::ExclusiveSum(
          temp.get(), prefix_temp_size, modified_in_deg,
          subindptr_aligned.get(), num_rows + 1, stream));

      device->CopyDataFromTo(
          subindptr_aligned.get(), num_rows * sizeof(hop_size), &hop_size, 0,
          sizeof(hop_size), ctx, DGLContext{kDGLCPU, 0}, mat.indptr->dtype);
    }
    const dim3 block(BLOCK_SIZE);
    const dim3 grid((hop_size + BLOCK_SIZE - 1) / BLOCK_SIZE);
    CUDA_KERNEL_CALL(
        (_CSRRowWiseOneHopExtractorAlignedKernel<IdType>), grid, block, 0,
        stream, hop_size, num_rows, indptr.get(), subindptr,
        subindptr_aligned.get(), indices_, hop_1, perm);
  }
  const auto indices = is_pinned ? hop_1 : indices_;

  auto rands =
      allocator.alloc_unique<FloatType>(importance_sampling ? hop_size : 1);
  auto probs_found =
      allocator.alloc_unique<FloatType>(importance_sampling ? hop_size : 1);

  if (weighted) {
    // Recompute c for weighted graphs.
    constexpr int BLOCK_CTAS = BLOCK_SIZE / CTA_SIZE;
    // the number of rows each thread block will cover
    constexpr int TILE_SIZE = BLOCK_CTAS;
    const dim3 block(CTA_SIZE, BLOCK_CTAS);
    const dim3 grid((num_rows + TILE_SIZE - 1) / TILE_SIZE);
    CUDA_KERNEL_CALL(
        (_CSRRowWiseLayerSampleDegreeKernel<
            IdType, FloatType, BLOCK_CTAS, TILE_SIZE>),
        grid, block, 0, stream, (IdType)num_picks, num_rows, cs, ds, d2s,
        indptr.get(), nullptr, A, subindptr);
  }

  const continuous_seed random_seed =
      IsNullArray(random_seed_arr)
          ? continuous_seed(RandomEngine::ThreadLocal()->RandInt(1000000000))
          : continuous_seed(random_seed_arr, seed2_contribution);

  if (importance_sampling)
    compute_importance_sampling_probabilities<
        IdType, FloatType, decltype(exec_policy)>(
        mat, hop_size, stream, random_seed, num_rows, indptr.get(), subindptr,
        indices, idx_coo_arr, nids, cs_arr, weighted, A, ds, d2s,
        (IdType)num_picks, ctx, allocator, exec_policy, importance_sampling,
        hop_1, rands.get(), probs_found.get());

  IdArray picked_row = NewIdArray(hop_size, ctx, sizeof(IdType) * 8);
  IdArray picked_col = NewIdArray(hop_size, ctx, sizeof(IdType) * 8);
  IdArray picked_idx = NewIdArray(hop_size, ctx, sizeof(IdType) * 8);
  FloatArray picked_imp =
      importance_sampling || weighted
          ? NewFloatArray(hop_size, ctx, sizeof(FloatType) * 8)
          : NullArray();

  IdType* const picked_row_data = picked_row.Ptr<IdType>();
  IdType* const picked_col_data = picked_col.Ptr<IdType>();
  IdType* const picked_idx_data = picked_idx.Ptr<IdType>();
  FloatType* const picked_imp_data = picked_imp.Ptr<FloatType>();

  auto picked_inrow = allocator.alloc_unique<IdType>(
      importance_sampling || weighted ? hop_size : 1);

  // Sample edges here
  IdType num_edges;
  {
    thrust::constant_iterator<FloatType> one(1);
    if (importance_sampling) {
      auto output = thrust::make_zip_iterator(
          picked_inrow.get(), picked_row_data, picked_col_data, picked_idx_data,
          picked_imp_data);
      if (weighted) {
        auto transformed_output = thrust::make_transform_output_iterator(
            output,
            TransformOpImp<
                IdType, FloatType, FloatType*, FloatType*, decltype(one)>{
                probs_found.get(), A, one, idx_coo, rows, cs, indptr.get(),
                subindptr, indices, data, is_pinned});
        auto stencil =
            thrust::make_zip_iterator(idx_coo, probs_found.get(), rands.get());
        num_edges =
            thrust::copy_if(
                exec_policy, iota, iota + hop_size, stencil, transformed_output,
                thrust::make_zip_function(StencilOp<FloatType>{cs})) -
            transformed_output;
      } else {
        auto transformed_output = thrust::make_transform_output_iterator(
            output,
            TransformOpImp<
                IdType, FloatType, FloatType*, decltype(one), decltype(one)>{
                probs_found.get(), one, one, idx_coo, rows, cs, indptr.get(),
                subindptr, indices, data, is_pinned});
        auto stencil =
            thrust::make_zip_iterator(idx_coo, probs_found.get(), rands.get());
        num_edges =
            thrust::copy_if(
                exec_policy, iota, iota + hop_size, stencil, transformed_output,
                thrust::make_zip_function(StencilOp<FloatType>{cs})) -
            transformed_output;
      }
    } else {
      if (weighted) {
        auto output = thrust::make_zip_iterator(
            picked_inrow.get(), picked_row_data, picked_col_data,
            picked_idx_data, picked_imp_data);
        auto transformed_output = thrust::make_transform_output_iterator(
            output,
            TransformOpImp<
                IdType, FloatType, decltype(one), FloatType*, FloatType*>{
                one, A, A, idx_coo, rows, cs, indptr.get(), subindptr, indices,
                data, is_pinned});
        const auto pred =
            StencilOpFused<IdType, FloatType, decltype(one), FloatType*>{
                random_seed, idx_coo,      cs,      one,  A,
                subindptr,   indptr.get(), indices, nids, is_pinned};
        num_edges = thrust::copy_if(
                        exec_policy, iota, iota + hop_size, iota,
                        transformed_output, pred) -
                    transformed_output;
      } else {
        auto output = thrust::make_zip_iterator(
            picked_row_data, picked_col_data, picked_idx_data);
        auto transformed_output = thrust::make_transform_output_iterator(
            output, TransformOp<IdType>{
                        idx_coo, rows, indptr.get(), subindptr, indices, data,
                        is_pinned});
        const auto pred =
            StencilOpFused<IdType, FloatType, decltype(one), decltype(one)>{
                random_seed, idx_coo,      cs,      one,  one,
                subindptr,   indptr.get(), indices, nids, is_pinned};
        num_edges = thrust::copy_if(
                        exec_policy, iota, iota + hop_size, iota,
                        transformed_output, pred) -
                    transformed_output;
      }
    }
  }

  // Normalize edge weights here
  if (importance_sampling || weighted) {
    thrust::constant_iterator<IdType> one(1);
    // contains degree information
    auto ds = allocator.alloc_unique<IdType>(num_rows);
    // contains sum of edge weights
    auto ws = allocator.alloc_unique<FloatType>(num_rows);
    // contains degree information only for vertices with nonzero degree
    auto ds_2 = allocator.alloc_unique<IdType>(num_rows);
    // contains sum of edge weights only for vertices with nonzero degree
    auto ws_2 = allocator.alloc_unique<FloatType>(num_rows);
    auto output_ = thrust::make_zip_iterator(ds.get(), ws.get());
    // contains row ids only for vertices with nonzero degree
    auto keys = allocator.alloc_unique<IdType>(num_rows);
    auto input = thrust::make_zip_iterator(one, picked_imp_data);
    auto new_end = thrust::reduce_by_key(
        exec_policy, picked_inrow.get(), picked_inrow.get() + num_edges, input,
        keys.get(), output_, thrust::equal_to<IdType>{}, TupleSum{});
    {
      thrust::constant_iterator<IdType> zero_int(0);
      thrust::constant_iterator<FloatType> zero_float(0);
      auto input = thrust::make_zip_iterator(zero_int, zero_float);
      auto output = thrust::make_zip_iterator(ds_2.get(), ws_2.get());
      thrust::copy(exec_policy, input, input + num_rows, output);
      {
        const auto num_rows_2 = new_end.first - keys.get();
        thrust::scatter(
            exec_policy, output_, output_ + num_rows_2, keys.get(), output);
      }
    }
    {
      auto input =
          thrust::make_zip_iterator(picked_inrow.get(), picked_imp_data);
      auto transformed_input = thrust::make_transform_iterator(
          input, thrust::make_zip_function(TransformOpMean<IdType, FloatType>{
                     ds_2.get(), ws_2.get()}));
      thrust::copy(
          exec_policy, transformed_input, transformed_input + num_edges,
          picked_imp_data);
    }
  }

  picked_row = picked_row.CreateView({num_edges}, picked_row->dtype);
  picked_col = picked_col.CreateView({num_edges}, picked_col->dtype);
  picked_idx = picked_idx.CreateView({num_edges}, picked_idx->dtype);
  if (importance_sampling || weighted)
    picked_imp = picked_imp.CreateView({num_edges}, picked_imp->dtype);

  return std::make_pair(
      COOMatrix(mat.num_rows, mat.num_cols, picked_row, picked_col, picked_idx),
      picked_imp);
}

template std::pair<COOMatrix, FloatArray>
CSRLaborSampling<kDGLCUDA, int32_t, float>(
    CSRMatrix, IdArray, int64_t, FloatArray, int, IdArray, float, IdArray);
template std::pair<COOMatrix, FloatArray>
CSRLaborSampling<kDGLCUDA, int64_t, float>(
    CSRMatrix, IdArray, int64_t, FloatArray, int, IdArray, float, IdArray);
template std::pair<COOMatrix, FloatArray>
CSRLaborSampling<kDGLCUDA, int32_t, double>(
    CSRMatrix, IdArray, int64_t, FloatArray, int, IdArray, float, IdArray);
template std::pair<COOMatrix, FloatArray>
CSRLaborSampling<kDGLCUDA, int64_t, double>(
    CSRMatrix, IdArray, int64_t, FloatArray, int, IdArray, float, IdArray);

}  // namespace impl
}  // namespace aten
}  // namespace dgl
