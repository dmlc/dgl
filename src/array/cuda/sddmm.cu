#include "hip/hip_runtime.h"
/**
 *  Copyright (c) 2020 by Contributors
 * @file array/cuda/sddmm.cu
 * @brief SDDMM C APIs and definitions.
 */
#include <dgl/array.h>

#include "./functor.cuh"
#include "./sddmm.cuh"

namespace dgl {
namespace aten {

/**
 * @brief CUDA implementation of g-SDDMM on Csr format.
 */
template <int XPU, typename IdType, typename DType>
void SDDMMCsr(
    const std::string& op, const BcastOff& bcast, const CSRMatrix& csr,
    NDArray lhs, NDArray rhs, NDArray out, int lhs_target, int rhs_target) {
  SWITCH_OP(op, Op, {
    SWITCH_TARGET(lhs_target, rhs_target, LhsTarget, RhsTarget, {
      cuda::SDDMMCsr<IdType, DType, Op, LhsTarget, RhsTarget>(
          bcast, csr, lhs, rhs, out);
    });
  });
}

/**
 * @brief CUDA implementation of g-SDDMM on Coo format.
 */
template <int XPU, typename IdType, typename DType>
void SDDMMCoo(
    const std::string& op, const BcastOff& bcast, const COOMatrix& coo,
    NDArray lhs, NDArray rhs, NDArray out, int lhs_target, int rhs_target) {
  SWITCH_OP(op, Op, {
    SWITCH_TARGET(lhs_target, rhs_target, LhsTarget, RhsTarget, {
      cuda::SDDMMCoo<IdType, DType, Op, LhsTarget, RhsTarget>(
          bcast, coo, lhs, rhs, out);
    });
  });
}

template void SDDMMCsr<kDGLCUDA, int32_t, __half>(
    const std::string& op, const BcastOff& bcast, const CSRMatrix& csr,
    NDArray lhs, NDArray rhs, NDArray out, int lhs_target, int rhs_target);
template void SDDMMCsr<kDGLCUDA, int64_t, __half>(
    const std::string& op, const BcastOff& bcast, const CSRMatrix& csr,
    NDArray lhs, NDArray rhs, NDArray out, int lhs_target, int rhs_target);
template void SDDMMCsr<kDGLCUDA, int32_t, __hip_bfloat16>(
    const std::string& op, const BcastOff& bcast, const CSRMatrix& csr,
    NDArray lhs, NDArray rhs, NDArray out, int lhs_target, int rhs_target);
template void SDDMMCsr<kDGLCUDA, int64_t, __hip_bfloat16>(
    const std::string& op, const BcastOff& bcast, const CSRMatrix& csr,
    NDArray lhs, NDArray rhs, NDArray out, int lhs_target, int rhs_target);
template void SDDMMCsr<kDGLCUDA, int32_t, float>(
    const std::string& op, const BcastOff& bcast, const CSRMatrix& csr,
    NDArray lhs, NDArray rhs, NDArray out, int lhs_target, int rhs_target);
template void SDDMMCsr<kDGLCUDA, int64_t, float>(
    const std::string& op, const BcastOff& bcast, const CSRMatrix& csr,
    NDArray lhs, NDArray rhs, NDArray out, int lhs_target, int rhs_target);
template void SDDMMCsr<kDGLCUDA, int32_t, double>(
    const std::string& op, const BcastOff& bcast, const CSRMatrix& csr,
    NDArray lhs, NDArray rhs, NDArray out, int lhs_target, int rhs_target);
template void SDDMMCsr<kDGLCUDA, int64_t, double>(
    const std::string& op, const BcastOff& bcast, const CSRMatrix& csr,
    NDArray lhs, NDArray rhs, NDArray out, int lhs_target, int rhs_target);

template void SDDMMCoo<kDGLCUDA, int32_t, __half>(
    const std::string& op, const BcastOff& bcast, const COOMatrix& coo,
    NDArray lhs, NDArray rhs, NDArray out, int lhs_target, int rhs_target);
template void SDDMMCoo<kDGLCUDA, int64_t, __half>(
    const std::string& op, const BcastOff& bcast, const COOMatrix& coo,
    NDArray lhs, NDArray rhs, NDArray out, int lhs_target, int rhs_target);
template void SDDMMCoo<kDGLCUDA, int32_t, __hip_bfloat16>(
    const std::string& op, const BcastOff& bcast, const COOMatrix& coo,
    NDArray lhs, NDArray rhs, NDArray out, int lhs_target, int rhs_target);
template void SDDMMCoo<kDGLCUDA, int64_t, __hip_bfloat16>(
    const std::string& op, const BcastOff& bcast, const COOMatrix& coo,
    NDArray lhs, NDArray rhs, NDArray out, int lhs_target, int rhs_target);
template void SDDMMCoo<kDGLCUDA, int32_t, float>(
    const std::string& op, const BcastOff& bcast, const COOMatrix& coo,
    NDArray lhs, NDArray rhs, NDArray out, int lhs_target, int rhs_target);
template void SDDMMCoo<kDGLCUDA, int64_t, float>(
    const std::string& op, const BcastOff& bcast, const COOMatrix& coo,
    NDArray lhs, NDArray rhs, NDArray out, int lhs_target, int rhs_target);
template void SDDMMCoo<kDGLCUDA, int32_t, double>(
    const std::string& op, const BcastOff& bcast, const COOMatrix& coo,
    NDArray lhs, NDArray rhs, NDArray out, int lhs_target, int rhs_target);
template void SDDMMCoo<kDGLCUDA, int64_t, double>(
    const std::string& op, const BcastOff& bcast, const COOMatrix& coo,
    NDArray lhs, NDArray rhs, NDArray out, int lhs_target, int rhs_target);

}  // namespace aten
}  // namespace dgl
