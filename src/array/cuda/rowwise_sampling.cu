#include "hip/hip_runtime.h"
/**
 *  Copyright (c) 2021 by Contributors
 * @file array/cuda/rowwise_sampling.cu
 * @brief uniform rowwise sampling
 */

#include <hiprand/hiprand_kernel.h>
#include <dgl/random.h>
#include <dgl/runtime/device_api.h>
#include <dgl/runtime/tensordispatch.h>

#include <hipcub/hipcub.hpp>
#include <numeric>

#include "../../array/cuda/atomic.cuh"
#include "../../runtime/cuda/cuda_common.h"
#include "./utils.h"

using namespace dgl::cuda;
using namespace dgl::aten::cuda;
using TensorDispatcher = dgl::runtime::TensorDispatcher;

namespace dgl {
namespace aten {
namespace impl {

namespace {

constexpr int BLOCK_SIZE = 128;

/**
 * @brief Compute the size of each row in the sampled CSR, without replacement.
 *
 * @tparam IdType The type of node and edge indexes.
 * @param num_picks The number of non-zero entries to pick per row.
 * @param num_rows The number of rows to pick.
 * @param in_rows The set of rows to pick.
 * @param in_ptr The index where each row's edges start.
 * @param out_deg The size of each row in the sampled matrix, as indexed by
 * `in_rows` (output).
 */
template <typename IdType>
__global__ void _CSRRowWiseSampleDegreeKernel(
    const int64_t num_picks, const int64_t num_rows,
    const IdType* const in_rows, const IdType* const in_ptr,
    IdType* const out_deg) {
  const int tIdx = threadIdx.x + blockIdx.x * blockDim.x;

  if (tIdx < num_rows) {
    const int in_row = in_rows[tIdx];
    const int out_row = tIdx;
    out_deg[out_row] = min(
        static_cast<IdType>(num_picks), in_ptr[in_row + 1] - in_ptr[in_row]);

    if (out_row == num_rows - 1) {
      // make the prefixsum work
      out_deg[num_rows] = 0;
    }
  }
}

/**
 * @brief Compute the size of each row in the sampled CSR, with replacement.
 *
 * @tparam IdType The type of node and edge indexes.
 * @param num_picks The number of non-zero entries to pick per row.
 * @param num_rows The number of rows to pick.
 * @param in_rows The set of rows to pick.
 * @param in_ptr The index where each row's edges start.
 * @param out_deg The size of each row in the sampled matrix, as indexed by
 * `in_rows` (output).
 */
template <typename IdType>
__global__ void _CSRRowWiseSampleDegreeReplaceKernel(
    const int64_t num_picks, const int64_t num_rows,
    const IdType* const in_rows, const IdType* const in_ptr,
    IdType* const out_deg) {
  const int tIdx = threadIdx.x + blockIdx.x * blockDim.x;

  if (tIdx < num_rows) {
    const int64_t in_row = in_rows[tIdx];
    const int64_t out_row = tIdx;

    if (in_ptr[in_row + 1] - in_ptr[in_row] == 0) {
      out_deg[out_row] = 0;
    } else {
      out_deg[out_row] = static_cast<IdType>(num_picks);
    }

    if (out_row == num_rows - 1) {
      // make the prefixsum work
      out_deg[num_rows] = 0;
    }
  }
}

/**
 * @brief Perform row-wise uniform sampling on a CSR matrix,
 * and generate a COO matrix, without replacement.
 *
 * @tparam IdType The ID type used for matrices.
 * @tparam TILE_SIZE The number of rows covered by each threadblock.
 * @param rand_seed The random seed to use.
 * @param num_picks The number of non-zeros to pick per row.
 * @param num_rows The number of rows to pick.
 * @param in_rows The set of rows to pick.
 * @param in_ptr The indptr array of the input CSR.
 * @param in_index The indices array of the input CSR.
 * @param data The data array of the input CSR.
 * @param out_ptr The offset to write each row to in the output COO.
 * @param out_rows The rows of the output COO (output).
 * @param out_cols The columns of the output COO (output).
 * @param out_idxs The data array of the output COO (output).
 */
template <typename IdType, int TILE_SIZE>
__global__ void _CSRRowWiseSampleUniformKernel(
    const uint64_t rand_seed, const int64_t num_picks, const int64_t num_rows,
    const IdType* const in_rows, const IdType* const in_ptr,
    const IdType* const in_index, const IdType* const data,
    const IdType* const out_ptr, IdType* const out_rows, IdType* const out_cols,
    IdType* const out_idxs) {
  // we assign one warp per row
  assert(blockDim.x == BLOCK_SIZE);

  int64_t out_row = blockIdx.x * TILE_SIZE;
  const int64_t last_row =
      min(static_cast<int64_t>(blockIdx.x + 1) * TILE_SIZE, num_rows);

  hiprandStatePhilox4_32_10_t rng;
  hiprand_init(rand_seed * gridDim.x + blockIdx.x, threadIdx.x, 0, &rng);

  while (out_row < last_row) {
    const int64_t row = in_rows[out_row];
    const int64_t in_row_start = in_ptr[row];
    const int64_t deg = in_ptr[row + 1] - in_row_start;
    const int64_t out_row_start = out_ptr[out_row];

    if (deg <= num_picks) {
      // just copy row when there is not enough nodes to sample.
      for (int idx = threadIdx.x; idx < deg; idx += BLOCK_SIZE) {
        const IdType in_idx = in_row_start + idx;
        out_rows[out_row_start + idx] = row;
        out_cols[out_row_start + idx] = in_index[in_idx];
        out_idxs[out_row_start + idx] = data ? data[in_idx] : in_idx;
      }
    } else {
      // generate permutation list via reservoir algorithm
      for (int idx = threadIdx.x; idx < num_picks; idx += BLOCK_SIZE) {
        out_idxs[out_row_start + idx] = idx;
      }
      __syncthreads();

      for (int idx = num_picks + threadIdx.x; idx < deg; idx += BLOCK_SIZE) {
        const int num = hiprand(&rng) % (idx + 1);
        if (num < num_picks) {
          // use max so as to achieve the replacement order the serial
          // algorithm would have
          AtomicMax(out_idxs + out_row_start + num, idx);
        }
      }
      __syncthreads();

      // copy permutation over
      for (int idx = threadIdx.x; idx < num_picks; idx += BLOCK_SIZE) {
        const IdType perm_idx = out_idxs[out_row_start + idx] + in_row_start;
        out_rows[out_row_start + idx] = row;
        out_cols[out_row_start + idx] = in_index[perm_idx];
        out_idxs[out_row_start + idx] = data ? data[perm_idx] : perm_idx;
      }
    }
    out_row += 1;
  }
}

/**
 * @brief Perform row-wise uniform sampling on a CSR matrix,
 * and generate a COO matrix, with replacement.
 *
 * @tparam IdType The ID type used for matrices.
 * @tparam TILE_SIZE The number of rows covered by each threadblock.
 * @param rand_seed The random seed to use.
 * @param num_picks The number of non-zeros to pick per row.
 * @param num_rows The number of rows to pick.
 * @param in_rows The set of rows to pick.
 * @param in_ptr The indptr array of the input CSR.
 * @param in_index The indices array of the input CSR.
 * @param data The data array of the input CSR.
 * @param out_ptr The offset to write each row to in the output COO.
 * @param out_rows The rows of the output COO (output).
 * @param out_cols The columns of the output COO (output).
 * @param out_idxs The data array of the output COO (output).
 */
template <typename IdType, int TILE_SIZE>
__global__ void _CSRRowWiseSampleUniformReplaceKernel(
    const uint64_t rand_seed, const int64_t num_picks, const int64_t num_rows,
    const IdType* const in_rows, const IdType* const in_ptr,
    const IdType* const in_index, const IdType* const data,
    const IdType* const out_ptr, IdType* const out_rows, IdType* const out_cols,
    IdType* const out_idxs) {
  // we assign one warp per row
  assert(blockDim.x == BLOCK_SIZE);

  int64_t out_row = blockIdx.x * TILE_SIZE;
  const int64_t last_row =
      min(static_cast<int64_t>(blockIdx.x + 1) * TILE_SIZE, num_rows);

  hiprandStatePhilox4_32_10_t rng;
  hiprand_init(rand_seed * gridDim.x + blockIdx.x, threadIdx.x, 0, &rng);

  while (out_row < last_row) {
    const int64_t row = in_rows[out_row];
    const int64_t in_row_start = in_ptr[row];
    const int64_t out_row_start = out_ptr[out_row];
    const int64_t deg = in_ptr[row + 1] - in_row_start;

    if (deg > 0) {
      // each thread then blindly copies in rows only if deg > 0.
      for (int idx = threadIdx.x; idx < num_picks; idx += BLOCK_SIZE) {
        const int64_t edge = hiprand(&rng) % deg;
        const int64_t out_idx = out_row_start + idx;
        out_rows[out_idx] = row;
        out_cols[out_idx] = in_index[in_row_start + edge];
        out_idxs[out_idx] =
            data ? data[in_row_start + edge] : in_row_start + edge;
      }
    }
    out_row += 1;
  }
}

}  // namespace

///////////////////////////// CSR sampling //////////////////////////

template <DGLDeviceType XPU, typename IdType>
COOMatrix _CSRRowWiseSamplingUniform(
    CSRMatrix mat, IdArray rows, const int64_t num_picks, const bool replace) {
  const auto& ctx = rows->ctx;
  auto device = runtime::DeviceAPI::Get(ctx);
  hipStream_t stream = runtime::getCurrentCUDAStream();

  const int64_t num_rows = rows->shape[0];
  const IdType* const slice_rows = static_cast<const IdType*>(rows->data);

  IdArray picked_row =
      NewIdArray(num_rows * num_picks, ctx, sizeof(IdType) * 8);
  IdArray picked_col =
      NewIdArray(num_rows * num_picks, ctx, sizeof(IdType) * 8);
  IdArray picked_idx =
      NewIdArray(num_rows * num_picks, ctx, sizeof(IdType) * 8);
  IdType* const out_rows = static_cast<IdType*>(picked_row->data);
  IdType* const out_cols = static_cast<IdType*>(picked_col->data);
  IdType* const out_idxs = static_cast<IdType*>(picked_idx->data);

  const IdType* in_ptr = static_cast<IdType*>(GetDevicePointer(mat.indptr));
  const IdType* in_cols = static_cast<IdType*>(GetDevicePointer(mat.indices));
  const IdType* data = CSRHasData(mat)
                           ? static_cast<IdType*>(GetDevicePointer(mat.data))
                           : nullptr;

  // compute degree
  IdType* out_deg = static_cast<IdType*>(
      device->AllocWorkspace(ctx, (num_rows + 1) * sizeof(IdType)));
  if (replace) {
    const dim3 block(512);
    const dim3 grid((num_rows + block.x - 1) / block.x);
    CUDA_KERNEL_CALL(
        _CSRRowWiseSampleDegreeReplaceKernel, grid, block, 0, stream, num_picks,
        num_rows, slice_rows, in_ptr, out_deg);
  } else {
    const dim3 block(512);
    const dim3 grid((num_rows + block.x - 1) / block.x);
    CUDA_KERNEL_CALL(
        _CSRRowWiseSampleDegreeKernel, grid, block, 0, stream, num_picks,
        num_rows, slice_rows, in_ptr, out_deg);
  }

  // fill out_ptr
  IdType* out_ptr = static_cast<IdType*>(
      device->AllocWorkspace(ctx, (num_rows + 1) * sizeof(IdType)));
  size_t prefix_temp_size = 0;
  CUDA_CALL(hipcub::DeviceScan::ExclusiveSum(
      nullptr, prefix_temp_size, out_deg, out_ptr, num_rows + 1, stream));
  void* prefix_temp = device->AllocWorkspace(ctx, prefix_temp_size);
  CUDA_CALL(hipcub::DeviceScan::ExclusiveSum(
      prefix_temp, prefix_temp_size, out_deg, out_ptr, num_rows + 1, stream));
  device->FreeWorkspace(ctx, prefix_temp);
  device->FreeWorkspace(ctx, out_deg);

  hipEvent_t copyEvent;
  CUDA_CALL(hipEventCreate(&copyEvent));

  NDArray new_len_tensor;
  if (TensorDispatcher::Global()->IsAvailable()) {
    new_len_tensor = NDArray::PinnedEmpty(
        {1}, DGLDataTypeTraits<IdType>::dtype, DGLContext{kDGLCPU, 0});
  } else {
    // use pageable memory, it will unecessarily block but be functional
    new_len_tensor = NDArray::Empty(
        {1}, DGLDataTypeTraits<IdType>::dtype, DGLContext{kDGLCPU, 0});
  }

  // copy using the internal current stream
  CUDA_CALL(hipMemcpyAsync(
      new_len_tensor->data, out_ptr + num_rows, sizeof(IdType),
      hipMemcpyDeviceToHost, stream));
  CUDA_CALL(hipEventRecord(copyEvent, stream));

  const uint64_t random_seed = RandomEngine::ThreadLocal()->RandInt(1000000000);

  // select edges
  // the number of rows each thread block will cover
  constexpr int TILE_SIZE = 128 / BLOCK_SIZE;
  if (replace) {  // with replacement
    const dim3 block(BLOCK_SIZE);
    const dim3 grid((num_rows + TILE_SIZE - 1) / TILE_SIZE);
    CUDA_KERNEL_CALL(
        (_CSRRowWiseSampleUniformReplaceKernel<IdType, TILE_SIZE>), grid, block,
        0, stream, random_seed, num_picks, num_rows, slice_rows, in_ptr,
        in_cols, data, out_ptr, out_rows, out_cols, out_idxs);
  } else {  // without replacement
    const dim3 block(BLOCK_SIZE);
    const dim3 grid((num_rows + TILE_SIZE - 1) / TILE_SIZE);
    CUDA_KERNEL_CALL(
        (_CSRRowWiseSampleUniformKernel<IdType, TILE_SIZE>), grid, block, 0,
        stream, random_seed, num_picks, num_rows, slice_rows, in_ptr, in_cols,
        data, out_ptr, out_rows, out_cols, out_idxs);
  }
  device->FreeWorkspace(ctx, out_ptr);

  // wait for copying `new_len` to finish
  CUDA_CALL(hipEventSynchronize(copyEvent));
  CUDA_CALL(hipEventDestroy(copyEvent));

  const IdType new_len = static_cast<const IdType*>(new_len_tensor->data)[0];
  picked_row = picked_row.CreateView({new_len}, picked_row->dtype);
  picked_col = picked_col.CreateView({new_len}, picked_col->dtype);
  picked_idx = picked_idx.CreateView({new_len}, picked_idx->dtype);

  return COOMatrix(
      mat.num_rows, mat.num_cols, picked_row, picked_col, picked_idx);
}

template <DGLDeviceType XPU, typename IdType>
COOMatrix CSRRowWiseSamplingUniform(
    CSRMatrix mat, IdArray rows, const int64_t num_picks, const bool replace) {
  if (num_picks == -1) {
    // Basically this is UnitGraph::InEdges().
    COOMatrix coo = CSRToCOO(CSRSliceRows(mat, rows), false);
    IdArray sliced_rows = IndexSelect(rows, coo.row);
    return COOMatrix(
        mat.num_rows, mat.num_cols, sliced_rows, coo.col, coo.data);
  } else {
    return _CSRRowWiseSamplingUniform<XPU, IdType>(
        mat, rows, num_picks, replace);
  }
}

template COOMatrix CSRRowWiseSamplingUniform<kDGLCUDA, int32_t>(
    CSRMatrix, IdArray, int64_t, bool);
template COOMatrix CSRRowWiseSamplingUniform<kDGLCUDA, int64_t>(
    CSRMatrix, IdArray, int64_t, bool);

}  // namespace impl
}  // namespace aten
}  // namespace dgl
