#include "hip/hip_runtime.h"
/**
 *  Copyright (c) 2020 by Contributors
 * @file array/cuda/coo2csr.cc
 * @brief COO2CSR
 */
#include <dgl/array.h>

#include "../../runtime/cuda/cuda_common.h"
#include "./utils.h"

namespace dgl {

using runtime::NDArray;

namespace aten {
namespace impl {

template <DGLDeviceType XPU, typename IdType>
CSRMatrix COOToCSR(COOMatrix coo) {
  LOG(FATAL) << "Unreachable code.";
  return {};
}

template <>
CSRMatrix COOToCSR<kDGLCUDA, int32_t>(COOMatrix coo) {
  auto* thr_entry = runtime::CUDAThreadEntry::ThreadLocal();
  hipStream_t stream = runtime::getCurrentCUDAStream();
  // allocate cusparse handle if needed
  if (!thr_entry->cusparse_handle) {
    CUSPARSE_CALL(hipsparseCreate(&(thr_entry->cusparse_handle)));
  }
  CUSPARSE_CALL(hipsparseSetStream(thr_entry->cusparse_handle, stream));

  bool row_sorted = coo.row_sorted;
  bool col_sorted = coo.col_sorted;
  if (!row_sorted) {
    // we only need to sort the rows to perform conversion
    coo = COOSort(coo, false);
    col_sorted = coo.col_sorted;
  }

  const int64_t nnz = coo.row->shape[0];
  CHECK_NO_OVERFLOW(coo.row->dtype, nnz);
  // TODO(minjie): Many of our current implementation assumes that CSR must have
  //   a data array. This is a temporary workaround. Remove this after:
  //   - The old immutable graph implementation is deprecated.
  //   - The old binary reduce kernel is deprecated.
  if (!COOHasData(coo))
    coo.data = aten::Range(0, nnz, coo.row->dtype.bits, coo.row->ctx);

  NDArray indptr =
      aten::NewIdArray(coo.num_rows + 1, coo.row->ctx, coo.row->dtype.bits);
  int32_t* indptr_ptr = static_cast<int32_t*>(indptr->data);
  CUSPARSE_CALL(hipsparseXcoo2csr(
      thr_entry->cusparse_handle, coo.row.Ptr<int32_t>(), nnz, coo.num_rows,
      indptr_ptr, HIPSPARSE_INDEX_BASE_ZERO));

  return CSRMatrix(
      coo.num_rows, coo.num_cols, indptr, coo.col, coo.data, col_sorted);
}

/**
 * @brief Search for the insertion positions for needle in the hay.
 *
 * The hay is a list of sorted elements and the result is the insertion position
 * of each needle so that the insertion still gives sorted order.
 *
 * It essentially perform binary search to find upper bound for each needle
 * elements.
 *
 * For example:
 * hay = [0, 0, 1, 2, 2]
 * needle = [0, 1, 2, 3]
 * then,
 * out = [2, 3, 5, 5]
 */
template <typename IdType>
__global__ void _SortedSearchKernelUpperBound(
    const IdType* hay, int64_t hay_size, const IdType* needles,
    int64_t num_needles, IdType* pos) {
  int tx = blockIdx.x * blockDim.x + threadIdx.x;
  const int stride_x = gridDim.x * blockDim.x;
  while (tx < num_needles) {
    const IdType ele = needles[tx];
    // binary search
    IdType lo = 0, hi = hay_size;
    while (lo < hi) {
      IdType mid = (lo + hi) >> 1;
      if (hay[mid] <= ele) {
        lo = mid + 1;
      } else {
        hi = mid;
      }
    }
    pos[tx] = lo;
    tx += stride_x;
  }
}

template <>
CSRMatrix COOToCSR<kDGLCUDA, int64_t>(COOMatrix coo) {
  const auto& ctx = coo.row->ctx;
  const auto nbits = coo.row->dtype.bits;
  hipStream_t stream = runtime::getCurrentCUDAStream();
  bool row_sorted = coo.row_sorted;
  bool col_sorted = coo.col_sorted;
  if (!row_sorted) {
    coo = COOSort(coo, false);
    col_sorted = coo.col_sorted;
  }

  const int64_t nnz = coo.row->shape[0];
  // TODO(minjie): Many of our current implementation assumes that CSR must have
  //   a data array. This is a temporary workaround. Remove this after:
  //   - The old immutable graph implementation is deprecated.
  //   - The old binary reduce kernel is deprecated.
  if (!COOHasData(coo))
    coo.data = aten::Range(0, nnz, coo.row->dtype.bits, coo.row->ctx);

  IdArray rowids = Range(0, coo.num_rows, nbits, ctx);
  const int nt = cuda::FindNumThreads(coo.num_rows);
  const int nb = (coo.num_rows + nt - 1) / nt;
  IdArray indptr = Full(0, coo.num_rows + 1, nbits, ctx);
  CUDA_KERNEL_CALL(
      _SortedSearchKernelUpperBound, nb, nt, 0, stream, coo.row.Ptr<int64_t>(),
      nnz, rowids.Ptr<int64_t>(), coo.num_rows, indptr.Ptr<int64_t>() + 1);

  return CSRMatrix(
      coo.num_rows, coo.num_cols, indptr, coo.col, coo.data, col_sorted);
}

template CSRMatrix COOToCSR<kDGLCUDA, int32_t>(COOMatrix coo);
template CSRMatrix COOToCSR<kDGLCUDA, int64_t>(COOMatrix coo);

}  // namespace impl
}  // namespace aten
}  // namespace dgl
