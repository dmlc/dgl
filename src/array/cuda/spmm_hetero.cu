#include "hip/hip_runtime.h"
/**
 *  Copyright (c) 2020 by Contributors
 * @file array/cuda/spmm.cu
 * @brief SPMM C APIs and definitions.
 */
#include <dgl/array.h>

#include <cstdlib>

#include "../../runtime/cuda/cuda_common.h"
#include "./functor.cuh"
#include "./ge_spmm.cuh"
#include "./spmm.cuh"

namespace dgl {

using namespace cuda;

namespace aten {

/**
 * @brief CUDA implementation of g-SpMM on Csr format.
 * @note use cusparse if the reduce operator is `sum` and there is
 *       no broadcast, use dgl's kernel in other cases.
 */
template <int XPU, typename IdType, typename DType>
void SpMMCsrHetero(
    const std::string& op, const std::string& reduce, const BcastOff& bcast,
    const std::vector<CSRMatrix>& vec_csr,
    const std::vector<NDArray>& vec_ufeat,
    const std::vector<NDArray>& vec_efeat, std::vector<NDArray>* vec_out,
    std::vector<std::vector<NDArray>>* out_aux,
    const std::vector<dgl_type_t>& ufeat_ntids,  // ufeat node type id
    const std::vector<dgl_type_t>& out_ntids) {  // output node type id
  bool is_scalar_efeat =
      vec_efeat[0].NumElements() == vec_csr[0].indices->shape[0];
  bool use_efeat = op != "copy_lhs";
  auto device = runtime::DeviceAPI::Get(vec_csr[0].indptr->ctx);
  std::vector<DType*> trans_out((*vec_out).size(), NULL);
  bool use_deterministic_alg_only = false;
  if (NULL != std::getenv("USE_DETERMINISTIC_ALG"))
    use_deterministic_alg_only = true;

  bool use_legacy_cusparsemm =
      (CUDART_VERSION < 11000) && (reduce == "sum") &&
      // legacy cuSPARSE does not care about NNZ, hence the argument "false".
      ((op == "copy_lhs" && cusparse_available<DType, IdType>(false)) ||
       (op == "mul" && is_scalar_efeat &&
        cusparse_available<DType, IdType>(false)));
  // Create temporary output buffer to store non-transposed output
  if (use_legacy_cusparsemm) {
    for (dgl_type_t ntype = 0; ntype < (*vec_out).size(); ++ntype) {
      const int m = (*vec_out)[ntype]->shape[0];
      const int n = (*vec_out)[ntype]->shape[1];
      if (m == 0) continue;
      DType* out = static_cast<DType*>(device->AllocWorkspace(
          vec_csr[0].indptr->ctx, m * n * sizeof(DType)));
      CUDA_CALL(hipMemset(out, 0, m * n * sizeof(DType)));
      trans_out[ntype] = out;
    }
  }
  // Check shape of ufeat for all relation type and compute feature size
  int64_t x_length = 1;
  for (dgl_type_t etype = 0; etype < (ufeat_ntids.size() - 1); ++etype) {
    NDArray ufeat = vec_ufeat[ufeat_ntids[etype]];
    NDArray next_ufeat = vec_ufeat[ufeat_ntids[etype + 1]];
    CHECK_EQ(ufeat->ndim, next_ufeat->ndim)
        << "Input features have different shapes";
    for (int i = 1; i < ufeat->ndim; ++i) {
      if (ufeat->shape[i] != next_ufeat->shape[i]) {
        if (ufeat->shape[i] == 1 || next_ufeat->shape[i] == 1)
          LOG(FATAL) << "Homogenized message passing on heterogeneous graphs "
                        "does not support "
                     << "automatic broadcasting.  Please manually broadcast it "
                        "before calling "
                     << "message passing functions.";
        else
          LOG(FATAL) << "Input features have different shapes.";
        return;
      }

      if (etype == 0) x_length *= ufeat->shape[i];
    }
  }
  // TODO(Israt): Can python do the following initializations while creating the
  // tensors?
  if (reduce == "max" || reduce == "min") {
    const int64_t dim = bcast.out_len;
    std::vector<bool> updated((*vec_out).size(), false);
    for (dgl_type_t etype = 0; etype < ufeat_ntids.size(); ++etype) {
      DType* out_off = (*vec_out)[out_ntids[etype]].Ptr<DType>();
      if (reduce == "max")
        _Fill(
            out_off, vec_csr[etype].num_rows * dim,
            cuda::reduce::Max<IdType, DType>::zero());
      else  // min
        _Fill(
            out_off, vec_csr[etype].num_rows * dim,
            cuda::reduce::Min<IdType, DType>::zero());
      const dgl_type_t dst_id = out_ntids[etype];
      if (!updated[dst_id]) {
        updated[dst_id] = true;
        if (op == "copy_lhs") {
          IdType* argu_ntype = (*out_aux)[2][dst_id].Ptr<IdType>();
          _Fill(
              argu_ntype, vec_csr[etype].num_rows * dim,
              static_cast<IdType>(-1));
        }
        if (op == "copy_rhs") {
          IdType* arge_etype = (*out_aux)[3][dst_id].Ptr<IdType>();
          _Fill(
              arge_etype, vec_csr[etype].num_rows * dim,
              static_cast<IdType>(-1));
        }
      }
    }
  }

  hipStream_t stream = runtime::getCurrentCUDAStream();
  for (dgl_type_t etype = 0; etype < ufeat_ntids.size(); ++etype) {
    const dgl_type_t src_id = ufeat_ntids[etype];
    const dgl_type_t dst_id = out_ntids[etype];
    CSRMatrix csr = vec_csr[etype];
    if (reduce == "sum") {
      bool more_nnz = (csr.indices->shape[0] > csr.num_rows * csr.num_cols);
      /* Call  SpMM for each relation type */
      if (op == "copy_lhs" &&
          cusparse_available<DType, IdType>(more_nnz)) {  // cusparse
        /* If CUDA is less than 11.0, put the output in trans_out for later
         * transposition */
        DType* out = (CUDART_VERSION < 11000)
                         ? trans_out[dst_id]
                         : static_cast<DType*>((*vec_out)[dst_id]->data);
        CusparseCsrmm2Hetero<DType, IdType>(
            csr.indptr->ctx, csr, static_cast<DType*>(vec_ufeat[src_id]->data),
            nullptr, out, x_length, stream, use_deterministic_alg_only);
      } else if (
          op == "mul" && is_scalar_efeat &&
          cusparse_available<DType, IdType>(more_nnz)) {  // cusparse
        NDArray efeat = vec_efeat[etype];
        if (!IsNullArray(csr.data)) efeat = IndexSelect(efeat, csr.data);
        CusparseCsrmm2Hetero<DType, IdType>(
            csr.indptr->ctx, csr, static_cast<DType*>(vec_ufeat[src_id]->data),
            static_cast<DType*>(efeat->data),
            // TODO(Israt): Change (*vec_out) to trans_out to support CUDA
            // version < 11
            static_cast<DType*>((*vec_out)[dst_id]->data), x_length, stream,
            use_deterministic_alg_only);
      } else {  // general kernel
        NDArray ufeat =
            (vec_ufeat.size() == 0) ? NullArray() : vec_ufeat[src_id];
        NDArray efeat =
            (vec_efeat.size() == 0) ? NullArray() : vec_efeat[etype];
        SWITCH_OP(op, Op, {
          cuda::SpMMCsr<IdType, DType, Op, cuda::reduce::Sum<IdType, DType>>(
              bcast, csr, ufeat, efeat, (*vec_out)[dst_id], NullArray(),
              NullArray());
        });
      }
    } else if (reduce == "max") {
      SWITCH_OP(op, Op, {
        NDArray ufeat =
            (vec_ufeat.size() == 0) ? NullArray() : vec_ufeat[src_id];
        NDArray efeat =
            (vec_efeat.size() == 0) ? NullArray() : vec_efeat[etype];
        cuda::SpMMCmpCsrHetero<
            IdType, DType, Op, cuda::reduce::Max<IdType, DType>>(
            bcast, csr, ufeat, efeat, (*vec_out)[dst_id], (*out_aux)[0][dst_id],
            (*out_aux)[1][dst_id], (*out_aux)[2][dst_id], (*out_aux)[3][dst_id],
            src_id, etype);
      });
    } else if (reduce == "min") {
      SWITCH_OP(op, Op, {
        NDArray ufeat =
            (vec_ufeat.size() == 0) ? NullArray() : vec_ufeat[src_id];
        NDArray efeat =
            (vec_efeat.size() == 0) ? NullArray() : vec_efeat[etype];
        cuda::SpMMCmpCsrHetero<
            IdType, DType, Op, cuda::reduce::Min<IdType, DType>>(
            bcast, csr, ufeat, efeat, (*vec_out)[dst_id], (*out_aux)[0][dst_id],
            (*out_aux)[1][dst_id], (*out_aux)[2][dst_id], (*out_aux)[3][dst_id],
            src_id, etype);
      });
    } else {
      LOG(FATAL) << "Not implemented";
    }
  }

  if (use_legacy_cusparsemm) {
    // transpose output
    for (dgl_type_t ntype = 0; ntype < (*vec_out).size(); ++ntype) {
      const int m = (*vec_out)[ntype]->shape[0];
      const int n = (*vec_out)[ntype]->shape[1];
      if (m == 0) continue;
      DType* C_data = static_cast<DType*>((*vec_out)[ntype]->data);
      _Transpose(trans_out[ntype], C_data, n, m);
      device->FreeWorkspace(vec_csr[0].indptr->ctx, trans_out[ntype]);
    }
  }
}

template void SpMMCsrHetero<kDGLCUDA, int32_t, __half>(
    const std::string& op, const std::string& reduce, const BcastOff& bcast,
    const std::vector<CSRMatrix>& csr, const std::vector<NDArray>& ufeat,
    const std::vector<NDArray>& efeat, std::vector<NDArray>* out,
    std::vector<std::vector<NDArray>>* out_aux,
    const std::vector<dgl_type_t>& ufeat_ntids,
    const std::vector<dgl_type_t>& out_ntids);
template void SpMMCsrHetero<kDGLCUDA, int64_t, __half>(
    const std::string& op, const std::string& reduce, const BcastOff& bcast,
    const std::vector<CSRMatrix>& csr, const std::vector<NDArray>& ufeat,
    const std::vector<NDArray>& efeat, std::vector<NDArray>* out,
    std::vector<std::vector<NDArray>>* out_aux,
    const std::vector<dgl_type_t>& ufeat_ntids,
    const std::vector<dgl_type_t>& out_ntids);
#if BF16_ENABLED
template void SpMMCsrHetero<kDGLCUDA, int32_t, __hip_bfloat16>(
    const std::string& op, const std::string& reduce, const BcastOff& bcast,
    const std::vector<CSRMatrix>& csr, const std::vector<NDArray>& ufeat,
    const std::vector<NDArray>& efeat, std::vector<NDArray>* out,
    std::vector<std::vector<NDArray>>* out_aux,
    const std::vector<dgl_type_t>& ufeat_ntids,
    const std::vector<dgl_type_t>& out_ntids);
template void SpMMCsrHetero<kDGLCUDA, int64_t, __hip_bfloat16>(
    const std::string& op, const std::string& reduce, const BcastOff& bcast,
    const std::vector<CSRMatrix>& csr, const std::vector<NDArray>& ufeat,
    const std::vector<NDArray>& efeat, std::vector<NDArray>* out,
    std::vector<std::vector<NDArray>>* out_aux,
    const std::vector<dgl_type_t>& ufeat_ntids,
    const std::vector<dgl_type_t>& out_ntids);
#endif  // BF16_ENABLED
template void SpMMCsrHetero<kDGLCUDA, int32_t, float>(
    const std::string& op, const std::string& reduce, const BcastOff& bcast,
    const std::vector<CSRMatrix>& csr, const std::vector<NDArray>& ufeat,
    const std::vector<NDArray>& efeat, std::vector<NDArray>* out,
    std::vector<std::vector<NDArray>>* out_aux,
    const std::vector<dgl_type_t>& ufeat_ntids,
    const std::vector<dgl_type_t>& out_ntids);
template void SpMMCsrHetero<kDGLCUDA, int64_t, float>(
    const std::string& op, const std::string& reduce, const BcastOff& bcast,
    const std::vector<CSRMatrix>& csr, const std::vector<NDArray>& ufeat,
    const std::vector<NDArray>& efeat, std::vector<NDArray>* out,
    std::vector<std::vector<NDArray>>* out_aux,
    const std::vector<dgl_type_t>& ufeat_ntids,
    const std::vector<dgl_type_t>& out_ntids);
template void SpMMCsrHetero<kDGLCUDA, int32_t, double>(
    const std::string& op, const std::string& reduce, const BcastOff& bcast,
    const std::vector<CSRMatrix>& csr, const std::vector<NDArray>& ufeat,
    const std::vector<NDArray>& efeat, std::vector<NDArray>* out,
    std::vector<std::vector<NDArray>>* out_aux,
    const std::vector<dgl_type_t>& ufeat_ntids,
    const std::vector<dgl_type_t>& out_ntids);
template void SpMMCsrHetero<kDGLCUDA, int64_t, double>(
    const std::string& op, const std::string& reduce, const BcastOff& bcast,
    const std::vector<CSRMatrix>& csr, const std::vector<NDArray>& ufeat,
    const std::vector<NDArray>& efeat, std::vector<NDArray>* out,
    std::vector<std::vector<NDArray>>* out_aux,
    const std::vector<dgl_type_t>& ufeat_ntids,
    const std::vector<dgl_type_t>& out_ntids);

}  // namespace aten
}  // namespace dgl
