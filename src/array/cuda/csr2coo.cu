#include "hip/hip_runtime.h"
/**
 *  Copyright (c) 2020 by Contributors
 * @file array/cuda/csr2coo.cc
 * @brief CSR2COO
 */
#include <dgl/array.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/transform_iterator.h>

#include <hipcub/hipcub.hpp>

#include "../../runtime/cuda/cuda_common.h"
#include "./utils.h"

namespace dgl {

using runtime::NDArray;

namespace aten {
namespace impl {

template <DGLDeviceType XPU, typename IdType>
COOMatrix CSRToCOO(CSRMatrix csr) {
  LOG(FATAL) << "Unreachable codes";
  return {};
}

template <>
COOMatrix CSRToCOO<kDGLCUDA, int32_t>(CSRMatrix csr) {
  auto* thr_entry = runtime::CUDAThreadEntry::ThreadLocal();
  hipStream_t stream = runtime::getCurrentCUDAStream();
  // allocate cusparse handle if needed
  if (!thr_entry->cusparse_handle) {
    CUSPARSE_CALL(hipsparseCreate(&(thr_entry->cusparse_handle)));
  }
  CUSPARSE_CALL(hipsparseSetStream(thr_entry->cusparse_handle, stream));

  NDArray indptr = csr.indptr, indices = csr.indices, data = csr.data;
  const int32_t* indptr_ptr = static_cast<int32_t*>(indptr->data);
  NDArray row =
      aten::NewIdArray(indices->shape[0], indptr->ctx, indptr->dtype.bits);
  int32_t* row_ptr = static_cast<int32_t*>(row->data);

  CUSPARSE_CALL(hipsparseXcsr2coo(
      thr_entry->cusparse_handle, indptr_ptr, indices->shape[0], csr.num_rows,
      row_ptr, HIPSPARSE_INDEX_BASE_ZERO));

  return COOMatrix(
      csr.num_rows, csr.num_cols, row, indices, data, true, csr.sorted);
}

struct RepeatIndex {
  template <typename IdType>
  __host__ __device__ auto operator()(IdType i) {
    return thrust::make_constant_iterator(i);
  }
};

template <typename IdType>
struct OutputBufferIndexer {
  const IdType* indptr;
  IdType* buffer;
  __host__ __device__ auto operator()(IdType i) { return buffer + indptr[i]; }
};

template <typename IdType>
struct AdjacentDifference {
  const IdType* indptr;
  __host__ __device__ auto operator()(IdType i) {
    return indptr[i + 1] - indptr[i];
  }
};

template <>
COOMatrix CSRToCOO<kDGLCUDA, int64_t>(CSRMatrix csr) {
  const auto& ctx = csr.indptr->ctx;
  hipStream_t stream = runtime::getCurrentCUDAStream();

  const int64_t nnz = csr.indices->shape[0];
  const auto nbits = csr.indptr->dtype.bits;
  IdArray ret_row = NewIdArray(nnz, ctx, nbits);

  runtime::CUDAWorkspaceAllocator allocator(csr.indptr->ctx);
  thrust::counting_iterator<int64_t> iota(0);

  auto input_buffer = thrust::make_transform_iterator(iota, RepeatIndex{});
  auto output_buffer = thrust::make_transform_iterator(
      iota, OutputBufferIndexer<int64_t>{
                csr.indptr.Ptr<int64_t>(), ret_row.Ptr<int64_t>()});
  auto buffer_sizes = thrust::make_transform_iterator(
      iota, AdjacentDifference<int64_t>{csr.indptr.Ptr<int64_t>()});

  constexpr int64_t max_copy_at_once = std::numeric_limits<int32_t>::max();
  for (int64_t i = 0; i < csr.num_rows; i += max_copy_at_once) {
    std::size_t temp_storage_bytes = 0;
    CUDA_CALL(hipcub::DeviceCopy::Batched(
        nullptr, temp_storage_bytes, input_buffer + i, output_buffer + i,
        buffer_sizes + i, std::min(csr.num_rows - i, max_copy_at_once),
        stream));

    auto temp = allocator.alloc_unique<char>(temp_storage_bytes);

    CUDA_CALL(hipcub::DeviceCopy::Batched(
        temp.get(), temp_storage_bytes, input_buffer + i, output_buffer + i,
        buffer_sizes + i, std::min(csr.num_rows - i, max_copy_at_once),
        stream));
  }

  return COOMatrix(
      csr.num_rows, csr.num_cols, ret_row, csr.indices, csr.data, true,
      csr.sorted);
}

template COOMatrix CSRToCOO<kDGLCUDA, int32_t>(CSRMatrix csr);
template COOMatrix CSRToCOO<kDGLCUDA, int64_t>(CSRMatrix csr);

template <DGLDeviceType XPU, typename IdType>
COOMatrix CSRToCOODataAsOrder(CSRMatrix csr) {
  LOG(FATAL) << "Unreachable codes";
  return {};
}

template <>
COOMatrix CSRToCOODataAsOrder<kDGLCUDA, int32_t>(CSRMatrix csr) {
  COOMatrix coo = CSRToCOO<kDGLCUDA, int32_t>(csr);
  if (aten::IsNullArray(coo.data)) return coo;

  auto* thr_entry = runtime::CUDAThreadEntry::ThreadLocal();
  auto device = runtime::DeviceAPI::Get(coo.row->ctx);
  hipStream_t stream = runtime::getCurrentCUDAStream();
  // allocate cusparse handle if needed
  if (!thr_entry->cusparse_handle) {
    CUSPARSE_CALL(hipsparseCreate(&(thr_entry->cusparse_handle)));
  }
  CUSPARSE_CALL(hipsparseSetStream(thr_entry->cusparse_handle, stream));

  NDArray row = coo.row, col = coo.col, data = coo.data;
  int32_t* row_ptr = static_cast<int32_t*>(row->data);
  int32_t* col_ptr = static_cast<int32_t*>(col->data);
  int32_t* data_ptr = static_cast<int32_t*>(data->data);

  size_t workspace_size = 0;
  CUSPARSE_CALL(hipsparseXcoosort_bufferSizeExt(
      thr_entry->cusparse_handle, coo.num_rows, coo.num_cols, row->shape[0],
      data_ptr, row_ptr, &workspace_size));
  void* workspace = device->AllocWorkspace(row->ctx, workspace_size);
  CUSPARSE_CALL(hipsparseXcoosortByRow(
      thr_entry->cusparse_handle, coo.num_rows, coo.num_cols, row->shape[0],
      data_ptr, row_ptr, col_ptr, workspace));
  device->FreeWorkspace(row->ctx, workspace);

  // The row and column field have already been reordered according
  // to data, thus the data field will be deprecated.
  coo.data = aten::NullArray();
  coo.row_sorted = false;
  coo.col_sorted = false;
  return coo;
}

template <>
COOMatrix CSRToCOODataAsOrder<kDGLCUDA, int64_t>(CSRMatrix csr) {
  COOMatrix coo = CSRToCOO<kDGLCUDA, int64_t>(csr);
  if (aten::IsNullArray(coo.data)) return coo;
  const auto& sorted = Sort(coo.data);

  coo.row = IndexSelect(coo.row, sorted.second);
  coo.col = IndexSelect(coo.col, sorted.second);

  // The row and column field have already been reordered according
  // to data, thus the data field will be deprecated.
  coo.data = aten::NullArray();
  coo.row_sorted = false;
  coo.col_sorted = false;
  return coo;
}

template COOMatrix CSRToCOODataAsOrder<kDGLCUDA, int32_t>(CSRMatrix csr);
template COOMatrix CSRToCOODataAsOrder<kDGLCUDA, int64_t>(CSRMatrix csr);

}  // namespace impl
}  // namespace aten
}  // namespace dgl
