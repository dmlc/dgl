#include "hip/hip_runtime.h"
/**
 *  Copyright (c) 2020-2021 by Contributors
 * @file array/cuda/array_op_impl.cu
 * @brief Array operator GPU implementation
 */
#include <dgl/array.h>

#include "../../runtime/cuda/cuda_common.h"
#include "../../runtime/cuda/cuda_hashtable.cuh"
#include "../arith.h"
#include "./utils.h"

namespace dgl {
using runtime::NDArray;
using namespace runtime::cuda;
namespace aten {
namespace impl {

///////////////////////////// BinaryElewise /////////////////////////////

template <typename IdType, typename Op>
__global__ void _BinaryElewiseKernel(
    const IdType* lhs, const IdType* rhs, IdType* out, int64_t length) {
  int tx = blockIdx.x * blockDim.x + threadIdx.x;
  int stride_x = gridDim.x * blockDim.x;
  while (tx < length) {
    out[tx] = Op::Call(lhs[tx], rhs[tx]);
    tx += stride_x;
  }
}

template <DGLDeviceType XPU, typename IdType, typename Op>
IdArray BinaryElewise(IdArray lhs, IdArray rhs) {
  const int64_t len = lhs->shape[0];
  IdArray ret = NewIdArray(lhs->shape[0], lhs->ctx, lhs->dtype.bits);
  const IdType* lhs_data = static_cast<IdType*>(lhs->data);
  const IdType* rhs_data = static_cast<IdType*>(rhs->data);
  IdType* ret_data = static_cast<IdType*>(ret->data);
  hipStream_t stream = runtime::getCurrentCUDAStream();
  int nt = cuda::FindNumThreads(len);
  int nb = (len + nt - 1) / nt;
  CUDA_KERNEL_CALL(
      (_BinaryElewiseKernel<IdType, Op>), nb, nt, 0, stream, lhs_data, rhs_data,
      ret_data, len);
  return ret;
}

template IdArray BinaryElewise<kDGLCUDA, int32_t, arith::Add>(
    IdArray lhs, IdArray rhs);
template IdArray BinaryElewise<kDGLCUDA, int32_t, arith::Sub>(
    IdArray lhs, IdArray rhs);
template IdArray BinaryElewise<kDGLCUDA, int32_t, arith::Mul>(
    IdArray lhs, IdArray rhs);
template IdArray BinaryElewise<kDGLCUDA, int32_t, arith::Div>(
    IdArray lhs, IdArray rhs);
template IdArray BinaryElewise<kDGLCUDA, int32_t, arith::Mod>(
    IdArray lhs, IdArray rhs);
template IdArray BinaryElewise<kDGLCUDA, int32_t, arith::GT>(
    IdArray lhs, IdArray rhs);
template IdArray BinaryElewise<kDGLCUDA, int32_t, arith::LT>(
    IdArray lhs, IdArray rhs);
template IdArray BinaryElewise<kDGLCUDA, int32_t, arith::GE>(
    IdArray lhs, IdArray rhs);
template IdArray BinaryElewise<kDGLCUDA, int32_t, arith::LE>(
    IdArray lhs, IdArray rhs);
template IdArray BinaryElewise<kDGLCUDA, int32_t, arith::EQ>(
    IdArray lhs, IdArray rhs);
template IdArray BinaryElewise<kDGLCUDA, int32_t, arith::NE>(
    IdArray lhs, IdArray rhs);
template IdArray BinaryElewise<kDGLCUDA, int64_t, arith::Add>(
    IdArray lhs, IdArray rhs);
template IdArray BinaryElewise<kDGLCUDA, int64_t, arith::Sub>(
    IdArray lhs, IdArray rhs);
template IdArray BinaryElewise<kDGLCUDA, int64_t, arith::Mul>(
    IdArray lhs, IdArray rhs);
template IdArray BinaryElewise<kDGLCUDA, int64_t, arith::Div>(
    IdArray lhs, IdArray rhs);
template IdArray BinaryElewise<kDGLCUDA, int64_t, arith::Mod>(
    IdArray lhs, IdArray rhs);
template IdArray BinaryElewise<kDGLCUDA, int64_t, arith::GT>(
    IdArray lhs, IdArray rhs);
template IdArray BinaryElewise<kDGLCUDA, int64_t, arith::LT>(
    IdArray lhs, IdArray rhs);
template IdArray BinaryElewise<kDGLCUDA, int64_t, arith::GE>(
    IdArray lhs, IdArray rhs);
template IdArray BinaryElewise<kDGLCUDA, int64_t, arith::LE>(
    IdArray lhs, IdArray rhs);
template IdArray BinaryElewise<kDGLCUDA, int64_t, arith::EQ>(
    IdArray lhs, IdArray rhs);
template IdArray BinaryElewise<kDGLCUDA, int64_t, arith::NE>(
    IdArray lhs, IdArray rhs);

template <typename IdType, typename Op>
__global__ void _BinaryElewiseKernel(
    const IdType* lhs, IdType rhs, IdType* out, int64_t length) {
  int tx = blockIdx.x * blockDim.x + threadIdx.x;
  int stride_x = gridDim.x * blockDim.x;
  while (tx < length) {
    out[tx] = Op::Call(lhs[tx], rhs);
    tx += stride_x;
  }
}

template <DGLDeviceType XPU, typename IdType, typename Op>
IdArray BinaryElewise(IdArray lhs, IdType rhs) {
  const int64_t len = lhs->shape[0];
  IdArray ret = NewIdArray(lhs->shape[0], lhs->ctx, lhs->dtype.bits);
  const IdType* lhs_data = static_cast<IdType*>(lhs->data);
  IdType* ret_data = static_cast<IdType*>(ret->data);
  hipStream_t stream = runtime::getCurrentCUDAStream();
  int nt = cuda::FindNumThreads(len);
  int nb = (len + nt - 1) / nt;
  CUDA_KERNEL_CALL(
      (_BinaryElewiseKernel<IdType, Op>), nb, nt, 0, stream, lhs_data, rhs,
      ret_data, len);
  return ret;
}

template IdArray BinaryElewise<kDGLCUDA, int32_t, arith::Add>(
    IdArray lhs, int32_t rhs);
template IdArray BinaryElewise<kDGLCUDA, int32_t, arith::Sub>(
    IdArray lhs, int32_t rhs);
template IdArray BinaryElewise<kDGLCUDA, int32_t, arith::Mul>(
    IdArray lhs, int32_t rhs);
template IdArray BinaryElewise<kDGLCUDA, int32_t, arith::Div>(
    IdArray lhs, int32_t rhs);
template IdArray BinaryElewise<kDGLCUDA, int32_t, arith::Mod>(
    IdArray lhs, int32_t rhs);
template IdArray BinaryElewise<kDGLCUDA, int32_t, arith::GT>(
    IdArray lhs, int32_t rhs);
template IdArray BinaryElewise<kDGLCUDA, int32_t, arith::LT>(
    IdArray lhs, int32_t rhs);
template IdArray BinaryElewise<kDGLCUDA, int32_t, arith::GE>(
    IdArray lhs, int32_t rhs);
template IdArray BinaryElewise<kDGLCUDA, int32_t, arith::LE>(
    IdArray lhs, int32_t rhs);
template IdArray BinaryElewise<kDGLCUDA, int32_t, arith::EQ>(
    IdArray lhs, int32_t rhs);
template IdArray BinaryElewise<kDGLCUDA, int32_t, arith::NE>(
    IdArray lhs, int32_t rhs);
template IdArray BinaryElewise<kDGLCUDA, int64_t, arith::Add>(
    IdArray lhs, int64_t rhs);
template IdArray BinaryElewise<kDGLCUDA, int64_t, arith::Sub>(
    IdArray lhs, int64_t rhs);
template IdArray BinaryElewise<kDGLCUDA, int64_t, arith::Mul>(
    IdArray lhs, int64_t rhs);
template IdArray BinaryElewise<kDGLCUDA, int64_t, arith::Div>(
    IdArray lhs, int64_t rhs);
template IdArray BinaryElewise<kDGLCUDA, int64_t, arith::Mod>(
    IdArray lhs, int64_t rhs);
template IdArray BinaryElewise<kDGLCUDA, int64_t, arith::GT>(
    IdArray lhs, int64_t rhs);
template IdArray BinaryElewise<kDGLCUDA, int64_t, arith::LT>(
    IdArray lhs, int64_t rhs);
template IdArray BinaryElewise<kDGLCUDA, int64_t, arith::GE>(
    IdArray lhs, int64_t rhs);
template IdArray BinaryElewise<kDGLCUDA, int64_t, arith::LE>(
    IdArray lhs, int64_t rhs);
template IdArray BinaryElewise<kDGLCUDA, int64_t, arith::EQ>(
    IdArray lhs, int64_t rhs);
template IdArray BinaryElewise<kDGLCUDA, int64_t, arith::NE>(
    IdArray lhs, int64_t rhs);

template <typename IdType, typename Op>
__global__ void _BinaryElewiseKernel(
    IdType lhs, const IdType* rhs, IdType* out, int64_t length) {
  int tx = blockIdx.x * blockDim.x + threadIdx.x;
  int stride_x = gridDim.x * blockDim.x;
  while (tx < length) {
    out[tx] = Op::Call(lhs, rhs[tx]);
    tx += stride_x;
  }
}

template <DGLDeviceType XPU, typename IdType, typename Op>
IdArray BinaryElewise(IdType lhs, IdArray rhs) {
  const int64_t len = rhs->shape[0];
  IdArray ret = NewIdArray(rhs->shape[0], rhs->ctx, rhs->dtype.bits);
  const IdType* rhs_data = static_cast<IdType*>(rhs->data);
  IdType* ret_data = static_cast<IdType*>(ret->data);
  hipStream_t stream = runtime::getCurrentCUDAStream();
  int nt = cuda::FindNumThreads(len);
  int nb = (len + nt - 1) / nt;
  CUDA_KERNEL_CALL(
      (_BinaryElewiseKernel<IdType, Op>), nb, nt, 0, stream, lhs, rhs_data,
      ret_data, len);
  return ret;
}

template IdArray BinaryElewise<kDGLCUDA, int32_t, arith::Add>(
    int32_t lhs, IdArray rhs);
template IdArray BinaryElewise<kDGLCUDA, int32_t, arith::Sub>(
    int32_t lhs, IdArray rhs);
template IdArray BinaryElewise<kDGLCUDA, int32_t, arith::Mul>(
    int32_t lhs, IdArray rhs);
template IdArray BinaryElewise<kDGLCUDA, int32_t, arith::Div>(
    int32_t lhs, IdArray rhs);
template IdArray BinaryElewise<kDGLCUDA, int32_t, arith::Mod>(
    int32_t lhs, IdArray rhs);
template IdArray BinaryElewise<kDGLCUDA, int32_t, arith::GT>(
    int32_t lhs, IdArray rhs);
template IdArray BinaryElewise<kDGLCUDA, int32_t, arith::LT>(
    int32_t lhs, IdArray rhs);
template IdArray BinaryElewise<kDGLCUDA, int32_t, arith::GE>(
    int32_t lhs, IdArray rhs);
template IdArray BinaryElewise<kDGLCUDA, int32_t, arith::LE>(
    int32_t lhs, IdArray rhs);
template IdArray BinaryElewise<kDGLCUDA, int32_t, arith::EQ>(
    int32_t lhs, IdArray rhs);
template IdArray BinaryElewise<kDGLCUDA, int32_t, arith::NE>(
    int32_t lhs, IdArray rhs);
template IdArray BinaryElewise<kDGLCUDA, int64_t, arith::Add>(
    int64_t lhs, IdArray rhs);
template IdArray BinaryElewise<kDGLCUDA, int64_t, arith::Sub>(
    int64_t lhs, IdArray rhs);
template IdArray BinaryElewise<kDGLCUDA, int64_t, arith::Mul>(
    int64_t lhs, IdArray rhs);
template IdArray BinaryElewise<kDGLCUDA, int64_t, arith::Div>(
    int64_t lhs, IdArray rhs);
template IdArray BinaryElewise<kDGLCUDA, int64_t, arith::Mod>(
    int64_t lhs, IdArray rhs);
template IdArray BinaryElewise<kDGLCUDA, int64_t, arith::GT>(
    int64_t lhs, IdArray rhs);
template IdArray BinaryElewise<kDGLCUDA, int64_t, arith::LT>(
    int64_t lhs, IdArray rhs);
template IdArray BinaryElewise<kDGLCUDA, int64_t, arith::GE>(
    int64_t lhs, IdArray rhs);
template IdArray BinaryElewise<kDGLCUDA, int64_t, arith::LE>(
    int64_t lhs, IdArray rhs);
template IdArray BinaryElewise<kDGLCUDA, int64_t, arith::EQ>(
    int64_t lhs, IdArray rhs);
template IdArray BinaryElewise<kDGLCUDA, int64_t, arith::NE>(
    int64_t lhs, IdArray rhs);

template <typename IdType, typename Op>
__global__ void _UnaryElewiseKernel(
    const IdType* lhs, IdType* out, int64_t length) {
  int tx = blockIdx.x * blockDim.x + threadIdx.x;
  int stride_x = gridDim.x * blockDim.x;
  while (tx < length) {
    out[tx] = Op::Call(lhs[tx]);
    tx += stride_x;
  }
}

template <DGLDeviceType XPU, typename IdType, typename Op>
IdArray UnaryElewise(IdArray lhs) {
  const int64_t len = lhs->shape[0];
  IdArray ret = NewIdArray(lhs->shape[0], lhs->ctx, lhs->dtype.bits);
  const IdType* lhs_data = static_cast<IdType*>(lhs->data);
  IdType* ret_data = static_cast<IdType*>(ret->data);
  hipStream_t stream = runtime::getCurrentCUDAStream();
  int nt = cuda::FindNumThreads(len);
  int nb = (len + nt - 1) / nt;
  CUDA_KERNEL_CALL(
      (_UnaryElewiseKernel<IdType, Op>), nb, nt, 0, stream, lhs_data, ret_data,
      len);
  return ret;
}

template IdArray UnaryElewise<kDGLCUDA, int32_t, arith::Neg>(IdArray lhs);
template IdArray UnaryElewise<kDGLCUDA, int64_t, arith::Neg>(IdArray lhs);

///////////////////////////// Full /////////////////////////////

template <typename DType>
__global__ void _FullKernel(DType* out, int64_t length, DType val) {
  int tx = blockIdx.x * blockDim.x + threadIdx.x;
  int stride_x = gridDim.x * blockDim.x;
  while (tx < length) {
    out[tx] = val;
    tx += stride_x;
  }
}

template <DGLDeviceType XPU, typename DType>
NDArray Full(DType val, int64_t length, DGLContext ctx) {
  NDArray ret = NDArray::Empty({length}, DGLDataTypeTraits<DType>::dtype, ctx);
  DType* ret_data = static_cast<DType*>(ret->data);
  hipStream_t stream = runtime::getCurrentCUDAStream();
  int nt = cuda::FindNumThreads(length);
  int nb = (length + nt - 1) / nt;
  CUDA_KERNEL_CALL(
      (_FullKernel<DType>), nb, nt, 0, stream, ret_data, length, val);
  return ret;
}

template IdArray Full<kDGLCUDA, int32_t>(
    int32_t val, int64_t length, DGLContext ctx);
template IdArray Full<kDGLCUDA, int64_t>(
    int64_t val, int64_t length, DGLContext ctx);
template IdArray Full<kDGLCUDA, __half>(
    __half val, int64_t length, DGLContext ctx);
template IdArray Full<kDGLCUDA, __hip_bfloat16>(
    __hip_bfloat16 val, int64_t length, DGLContext ctx);
template IdArray Full<kDGLCUDA, float>(
    float val, int64_t length, DGLContext ctx);
template IdArray Full<kDGLCUDA, double>(
    double val, int64_t length, DGLContext ctx);

///////////////////////////// Range /////////////////////////////

template <typename IdType>
__global__ void _RangeKernel(IdType* out, IdType low, IdType length) {
  int tx = blockIdx.x * blockDim.x + threadIdx.x;
  int stride_x = gridDim.x * blockDim.x;
  while (tx < length) {
    out[tx] = low + tx;
    tx += stride_x;
  }
}

template <DGLDeviceType XPU, typename IdType>
IdArray Range(IdType low, IdType high, DGLContext ctx) {
  CHECK(high >= low) << "high must be bigger than low";
  const IdType length = high - low;
  IdArray ret = NewIdArray(length, ctx, sizeof(IdType) * 8);
  if (length == 0) return ret;
  IdType* ret_data = static_cast<IdType*>(ret->data);
  hipStream_t stream = runtime::getCurrentCUDAStream();
  int nt = cuda::FindNumThreads(length);
  int nb = (length + nt - 1) / nt;
  CUDA_KERNEL_CALL(
      (_RangeKernel<IdType>), nb, nt, 0, stream, ret_data, low, length);
  return ret;
}

template IdArray Range<kDGLCUDA, int32_t>(int32_t, int32_t, DGLContext);
template IdArray Range<kDGLCUDA, int64_t>(int64_t, int64_t, DGLContext);

///////////////////////////// Relabel_ //////////////////////////////

template <typename IdType>
__global__ void _RelabelKernel(
    IdType* out, int64_t length, DeviceOrderedHashTable<IdType> table) {
  int tx = blockIdx.x * blockDim.x + threadIdx.x;
  int stride_x = gridDim.x * blockDim.x;

  while (tx < length) {
    out[tx] = table.Search(out[tx])->local;
    tx += stride_x;
  }
}

template <DGLDeviceType XPU, typename IdType>
IdArray Relabel_(const std::vector<IdArray>& arrays) {
  IdArray all_nodes = Concat(arrays);
  const int64_t total_length = all_nodes->shape[0];

  if (total_length == 0) {
    return all_nodes;
  }

  const auto& ctx = arrays[0]->ctx;
  auto device = runtime::DeviceAPI::Get(ctx);
  hipStream_t stream = runtime::getCurrentCUDAStream();

  // build node maps and get the induced nodes
  OrderedHashTable<IdType> node_map(total_length, ctx, stream);
  int64_t num_induced = 0;
  int64_t* num_induced_device =
      static_cast<int64_t*>(device->AllocWorkspace(ctx, sizeof(int64_t)));
  IdArray induced_nodes = NewIdArray(total_length, ctx, sizeof(IdType) * 8);

  CUDA_CALL(hipMemsetAsync(
      num_induced_device, 0, sizeof(*num_induced_device), stream));

  node_map.FillWithDuplicates(
      all_nodes.Ptr<IdType>(), all_nodes->shape[0], induced_nodes.Ptr<IdType>(),
      num_induced_device, stream);
  // copy using the internal current stream
  device->CopyDataFromTo(
      num_induced_device, 0, &num_induced, 0, sizeof(num_induced), ctx,
      DGLContext{kDGLCPU, 0}, DGLDataType{kDGLInt, 64, 1});

  device->StreamSync(ctx, stream);
  device->FreeWorkspace(ctx, num_induced_device);

  // resize the induced nodes
  induced_nodes->shape[0] = num_induced;

  // relabel
  const int nt = 128;
  for (IdArray arr : arrays) {
    const int64_t length = arr->shape[0];
    int nb = (length + nt - 1) / nt;
    CUDA_KERNEL_CALL(
        (_RelabelKernel<IdType>), nb, nt, 0, stream, arr.Ptr<IdType>(), length,
        node_map.DeviceHandle());
  }

  return induced_nodes;
}

template IdArray Relabel_<kDGLCUDA, int32_t>(
    const std::vector<IdArray>& arrays);
template IdArray Relabel_<kDGLCUDA, int64_t>(
    const std::vector<IdArray>& arrays);

///////////////////////////// AsNumBits /////////////////////////////

template <typename InType, typename OutType>
__global__ void _CastKernel(const InType* in, OutType* out, size_t length) {
  int tx = blockIdx.x * blockDim.x + threadIdx.x;
  int stride_x = gridDim.x * blockDim.x;
  while (tx < length) {
    out[tx] = in[tx];
    tx += stride_x;
  }
}

template <DGLDeviceType XPU, typename IdType>
IdArray AsNumBits(IdArray arr, uint8_t bits) {
  const std::vector<int64_t> shape(arr->shape, arr->shape + arr->ndim);
  IdArray ret = IdArray::Empty(shape, DGLDataType{kDGLInt, bits, 1}, arr->ctx);
  const int64_t length = ret.NumElements();
  hipStream_t stream = runtime::getCurrentCUDAStream();
  int nt = cuda::FindNumThreads(length);
  int nb = (length + nt - 1) / nt;
  if (bits == 32) {
    CUDA_KERNEL_CALL(
        (_CastKernel<IdType, int32_t>), nb, nt, 0, stream,
        static_cast<IdType*>(arr->data), static_cast<int32_t*>(ret->data),
        length);
  } else {
    CUDA_KERNEL_CALL(
        (_CastKernel<IdType, int64_t>), nb, nt, 0, stream,
        static_cast<IdType*>(arr->data), static_cast<int64_t*>(ret->data),
        length);
  }
  return ret;
}

template IdArray AsNumBits<kDGLCUDA, int32_t>(IdArray arr, uint8_t bits);
template IdArray AsNumBits<kDGLCUDA, int64_t>(IdArray arr, uint8_t bits);

}  // namespace impl
}  // namespace aten
}  // namespace dgl
