#include "hip/hip_runtime.h"
/**
 *  Copyright (c) 2020 by Contributors
 * @file array/cuda/spmm.cu
 * @brief SpGEAM C APIs and definitions.
 */
#include <dgl/array.h>
#include <dgl/runtime/device_api.h>

#include "../../runtime/cuda/cuda_common.h"
#include "./cusparse_dispatcher.cuh"
#include "./functor.cuh"

namespace dgl {

using namespace dgl::runtime;

namespace aten {
namespace cusparse {

/** Cusparse implementation of SpSum on Csr format. */
template <typename DType, typename IdType>
std::pair<CSRMatrix, NDArray> CusparseCsrgeam2(
    const CSRMatrix& A, const NDArray A_weights_array, const CSRMatrix& B,
    const NDArray B_weights_array) {
  const int m = A.num_rows;
  const int n = A.num_cols;
  const int nnzA = A.indices->shape[0];
  const int nnzB = B.indices->shape[0];
  int nnzC;
  const DType alpha = 1.0;
  const DType beta = 1.0;
  auto ctx = A.indptr->ctx;
  auto device = runtime::DeviceAPI::Get(ctx);
  auto* thr_entry = runtime::CUDAThreadEntry::ThreadLocal();
  hipStream_t stream = runtime::getCurrentCUDAStream();
  const DType* A_weights = A_weights_array.Ptr<DType>();
  const DType* B_weights = B_weights_array.Ptr<DType>();
  // allocate cusparse handle if needed
  if (!thr_entry->cusparse_handle)
    CUSPARSE_CALL(hipsparseCreate(&(thr_entry->cusparse_handle)));
  CUSPARSE_CALL(hipsparseSetStream(thr_entry->cusparse_handle, stream));

  hipsparseMatDescr_t matA, matB, matC;
  CUSPARSE_CALL(hipsparseCreateMatDescr(&matA));
  CUSPARSE_CALL(hipsparseCreateMatDescr(&matB));
  CUSPARSE_CALL(hipsparseCreateMatDescr(&matC));

  hipsparseSetPointerMode(
      thr_entry->cusparse_handle, HIPSPARSE_POINTER_MODE_HOST);
  size_t workspace_size = 0;
  /* prepare output C */
  IdArray dC_csrOffsets = IdArray::Empty({m + 1}, A.indptr->dtype, ctx);
  IdType* dC_csrOffsets_data = dC_csrOffsets.Ptr<IdType>();
  IdArray dC_columns;
  NDArray dC_weights;
  IdType* dC_columns_data = dC_columns.Ptr<IdType>();
  DType* dC_weights_data = dC_weights.Ptr<DType>();
  /* prepare buffer */
  CUSPARSE_CALL(CSRGEAM<DType>::bufferSizeExt(
      thr_entry->cusparse_handle, m, n, &alpha, matA, nnzA, A_weights,
      A.indptr.Ptr<IdType>(), A.indices.Ptr<IdType>(), &beta, matB, nnzB,
      B_weights, B.indptr.Ptr<IdType>(), B.indices.Ptr<IdType>(), matC,
      dC_weights_data, dC_csrOffsets_data, dC_columns_data, &workspace_size));

  void* workspace = device->AllocWorkspace(ctx, workspace_size);
  CUSPARSE_CALL(CSRGEAM<DType>::nnz(
      thr_entry->cusparse_handle, m, n, matA, nnzA, A.indptr.Ptr<IdType>(),
      A.indices.Ptr<IdType>(), matB, nnzB, B.indptr.Ptr<IdType>(),
      B.indices.Ptr<IdType>(), matC, dC_csrOffsets_data, &nnzC, workspace));

  dC_columns = IdArray::Empty({nnzC}, A.indptr->dtype, ctx);
  dC_weights = NDArray::Empty({nnzC}, A_weights_array->dtype, ctx);
  dC_columns_data = dC_columns.Ptr<IdType>();
  dC_weights_data = dC_weights.Ptr<DType>();

  CUSPARSE_CALL(CSRGEAM<DType>::compute(
      thr_entry->cusparse_handle, m, n, &alpha, matA, nnzA, A_weights,
      A.indptr.Ptr<IdType>(), A.indices.Ptr<IdType>(), &beta, matB, nnzB,
      B_weights, B.indptr.Ptr<IdType>(), B.indices.Ptr<IdType>(), matC,
      dC_weights_data, dC_csrOffsets_data, dC_columns_data, workspace));

  device->FreeWorkspace(ctx, workspace);
  // destroy matrix/vector descriptors
  CUSPARSE_CALL(hipsparseDestroyMatDescr(matA));
  CUSPARSE_CALL(hipsparseDestroyMatDescr(matB));
  CUSPARSE_CALL(hipsparseDestroyMatDescr(matC));
  return {
      CSRMatrix(
          A.num_rows, A.num_cols, dC_csrOffsets, dC_columns,
          NullArray(dC_csrOffsets->dtype, dC_csrOffsets->ctx), true),
      dC_weights};
}
}  // namespace cusparse

template <int XPU, typename IdType, typename DType>
std::pair<CSRMatrix, NDArray> CSRSum(
    const std::vector<CSRMatrix>& As, const std::vector<NDArray>& A_weights) {
  const int64_t M = As[0].num_rows;
  const int64_t N = As[0].num_cols;
  const int64_t n = As.size();

  // Cast 64 bit indices to 32 bit
  std::vector<CSRMatrix> newAs;
  newAs.reserve(n);
  bool cast = false;
  if (As[0].indptr->dtype.bits == 64) {
    for (int i = 0; i < n; ++i)
      newAs.emplace_back(
          As[i].num_rows, As[i].num_cols, AsNumBits(As[i].indptr, 32),
          AsNumBits(As[i].indices, 32), AsNumBits(As[i].data, 32));
    cast = true;
  } else {
    for (int i = 0; i < n; ++i) newAs.push_back(As[i]);
  }

  // cuSPARSE csrgeam2 requires the CSR to be sorted.
  // TODO(BarclayII): ideally the sorted CSR should be cached but I'm not sure
  // how to do it.
  for (int i = 0; i < n; ++i) {
    if (!newAs[i].sorted) newAs[i] = CSRSort(newAs[i]);
  }

  // Reorder weights if A[i] has edge IDs
  std::vector<NDArray> A_weights_reordered(n);
  for (int i = 0; i < n; ++i) {
    if (CSRHasData(newAs[i]))
      A_weights_reordered[i] = IndexSelect(A_weights[i], newAs[i].data);
    else
      A_weights_reordered[i] = A_weights[i];
  }

  // Loop and sum
  auto result = std::make_pair(
      CSRMatrix(
          newAs[0].num_rows, newAs[0].num_cols, newAs[0].indptr,
          newAs[0].indices,
          NullArray(newAs[0].indptr->dtype, newAs[0].indptr->ctx)),
      A_weights_reordered[0]);  // Weights already reordered so we don't need
                                // As[0].data
  for (int64_t i = 1; i < n; ++i)
    result = cusparse::CusparseCsrgeam2<DType, int32_t>(
        result.first, result.second, newAs[i], A_weights_reordered[i]);

  // Cast 32 bit indices back to 64 bit if necessary
  if (cast) {
    CSRMatrix C = result.first;
    return {
        CSRMatrix(
            C.num_rows, C.num_cols, AsNumBits(C.indptr, 64),
            AsNumBits(C.indices, 64), AsNumBits(C.data, 64), true),
        result.second};
  } else {
    return result;
  }
}

template std::pair<CSRMatrix, NDArray> CSRSum<kDGLCUDA, int32_t, __half>(
    const std::vector<CSRMatrix>&, const std::vector<NDArray>&);
template std::pair<CSRMatrix, NDArray> CSRSum<kDGLCUDA, int64_t, __half>(
    const std::vector<CSRMatrix>&, const std::vector<NDArray>&);
template std::pair<CSRMatrix, NDArray> CSRSum<kDGLCUDA, int32_t, __hip_bfloat16>(
    const std::vector<CSRMatrix>&, const std::vector<NDArray>&);
template std::pair<CSRMatrix, NDArray> CSRSum<kDGLCUDA, int64_t, __hip_bfloat16>(
    const std::vector<CSRMatrix>&, const std::vector<NDArray>&);
template std::pair<CSRMatrix, NDArray> CSRSum<kDGLCUDA, int32_t, float>(
    const std::vector<CSRMatrix>&, const std::vector<NDArray>&);
template std::pair<CSRMatrix, NDArray> CSRSum<kDGLCUDA, int64_t, float>(
    const std::vector<CSRMatrix>&, const std::vector<NDArray>&);
template std::pair<CSRMatrix, NDArray> CSRSum<kDGLCUDA, int32_t, double>(
    const std::vector<CSRMatrix>&, const std::vector<NDArray>&);
template std::pair<CSRMatrix, NDArray> CSRSum<kDGLCUDA, int64_t, double>(
    const std::vector<CSRMatrix>&, const std::vector<NDArray>&);

}  // namespace aten
}  // namespace dgl
