#include "hip/hip_runtime.h"
/**
 *  Copyright (c) 2020 by Contributors
 * @file array/cuda/csr_sort.cc
 * @brief Sort CSR index
 */
#include <dgl/array.h>

#include <hipcub/hipcub.hpp>

#include "../../runtime/cuda/cuda_common.h"
#include "./utils.h"

namespace dgl {

using runtime::NDArray;

namespace aten {
namespace impl {

/**
 * @brief Check whether each row is sorted.
 */
template <typename IdType>
__global__ void _SegmentIsSorted(
    const IdType* indptr, const IdType* indices, int64_t num_rows,
    int8_t* flags) {
  int tx = blockIdx.x * blockDim.x + threadIdx.x;
  const int stride_x = gridDim.x * blockDim.x;
  while (tx < num_rows) {
    bool f = true;
    for (IdType i = indptr[tx] + 1; f && i < indptr[tx + 1]; ++i) {
      f = (indices[i - 1] <= indices[i]);
    }
    flags[tx] = static_cast<int8_t>(f);
    tx += stride_x;
  }
}

template <DGLDeviceType XPU, typename IdType>
bool CSRIsSorted(CSRMatrix csr) {
  const auto& ctx = csr.indptr->ctx;
  hipStream_t stream = runtime::getCurrentCUDAStream();
  auto device = runtime::DeviceAPI::Get(ctx);
  // We allocate a workspace of num_rows bytes. It wastes a little bit memory
  // but should be fine.
  int8_t* flags =
      static_cast<int8_t*>(device->AllocWorkspace(ctx, csr.num_rows));
  const int nt = cuda::FindNumThreads(csr.num_rows);
  const int nb = (csr.num_rows + nt - 1) / nt;
  CUDA_KERNEL_CALL(
      _SegmentIsSorted, nb, nt, 0, stream, csr.indptr.Ptr<IdType>(),
      csr.indices.Ptr<IdType>(), csr.num_rows, flags);
  bool ret = cuda::AllTrue(flags, csr.num_rows, ctx);
  device->FreeWorkspace(ctx, flags);
  return ret;
}

template bool CSRIsSorted<kDGLCUDA, int32_t>(CSRMatrix csr);
template bool CSRIsSorted<kDGLCUDA, int64_t>(CSRMatrix csr);

template <DGLDeviceType XPU, typename IdType>
void CSRSort_(CSRMatrix* csr) {
  LOG(FATAL) << "Unreachable codes";
}

template <>
void CSRSort_<kDGLCUDA, int32_t>(CSRMatrix* csr) {
  auto* thr_entry = runtime::CUDAThreadEntry::ThreadLocal();
  auto device = runtime::DeviceAPI::Get(csr->indptr->ctx);
  hipStream_t stream = runtime::getCurrentCUDAStream();
  // allocate cusparse handle if needed
  if (!thr_entry->cusparse_handle) {
    CUSPARSE_CALL(hipsparseCreate(&(thr_entry->cusparse_handle)));
  }
  CUSPARSE_CALL(hipsparseSetStream(thr_entry->cusparse_handle, stream));

  NDArray indptr = csr->indptr;
  NDArray indices = csr->indices;
  const auto& ctx = indptr->ctx;
  const int64_t nnz = indices->shape[0];
  if (!aten::CSRHasData(*csr))
    csr->data = aten::Range(0, nnz, indices->dtype.bits, ctx);
  NDArray data = csr->data;

  size_t workspace_size = 0;
  CUSPARSE_CALL(hipsparseXcsrsort_bufferSizeExt(
      thr_entry->cusparse_handle, csr->num_rows, csr->num_cols, nnz,
      indptr.Ptr<int32_t>(), indices.Ptr<int32_t>(), &workspace_size));
  void* workspace = device->AllocWorkspace(ctx, workspace_size);

  hipsparseMatDescr_t descr;
  CUSPARSE_CALL(hipsparseCreateMatDescr(&descr));
  CUSPARSE_CALL(hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL));
  CUSPARSE_CALL(hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO));
  CUSPARSE_CALL(hipsparseXcsrsort(
      thr_entry->cusparse_handle, csr->num_rows, csr->num_cols, nnz, descr,
      indptr.Ptr<int32_t>(), indices.Ptr<int32_t>(), data.Ptr<int32_t>(),
      workspace));

  csr->sorted = true;

  // free resources
  CUSPARSE_CALL(hipsparseDestroyMatDescr(descr));
  device->FreeWorkspace(ctx, workspace);
}

template <>
void CSRSort_<kDGLCUDA, int64_t>(CSRMatrix* csr) {
  hipStream_t stream = runtime::getCurrentCUDAStream();
  auto device = runtime::DeviceAPI::Get(csr->indptr->ctx);

  const auto& ctx = csr->indptr->ctx;
  const int64_t nnz = csr->indices->shape[0];
  const auto nbits = csr->indptr->dtype.bits;
  if (!aten::CSRHasData(*csr)) csr->data = aten::Range(0, nnz, nbits, ctx);

  IdArray new_indices = csr->indices.Clone();
  IdArray new_data = csr->data.Clone();

  const int64_t* offsets = csr->indptr.Ptr<int64_t>();
  const int64_t* key_in = csr->indices.Ptr<int64_t>();
  int64_t* key_out = new_indices.Ptr<int64_t>();
  const int64_t* value_in = csr->data.Ptr<int64_t>();
  int64_t* value_out = new_data.Ptr<int64_t>();

  // Allocate workspace
  size_t workspace_size = 0;
  CUDA_CALL(hipcub::DeviceSegmentedRadixSort::SortPairs(
      nullptr, workspace_size, key_in, key_out, value_in, value_out, nnz,
      csr->num_rows, offsets, offsets + 1, 0, sizeof(int64_t) * 8, stream));
  void* workspace = device->AllocWorkspace(ctx, workspace_size);

  // Compute
  CUDA_CALL(hipcub::DeviceSegmentedRadixSort::SortPairs(
      workspace, workspace_size, key_in, key_out, value_in, value_out, nnz,
      csr->num_rows, offsets, offsets + 1, 0, sizeof(int64_t) * 8, stream));

  csr->sorted = true;
  csr->indices = new_indices;
  csr->data = new_data;

  // free resources
  device->FreeWorkspace(ctx, workspace);
}

template void CSRSort_<kDGLCUDA, int32_t>(CSRMatrix* csr);
template void CSRSort_<kDGLCUDA, int64_t>(CSRMatrix* csr);

}  // namespace impl
}  // namespace aten
}  // namespace dgl
