#include "hip/hip_runtime.h"
/**
 *  Copyright (c) 2019 by Contributors
 * @file array/cpu/array_index_select.cu
 * @brief Array index select GPU implementation
 */
#include <dgl/array.h>

#include "../../runtime/cuda/cuda_common.h"
#include "./array_index_select.cuh"
#include "./utils.h"

namespace dgl {
using runtime::NDArray;
namespace aten {
namespace impl {

template <DGLDeviceType XPU, typename DType, typename IdType>
NDArray IndexSelect(NDArray array, IdArray index) {
  const int64_t arr_len = array->shape[0];
  const int64_t len = index->shape[0];
  int64_t num_feat = 1;
  std::vector<int64_t> shape{len};
  for (int d = 1; d < array->ndim; ++d) {
    num_feat *= array->shape[d];
    shape.emplace_back(array->shape[d]);
  }

  // use index->ctx for pinned array
  NDArray ret = NDArray::Empty(shape, array->dtype, index->ctx);
  if (len == 0 || arr_len * num_feat == 0) return ret;
  DType* ret_data = static_cast<DType*>(ret->data);

  const DType* array_data = static_cast<DType*>(cuda::GetDevicePointer(array));
  const IdType* idx_data = static_cast<IdType*>(index->data);

  hipStream_t stream = runtime::getCurrentCUDAStream();
  if (num_feat == 1) {
    const int nt = cuda::FindNumThreads(len);
    const int nb = (len + nt - 1) / nt;
    CUDA_KERNEL_CALL(
        IndexSelectSingleKernel, nb, nt, 0, stream, array_data, idx_data, len,
        arr_len, ret_data);
  } else {
    dim3 block(256, 1);
    while (static_cast<int64_t>(block.x) >= 2 * num_feat) {
      block.x /= 2;
      block.y *= 2;
    }
    const dim3 grid((len + block.y - 1) / block.y);
    CUDA_KERNEL_CALL(
        IndexSelectMultiKernel, grid, block, 0, stream, array_data, num_feat,
        idx_data, len, arr_len, ret_data);
  }
  return ret;
}

template NDArray IndexSelect<kDGLCUDA, int32_t, int32_t>(NDArray, IdArray);
template NDArray IndexSelect<kDGLCUDA, int32_t, int64_t>(NDArray, IdArray);
template NDArray IndexSelect<kDGLCUDA, int64_t, int32_t>(NDArray, IdArray);
template NDArray IndexSelect<kDGLCUDA, int64_t, int64_t>(NDArray, IdArray);
template NDArray IndexSelect<kDGLCUDA, __half, int32_t>(NDArray, IdArray);
template NDArray IndexSelect<kDGLCUDA, __half, int64_t>(NDArray, IdArray);
template NDArray IndexSelect<kDGLCUDA, __hip_bfloat16, int32_t>(
    NDArray, IdArray);
template NDArray IndexSelect<kDGLCUDA, __hip_bfloat16, int64_t>(
    NDArray, IdArray);
template NDArray IndexSelect<kDGLCUDA, float, int32_t>(NDArray, IdArray);
template NDArray IndexSelect<kDGLCUDA, float, int64_t>(NDArray, IdArray);
template NDArray IndexSelect<kDGLCUDA, double, int32_t>(NDArray, IdArray);
template NDArray IndexSelect<kDGLCUDA, double, int64_t>(NDArray, IdArray);

template <DGLDeviceType XPU, typename DType>
DType IndexSelect(NDArray array, int64_t index) {
  auto device = runtime::DeviceAPI::Get(array->ctx);
  DType ret = static_cast<DType>(0.0f);
  device->CopyDataFromTo(
      static_cast<DType*>(array->data) + index, 0, &ret, 0, sizeof(DType),
      array->ctx, DGLContext{kDGLCPU, 0}, array->dtype);
  return ret;
}

template int32_t IndexSelect<kDGLCUDA, int32_t>(NDArray array, int64_t index);
template int64_t IndexSelect<kDGLCUDA, int64_t>(NDArray array, int64_t index);
template uint32_t IndexSelect<kDGLCUDA, uint32_t>(NDArray array, int64_t index);
template uint64_t IndexSelect<kDGLCUDA, uint64_t>(NDArray array, int64_t index);
template __half IndexSelect<kDGLCUDA, __half>(NDArray array, int64_t index);
template __hip_bfloat16 IndexSelect<kDGLCUDA, __hip_bfloat16>(
    NDArray array, int64_t index);
template float IndexSelect<kDGLCUDA, float>(NDArray array, int64_t index);
template double IndexSelect<kDGLCUDA, double>(NDArray array, int64_t index);

}  // namespace impl
}  // namespace aten
}  // namespace dgl
