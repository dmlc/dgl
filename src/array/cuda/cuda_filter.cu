#include "hip/hip_runtime.h"
/**
 *  Copyright (c) 2021 by Contributors
 * @file array/cuda/cuda_filter.cc
 * @brief Object for selecting items in a set, or selecting items not in a set.
 */

#include <dgl/runtime/device_api.h>

#include <hipcub/hipcub.hpp>

#include "../../runtime/cuda/cuda_common.h"
#include "../../runtime/cuda/cuda_hashtable.cuh"
#include "../filter.h"

using namespace dgl::runtime::cuda;

namespace dgl {
namespace array {

namespace {

template <typename IdType, bool include>
__global__ void _IsInKernel(
    DeviceOrderedHashTable<IdType> table, const IdType* const array,
    const int64_t size, IdType* const mark) {
  const int64_t idx = threadIdx.x + blockDim.x * blockIdx.x;
  if (idx < size) {
    mark[idx] = table.Contains(array[idx]) ^ (!include);
  }
}

template <typename IdType>
__global__ void _InsertKernel(
    const IdType* const prefix, const int64_t size, IdType* const result) {
  const int64_t idx = threadIdx.x + blockDim.x * blockIdx.x;
  if (idx < size) {
    if (prefix[idx] != prefix[idx + 1]) {
      result[prefix[idx]] = idx;
    }
  }
}

template <typename IdType, bool include>
IdArray _PerformFilter(const OrderedHashTable<IdType>& table, IdArray test) {
  const auto& ctx = test->ctx;
  auto device = runtime::DeviceAPI::Get(ctx);
  const int64_t size = test->shape[0];
  hipStream_t cudaStream = runtime::getCurrentCUDAStream();

  if (size == 0) {
    return test;
  }

  // we need two arrays: 1) to act as a prefixsum
  // for the number of entries that will be inserted, and
  // 2) to collect the included items.
  IdType* prefix = static_cast<IdType*>(
      device->AllocWorkspace(ctx, sizeof(IdType) * (size + 1)));

  // will resize down later
  IdArray result = aten::NewIdArray(size, ctx, sizeof(IdType) * 8);

  // mark each index based on it's existence in the hashtable
  {
    const dim3 block(256);
    const dim3 grid((size + block.x - 1) / block.x);

    CUDA_KERNEL_CALL(
        (_IsInKernel<IdType, include>), grid, block, 0, cudaStream,
        table.DeviceHandle(), static_cast<const IdType*>(test->data), size,
        prefix);
  }

  // generate prefix-sum
  {
    size_t workspace_bytes;
    CUDA_CALL(hipcub::DeviceScan::ExclusiveSum(
        nullptr, workspace_bytes, static_cast<IdType*>(nullptr),
        static_cast<IdType*>(nullptr), size + 1, cudaStream));
    void* workspace = device->AllocWorkspace(ctx, workspace_bytes);

    CUDA_CALL(hipcub::DeviceScan::ExclusiveSum(
        workspace, workspace_bytes, prefix, prefix, size + 1, cudaStream));
    device->FreeWorkspace(ctx, workspace);
  }

  // copy number using the internal current stream;
  IdType num_unique;
  device->CopyDataFromTo(
      prefix + size, 0, &num_unique, 0, sizeof(num_unique), ctx,
      DGLContext{kDGLCPU, 0}, test->dtype);

  // insert items into set
  {
    const dim3 block(256);
    const dim3 grid((size + block.x - 1) / block.x);

    CUDA_KERNEL_CALL(
        _InsertKernel, grid, block, 0, cudaStream, prefix, size,
        static_cast<IdType*>(result->data));
  }
  device->FreeWorkspace(ctx, prefix);

  return result.CreateView({num_unique}, result->dtype);
}

template <typename IdType>
class CudaFilterSet : public Filter {
 public:
  explicit CudaFilterSet(IdArray array)
      : table_(array->shape[0], array->ctx, runtime::getCurrentCUDAStream()) {
    hipStream_t cudaStream = runtime::getCurrentCUDAStream();
    table_.FillWithUnique(
        static_cast<const IdType*>(array->data), array->shape[0], cudaStream);
  }

  IdArray find_included_indices(IdArray test) override {
    return _PerformFilter<IdType, true>(table_, test);
  }

  IdArray find_excluded_indices(IdArray test) override {
    return _PerformFilter<IdType, false>(table_, test);
  }

 private:
  OrderedHashTable<IdType> table_;
};

}  // namespace

template <DGLDeviceType XPU, typename IdType>
FilterRef CreateSetFilter(IdArray set) {
  return FilterRef(std::make_shared<CudaFilterSet<IdType>>(set));
}

template FilterRef CreateSetFilter<kDGLCUDA, int32_t>(IdArray set);
template FilterRef CreateSetFilter<kDGLCUDA, int64_t>(IdArray set);

}  // namespace array
}  // namespace dgl
