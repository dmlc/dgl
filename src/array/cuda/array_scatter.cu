#include "hip/hip_runtime.h"
/**
 *  Copyright (c) 2019 by Contributors
 * @file array/cuda/array_scatter.cu
 * @brief Array scatter GPU implementation
 */
#include <dgl/array.h>

#include "../../runtime/cuda/cuda_common.h"
#include "./utils.h"

namespace dgl {
using runtime::NDArray;
namespace aten {
namespace impl {

template <typename DType, typename IdType>
__global__ void _ScatterKernel(
    const IdType* index, const DType* value, int64_t length, DType* out) {
  int tx = blockIdx.x * blockDim.x + threadIdx.x;
  int stride_x = gridDim.x * blockDim.x;
  while (tx < length) {
    out[index[tx]] = value[tx];
    tx += stride_x;
  }
}

template <DGLDeviceType XPU, typename DType, typename IdType>
void Scatter_(IdArray index, NDArray value, NDArray out) {
  const int64_t len = index->shape[0];
  const IdType* idx = index.Ptr<IdType>();
  const DType* val = value.Ptr<DType>();
  DType* outd = out.Ptr<DType>();

  hipStream_t stream = runtime::getCurrentCUDAStream();
  const int nt = cuda::FindNumThreads(len);
  const int nb = (len + nt - 1) / nt;
  CUDA_KERNEL_CALL(_ScatterKernel, nb, nt, 0, stream, idx, val, len, outd);
}

template void Scatter_<kDGLCUDA, int32_t, int32_t>(IdArray, NDArray, NDArray);
template void Scatter_<kDGLCUDA, int64_t, int32_t>(IdArray, NDArray, NDArray);
template void Scatter_<kDGLCUDA, __half, int32_t>(IdArray, NDArray, NDArray);
template void Scatter_<kDGLCUDA, __hip_bfloat16, int32_t>(
    IdArray, NDArray, NDArray);
template void Scatter_<kDGLCUDA, float, int32_t>(IdArray, NDArray, NDArray);
template void Scatter_<kDGLCUDA, double, int32_t>(IdArray, NDArray, NDArray);
template void Scatter_<kDGLCUDA, int32_t, int64_t>(IdArray, NDArray, NDArray);
template void Scatter_<kDGLCUDA, int64_t, int64_t>(IdArray, NDArray, NDArray);
template void Scatter_<kDGLCUDA, __half, int64_t>(IdArray, NDArray, NDArray);
template void Scatter_<kDGLCUDA, __hip_bfloat16, int64_t>(
    IdArray, NDArray, NDArray);
template void Scatter_<kDGLCUDA, float, int64_t>(IdArray, NDArray, NDArray);
template void Scatter_<kDGLCUDA, double, int64_t>(IdArray, NDArray, NDArray);

};  // namespace impl
};  // namespace aten
};  // namespace dgl
