#include "hip/hip_runtime.h"
/**
 *  Copyright (c) 2020 by Contributors
 * @file array/cuda/gather_mm.cu
 * @brief GatherMM C APIs and definitions.
 */
#include <dgl/array.h>

#include <algorithm>  // std::swap

#include "./atomic.cuh"
#include "./functor.cuh"
#include "./utils.h"

namespace dgl {
using namespace cuda;
namespace aten {

namespace {

/** @brief Call cuBLAS GEMM API for dense matmul operation for float and double.
 */
template <typename DType>
hipblasStatus_t cublasGemm(
    hipblasHandle_t handle, hipblasOperation_t transa, hipblasOperation_t transb,
    int m, int n, int k, const DType* alpha, const DType* A, int lda,
    const DType* B, int ldb, const DType* beta, DType* C, int ldc) {
  LOG(INFO) << "Not supported dtype";
  return HIPBLAS_STATUS_EXECUTION_FAILED;
}

template <>
hipblasStatus_t cublasGemm<__half>(
    hipblasHandle_t handle, hipblasOperation_t transa, hipblasOperation_t transb,
    int m, int n, int k, const __half* alpha, const __half* A, int lda,
    const __half* B, int ldb, const __half* beta, __half* C, int ldc) {
  return hipblasHgemm(
      handle, transa, transb, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);
}

template <>
hipblasStatus_t cublasGemm<__hip_bfloat16>(
    hipblasHandle_t handle, hipblasOperation_t transa, hipblasOperation_t transb,
    int m, int n, int k, const __hip_bfloat16* alpha, const __hip_bfloat16* A,
    int lda, const __hip_bfloat16* B, int ldb, const __hip_bfloat16* beta,
    __hip_bfloat16* C, int ldc) {
  float alpha_float = __bfloat162float(*alpha);
  float beta_float = __bfloat162float(*beta);
  return hipblasGemmEx(
      handle, transa, transb, m, n, k, &alpha_float, A, HIP_R_16BF, lda, B,
      HIP_R_16BF, ldb, &beta_float, C, HIP_R_16BF, ldc, HIPBLAS_COMPUTE_32F,
      CUBLAS_GEMM_DEFAULT_TENSOR_OP);
}

template <>
hipblasStatus_t cublasGemm<float>(
    hipblasHandle_t handle, hipblasOperation_t transa, hipblasOperation_t transb,
    int m, int n, int k, const float* alpha, const float* A, int lda,
    const float* B, int ldb, const float* beta, float* C, int ldc) {
  return hipblasSgemm(
      handle, transa, transb, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);
}

template <>
hipblasStatus_t cublasGemm<double>(
    hipblasHandle_t handle, hipblasOperation_t transa, hipblasOperation_t transb,
    int m, int n, int k, const double* alpha, const double* A, int lda,
    const double* B, int ldb, const double* beta, double* C, int ldc) {
  return hipblasDgemm(
      handle, transa, transb, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);
}

}  // namespace

namespace cuda {

/**
 * @note Each row of A multiplies a segment of matrix of B of dimension in_len *
 * outlen. One warp is assigned to process one row of A. Each WARP sequentially
 * multiplies one element of A and a row of B to compute partial result of the
 * output. A is loaded in shared memory in a coalesced way. Output matrix is
 * loaded in registers. B should get benefit from L2 cache.
 */
template <typename Idx, typename DType>
__global__ void GatherMMScatterKernel(
    const DType* __restrict__ A, const DType* __restrict__ B,
    DType* __restrict__ C, const Idx* __restrict__ idx_a,
    const Idx* __restrict__ idx_b, const Idx* __restrict__ idx_c,
    const int64_t num_rows, const int64_t in_len, const int64_t out_len) {
  unsigned int tId = threadIdx.x;
  unsigned int laneId = tId & 31;
  unsigned int gId = (blockIdx.x * blockDim.x + threadIdx.x);
  unsigned int warpId = gId >> 5;
  unsigned int row = warpId;
  if (row < num_rows) {
    const unsigned int local_row =
        row & 3;  // hardcoded for TB size 128 (4 warps)
    const Idx cur_rowA = (idx_a) ? idx_a[row] : row;
    const Idx cur_rowB = (idx_b) ? idx_b[row] : row;
    const Idx cur_rowC = (idx_c) ? idx_c[row] : row;
    const Idx B_offset = cur_rowB * in_len * out_len;
    const int sh_a_tile = 64;
    __shared__ DType sh_A[4 * sh_a_tile];
    int a_tile = sh_a_tile;
    for (unsigned int k_start = 0; k_start < in_len; k_start += 64) {
      if ((in_len - k_start) < a_tile) a_tile = in_len - k_start;
      // Load A in shared mem in a coalesced way
      for (unsigned int l = laneId; l < a_tile; l += 32)
        sh_A[local_row * sh_a_tile + l] = A[cur_rowA * in_len + (k_start + l)];
      __syncwarp();

      for (unsigned int outloop = 0; outloop < out_len; outloop += 32) {
        DType out_reg = static_cast<DType>(0.0f);  // thread private
        const unsigned int l = laneId;
        if (l < out_len) {
          // iterate over elements of a row of A
          for (unsigned int i = 0; i < a_tile; i++) {
            const DType a_val = sh_A[local_row * sh_a_tile + i];
            // iterate over elements of a row of B in parallel
            out_reg +=
                a_val * B[B_offset + ((i + k_start) * out_len + (outloop + l))];
          }
          if (idx_c) {
            AtomicAdd(C + cur_rowC * out_len + (outloop + l), out_reg);
          } else {
            C[cur_rowC * out_len + (outloop + l)] += out_reg;
          }
        }
      }
    }
  }
}

/**
 * @note Output matrix is accumulated via atomic operations. Rest of the
 * strategies are similar to GatherMMKernel. One warp is assigned to process one
 * row of A. Each WARP sequentially multiplies one element of A and a row of B
 * to compute partial result of the output. A is loaded in shared memory in a
 * coalesced way. B should get benefit from L2 cache.
 */
template <typename Idx, typename DType>
__global__ void GatherMMScatterKernel2(
    const DType* __restrict__ A, const DType* __restrict__ B,
    DType* __restrict__ C, const Idx* __restrict__ idx_a,
    const Idx* __restrict__ idx_b, const Idx* __restrict__ idx_c,
    const int64_t num_rows, const int64_t in_len, const int64_t out_len) {
  unsigned int tId = threadIdx.x;
  unsigned int laneId = tId & 31;
  unsigned int gId = (blockIdx.x * blockDim.x + threadIdx.x);
  unsigned int warpId = gId >> 5;
  unsigned int row = warpId;
  if (row < num_rows) {
    const unsigned int local_row =
        row & 3;  // hardcoded for TB size 128 (4 warps)
    const Idx row_a = (idx_a) ? idx_a[row] : row;
    const Idx row_b = (idx_b) ? idx_b[row] : row;
    const Idx row_c = (idx_c) ? idx_c[row] : row;
    const Idx C_offset = row_c * in_len * out_len;
    const int sh_a_tile = 64;
    __shared__ DType sh_A[4 * sh_a_tile];
    int a_tile = sh_a_tile;
    for (unsigned int k_start = 0; k_start < in_len; k_start += 64) {
      if ((in_len - k_start) < a_tile) a_tile = in_len - k_start;
      /* Load A in shared mem in a coalesced way */
      for (unsigned int l = laneId; l < a_tile; l += 32)
        sh_A[local_row * sh_a_tile + l] = A[row_a * in_len + (k_start + l)];
      __syncwarp();

      for (unsigned int outloop = 0; outloop < out_len; outloop += 32) {
        DType out_reg = static_cast<DType>(0.0f);  // thread private
        const unsigned int l = laneId;
        if (l < out_len) {
          const DType b_val = B[row_b * out_len + (outloop + l)];
          /* iterate over elements of a row of A */
          for (unsigned int i = 0; i < a_tile; i++) {
            const DType a_val = sh_A[local_row * sh_a_tile + i];
            const Idx C_idx =
                C_offset + ((i + k_start) * out_len + (outloop + l));
            AtomicAdd(C + C_idx, a_val * b_val);
          }
        }
      }
    }
  }
}

}  // namespace cuda

/**
 * @brief Implementation of Gather_mm operator. The input matrix A is
 *        expected to be sorted according to relation type.
 * @param A The input dense matrix of dimension m x k
 * @param B The input dense matrix of dimension k x n
 * @param C The output dense matrix of dimension m x n
 * @param seglen_A The input vector of size R. Each element
 *        is the length of segments of input ``A``
 * @param a_trans Matrix A to be transposed
 * @param b_trans Matrix B to be transposed
 */
template <int XPU, typename IdType, typename DType>
void SegmentMM(
    const NDArray A, const NDArray B, NDArray C, const NDArray seglen_A,
    bool a_trans, bool b_trans) {
  auto device = runtime::DeviceAPI::Get(A->ctx);
  hipStream_t stream = runtime::getCurrentCUDAStream();
  const DType* A_data = A.Ptr<DType>();
  const DType* B_data = B.Ptr<DType>();
  const IdType* seglen_A_data = seglen_A.Ptr<IdType>();
  DType* C_data = C.Ptr<DType>();
  int64_t A_offset = 0, B_offset = 0, C_offset = 0;
  int64_t m, n, k;
  int64_t num_rel = seglen_A.NumElements();
  DType alpha = 1., beta = 0.;

  auto* thr_entry = runtime::CUDAThreadEntry::ThreadLocal();
  if (!thr_entry->cublas_handle)
    CUBLAS_CALL(hipblasCreate(&(thr_entry->cublas_handle)));
  CUBLAS_CALL(hipblasSetStream(thr_entry->cublas_handle, stream));

  IdType m_offset = 0;
  for (IdType etype = 0; etype < num_rel; ++etype) {
    m = seglen_A_data[etype];  // rows of A
    CHECK_LE(m_offset + m, A->shape[0])
        << "Segment index out of bound of A->shape[0].";
    n = B->shape[2];  // cols of B
    k = B->shape[1];  // cols of A == rows of B
    int ldb = n, lda = k, ldc = n;
    hipblasOperation_t transB = HIPBLAS_OP_N;
    hipblasOperation_t transA = HIPBLAS_OP_N;
    if (b_trans) {
      transB = HIPBLAS_OP_T;
      ldb = n, lda = n, ldc = k;
      std::swap(n, k);
    }
    CUBLAS_CALL(cublasGemm<DType>(
        thr_entry->cublas_handle, transB, transA, n, m, k, &alpha,
        B_data + B_offset, ldb, A_data + A_offset, lda, &beta,
        C_data + C_offset, ldc));
    A_offset += m * k;
    B_offset += k * n;
    C_offset += m * n;
    m_offset += m;
  }
}

template <int XPU, typename IdType, typename DType>
void SegmentMMBackwardB(
    const NDArray A, const NDArray dC, NDArray dB, const NDArray seglen) {
  auto device = runtime::DeviceAPI::Get(A->ctx);
  hipStream_t stream = runtime::getCurrentCUDAStream();
  const DType* A_data = A.Ptr<DType>();
  const DType* dC_data = dC.Ptr<DType>();
  const IdType* seglen_data = seglen.Ptr<IdType>();
  DType* dB_data = dB.Ptr<DType>();
  int64_t A_offset = 0, dC_offset = 0, dB_offset = 0;
  int64_t m, n, k;
  int64_t num_rel = seglen.NumElements();
  DType alpha = 1., beta = 0.;

  auto* thr_entry = runtime::CUDAThreadEntry::ThreadLocal();
  if (!thr_entry->cublas_handle)
    CUBLAS_CALL(hipblasCreate(&(thr_entry->cublas_handle)));
  CUBLAS_CALL(hipblasSetStream(thr_entry->cublas_handle, stream));

  IdType k_offset = 0;
  for (IdType etype = 0; etype < num_rel; ++etype) {
    m = dC->shape[1];
    n = A->shape[1];
    k = seglen_data[etype];
    CHECK_LE(k_offset + k, A->shape[0])
        << "Segement index out of bound of A->shape[0].";
    int lddC = m, ldA = n, lddB = m;
    hipblasOperation_t trans_dC = HIPBLAS_OP_N;
    hipblasOperation_t trans_A = HIPBLAS_OP_T;
    CUBLAS_CALL(cublasGemm<DType>(
        thr_entry->cublas_handle, trans_dC, trans_A, m, n, k, &alpha,
        dC_data + dC_offset, lddC, A_data + A_offset, ldA, &beta,
        dB_data + dB_offset, lddB));
    dC_offset += m * k;
    A_offset += n * k;
    dB_offset += m * n;
    k_offset += k;
  }
}

/**
 * @brief Implementation of Gather_mm operator. The input matrix A is
 *        expected to be sorted according to relation type.
 * @param A The input dense matrix of dimension m x k
 * @param B The input dense matrix of dimension k x n
 * @param C The output dense matrix of dimension m x n
 * @param idx_a The input vector to gather left hand operand on
 * @param idx_b The input vector to gather right hand operand on
 */

template <int XPU, typename IdType, typename DType>
void GatherMM(
    const NDArray A, const NDArray B, NDArray C, const NDArray idx_a,
    const NDArray idx_b) {
  auto device = runtime::DeviceAPI::Get(A->ctx);
  hipStream_t stream = runtime::getCurrentCUDAStream();
  int64_t out_len = B->shape[2];  // cols of B
  int64_t in_len = A->shape[1];   // cols of A
  const int64_t tot_num_rows = A->shape[0];
  const int ntx = 128;
  const int warp_size = 32;
  const int nbx = ((tot_num_rows * warp_size + ntx - 1) / ntx);
  const dim3 nblks(nbx);
  const dim3 nthrs(ntx);
  CUDA_KERNEL_CALL(
      (cuda::GatherMMScatterKernel<IdType, DType>), nblks, nthrs, 0, stream,
      A.Ptr<DType>(), B.Ptr<DType>(), C.Ptr<DType>(), idx_a.Ptr<IdType>(),
      idx_b.Ptr<IdType>(), nullptr, tot_num_rows, in_len, out_len);
}

/**
 * @brief Implementation of Gather_mm operator. The input matrix A is
 *        expected to be sorted according to relation type.
 * @param A The input dense matrix of dimension m x k
 * @param B The input dense matrix of dimension k x n
 * @param C The output dense matrix of dimension m x n
 * @param idx_a The input vector to gather left hand operand on
 * @param idx_b The input vector to gather right hand operand on
 * @param idx_c The input vector to gather output operand on
 * @param num_rel The number of idx types in idx_b
 * @param a_trans Matrix A to be transposed
 * @param b_trans Matrix B to be transposed
 */
template <int XPU, typename IdType, typename DType>
void GatherMMScatter(
    const NDArray A, const NDArray B, NDArray C, const NDArray idx_a,
    const NDArray idx_b, const NDArray idx_c) {
  auto device = runtime::DeviceAPI::Get(A->ctx);
  hipStream_t stream = runtime::getCurrentCUDAStream();
  const IdType* idx_c_data = idx_c.Ptr<IdType>();
  int64_t out_len = (B->ndim == 2) ? B->shape[1] : B->shape[2];  // cols of B
  int64_t in_len = A->shape[1];                                  // cols of A
  int64_t tot_num_rows = A->shape[0];
  const int ntx = 128;
  const int warp_size = 32;
  const int nbx = ((tot_num_rows * warp_size + ntx - 1) / ntx);
  const dim3 nblks(nbx);
  const dim3 nthrs(ntx);
  if (B->ndim == 3) {
    CUDA_KERNEL_CALL(
        (cuda::GatherMMScatterKernel<IdType, DType>), nblks, nthrs, 0, stream,
        A.Ptr<DType>(), B.Ptr<DType>(), C.Ptr<DType>(), idx_a.Ptr<IdType>(),
        idx_b.Ptr<IdType>(), idx_c.Ptr<IdType>(), tot_num_rows, in_len,
        out_len);
  } else {
    // Custom kernel for W_grad[idx_c[i]] = H^T[i] * C.grad[i]
    // This kernel accesses rows of A in a transposed way w/o explicitly
    // converting A
    CUDA_KERNEL_CALL(
        (cuda::GatherMMScatterKernel2<IdType, DType>), nblks, nthrs, 0, stream,
        A.Ptr<DType>(), B.Ptr<DType>(), C.Ptr<DType>(), idx_a.Ptr<IdType>(),
        idx_b.Ptr<IdType>(), idx_c.Ptr<IdType>(), tot_num_rows, in_len,
        out_len);
  }
}

template void GatherMM<kDGLCUDA, int32_t, __half>(
    const NDArray A, const NDArray B, NDArray C, const NDArray idx_a,
    const NDArray idx_b);
template void GatherMM<kDGLCUDA, int64_t, __half>(
    const NDArray A, const NDArray B, NDArray C, const NDArray idx_a,
    const NDArray idx_b);
template void GatherMM<kDGLCUDA, int32_t, __hip_bfloat16>(
    const NDArray A, const NDArray B, NDArray C, const NDArray idx_a,
    const NDArray idx_b);
template void GatherMM<kDGLCUDA, int64_t, __hip_bfloat16>(
    const NDArray A, const NDArray B, NDArray C, const NDArray idx_a,
    const NDArray idx_b);
template void GatherMM<kDGLCUDA, int32_t, float>(
    const NDArray A, const NDArray B, NDArray C, const NDArray idx_a,
    const NDArray idx_b);
template void GatherMM<kDGLCUDA, int64_t, float>(
    const NDArray A, const NDArray B, NDArray C, const NDArray idx_a,
    const NDArray idx_b);
template void GatherMM<kDGLCUDA, int32_t, double>(
    const NDArray A, const NDArray B, NDArray C, const NDArray idx_a,
    const NDArray idx_b);
template void GatherMM<kDGLCUDA, int64_t, double>(
    const NDArray A, const NDArray B, NDArray C, const NDArray idx_a,
    const NDArray idx_b);

template void GatherMMScatter<kDGLCUDA, int32_t, __half>(
    const NDArray A, const NDArray B, NDArray C, const NDArray idx_a,
    const NDArray idx_b, const NDArray idx_c);
template void GatherMMScatter<kDGLCUDA, int64_t, __half>(
    const NDArray A, const NDArray B, NDArray C, const NDArray idx_a,
    const NDArray idx_b, const NDArray idx_c);
template void GatherMMScatter<kDGLCUDA, int32_t, __hip_bfloat16>(
    const NDArray A, const NDArray B, NDArray C, const NDArray idx_a,
    const NDArray idx_b, const NDArray idx_c);
template void GatherMMScatter<kDGLCUDA, int64_t, __hip_bfloat16>(
    const NDArray A, const NDArray B, NDArray C, const NDArray idx_a,
    const NDArray idx_b, const NDArray idx_c);
template void GatherMMScatter<kDGLCUDA, int32_t, float>(
    const NDArray A, const NDArray B, NDArray C, const NDArray idx_a,
    const NDArray idx_b, const NDArray idx_c);
template void GatherMMScatter<kDGLCUDA, int64_t, float>(
    const NDArray A, const NDArray B, NDArray C, const NDArray idx_a,
    const NDArray idx_b, const NDArray idx_c);
template void GatherMMScatter<kDGLCUDA, int32_t, double>(
    const NDArray A, const NDArray B, NDArray C, const NDArray idx_a,
    const NDArray idx_b, const NDArray idx_c);
template void GatherMMScatter<kDGLCUDA, int64_t, double>(
    const NDArray A, const NDArray B, NDArray C, const NDArray idx_a,
    const NDArray idx_b, const NDArray idx_c);

template void SegmentMM<kDGLCUDA, int32_t, __half>(
    const NDArray A, const NDArray B, NDArray C, const NDArray seglen_A,
    bool a_trans, bool b_trans);
template void SegmentMM<kDGLCUDA, int64_t, __half>(
    const NDArray A, const NDArray B, NDArray C, const NDArray seglen_A,
    bool a_trans, bool b_trans);
template void SegmentMM<kDGLCUDA, int32_t, __hip_bfloat16>(
    const NDArray A, const NDArray B, NDArray C, const NDArray seglen_A,
    bool a_trans, bool b_trans);
template void SegmentMM<kDGLCUDA, int64_t, __hip_bfloat16>(
    const NDArray A, const NDArray B, NDArray C, const NDArray seglen_A,
    bool a_trans, bool b_trans);
template void SegmentMM<kDGLCUDA, int32_t, float>(
    const NDArray A, const NDArray B, NDArray C, const NDArray seglen_A,
    bool a_trans, bool b_trans);
template void SegmentMM<kDGLCUDA, int64_t, float>(
    const NDArray A, const NDArray B, NDArray C, const NDArray seglen_A,
    bool a_trans, bool b_trans);
template void SegmentMM<kDGLCUDA, int32_t, double>(
    const NDArray A, const NDArray B, NDArray C, const NDArray seglen_A,
    bool a_trans, bool b_trans);
template void SegmentMM<kDGLCUDA, int64_t, double>(
    const NDArray A, const NDArray B, NDArray C, const NDArray seglen_A,
    bool a_trans, bool b_trans);

template void SegmentMMBackwardB<kDGLCUDA, int32_t, __half>(
    const NDArray A, const NDArray dC, NDArray dB, const NDArray seglen);
template void SegmentMMBackwardB<kDGLCUDA, int64_t, __half>(
    const NDArray A, const NDArray dC, NDArray dB, const NDArray seglen);
template void SegmentMMBackwardB<kDGLCUDA, int32_t, __hip_bfloat16>(
    const NDArray A, const NDArray dC, NDArray dB, const NDArray seglen);
template void SegmentMMBackwardB<kDGLCUDA, int64_t, __hip_bfloat16>(
    const NDArray A, const NDArray dC, NDArray dB, const NDArray seglen);
template void SegmentMMBackwardB<kDGLCUDA, int32_t, float>(
    const NDArray A, const NDArray dC, NDArray dB, const NDArray seglen);
template void SegmentMMBackwardB<kDGLCUDA, int64_t, float>(
    const NDArray A, const NDArray dC, NDArray dB, const NDArray seglen);
template void SegmentMMBackwardB<kDGLCUDA, int32_t, double>(
    const NDArray A, const NDArray dC, NDArray dB, const NDArray seglen);
template void SegmentMMBackwardB<kDGLCUDA, int64_t, double>(
    const NDArray A, const NDArray dC, NDArray dB, const NDArray seglen);

}  // namespace aten
}  // namespace dgl
